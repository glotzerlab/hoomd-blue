#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "hip/hip_runtime.h"
#include "PCNDAngleForceGPU.cuh"

#include "hoomd/TextureTools.h"
#include "hoomd/RNGIdentifiers.h"
#include "hoomd/RandomNumbers.h"

using namespace hoomd;

#include <assert.h>

// small number. cutoff for ignoring the angle as being ill defined.
#define SMALL Scalar(0.001)

/*! \file PCNDAngleForceGPU.cu
    \brief Defines GPU kernel code for calculating the PCND angle forces. Used by PCNDAngleForceComputeGPU.
*/

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
//! Texture for reading angle parameters
// scalar2_tex_t angle_params_tex;

//! Texture for reading angle CGCMM S-R parameters
// scalar2_tex_t angle_CGCMMsr_tex; // MISSING EPSILON!!! sigma=.x, rcut=.y

//! Texture for reading angle CGCMM Epsilon-pow/pref parameters
// scalar4_tex_t angle_CGCMMepow_tex; // now with EPSILON=.x, pow1=.y, pow2=.z, pref=.w

//! Kernel for caculating PCND angle forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param box Box dimensions for periodic boundary condition handling
    \param alist Angle data to use in calculating the forces
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
*/
__global__ void gpu_compute_PCND_angle_forces_kernel(Scalar4* d_force,
                                                     Scalar* d_virial,
						     const unsigned int* d_tag,
                                                     const size_t virial_pitch,
                                                     const unsigned int N,
                                                     const Scalar4* d_pos,
                                                     BoxDim box,
                                                     const group_storage<3>* alist,
                                                     const unsigned int* apos_list,
                                                     const unsigned int pitch,
                                                     const unsigned int* n_angles_list,
                                                     Scalar2* d_params,
                                                     Scalar2* d_PCNDsr,
                                                     uint16_t* d_PCNDepow,
                                                     uint64_t timestep,
                                                     //float* devData,
                                                     uint64_t PCNDtimestep,
                                                     float* devCarryover)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= N)
        return;
    //hiprandState localState = state[idx];
    //hiprandState localState;  local state of generator
	
    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_angles = n_angles_list[idx];

    // initialize the force to 0
    Scalar4 force_idx = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
     
    //make counter for number of loops
    int counter = 0;
		
    // loop over all angles
    for (int angle_idx = 0; angle_idx < n_angles; angle_idx++)
	{
	int cur_angle_abc = apos_list[pitch * angle_idx + idx];
			
	if (cur_angle_abc == 1)
	    {
	    counter += 1;
	    //printf("forcew=%f\n",force_idx.w);
	    ////////////////////////////////////////////////////////////Get params
            group_storage<3> cur_angle = alist[pitch * angle_idx + idx];

	    //int cur_angle_x_idx = cur_angle.idx[0];
	    //int cur_angle_y_idx = cur_angle.idx[1];
	    int cur_angle_type = cur_angle.idx[2];
				
	    // get the angle parameters (MEM TRANSFER: 8 bytes)
	    //Scalar2 params = texFetchScalar2(d_params, angle_params_tex, cur_angle_type);
	    Scalar2 params = __ldg(d_params + cur_angle_type);
	    Scalar Xi = params.x; //K
	    Scalar Tau = params.y;//t_0
				
	    ////////////////// get sig params
	    //const Scalar2 cgSR = texFetchScalar2(d_CGCMMsr, angle_CGCMMsr_tex, cur_angle_type);
            const Scalar2 cgSR = __ldg(d_PCNDsr + cur_angle_type);

	    int number = cgSR.x;//sigma//number
	    //Scalar cgrcut = cgSR.y;
				
            /////////////get eps param
            //const Scalar4 cgEPOW = texFetchScalar4(d_CGCMMepow, angle_CGCMMepow_tex, cur_angle_type);
            const uint16_t cgEPOW = __ldg(d_PCNDepow + cur_angle_type);
	    uint16_t seed = cgEPOW;

	    // read in the tag of our particle.
	    unsigned int ptag = d_tag[idx];
	    // get the angle pow/pref parameters (MEM TRANSFER: 12 bytes)
	    //int seed = cgEPOW.x; //get parameter seed (epsilon)
            //Scalar cgpow1 = cgEPOW.y;
            //Scalar cgpow2 = cgEPOW.z;
            //Scalar cgpref = cgEPOW.w;
            //////////////////////////////////////////////////////////////

	    // Initialize the Random Number Generator and generate the 6 random numbers
	    RandomGenerator rng(hoomd::Seed(RNGIdentifier::PCNDAngleForceCompute, timestep, seed),
			       hoomd::Counter(ptag));
	    UniformDistribution<Scalar> uniform(Scalar(0), Scalar(1));

	    Scalar a_x = uniform(rng);
	    Scalar b_x = uniform(rng);
	    Scalar a_y = uniform(rng);
	    Scalar b_y = uniform(rng);
	    Scalar a_z = uniform(rng);
	    Scalar b_z = uniform(rng);
						
	    if (PCNDtimestep == 0)
	        {
		devCarryover[(number) * 6 + counter * 3] = Xi * sqrt(-2 * log(a_x)) * cosf(2 * 3.1415926535897 * b_x);
		devCarryover[(number) * 6 + 1 + counter * 3] = Xi * sqrt(-2 * log(a_y)) * cosf(2 * 3.1415926535897 * b_y);
		devCarryover[(number) * 6 + 2 + counter * 3] = Xi * sqrt(-2 * log(a_z)) * cosf(2 * 3.1415926535897 * b_z);				
		force_idx.x += devCarryover[(number) * 6 + counter * 3];
		force_idx.y += devCarryover[(number) * 6 + 1 + counter * 3];
		force_idx.z += devCarryover[(number) * 6 + 2 + counter * 3];
	        }
	    else if (PCNDtimestep != 0)
	        {
		Scalar magx = devCarryover[(number) * 6 + counter * 3];
		Scalar magy = devCarryover[(number) * 6 + 1 + counter * 3];
		Scalar magz = devCarryover[(number) * 6 + 2 + counter * 3];
	        Scalar E = exp(-1 / Tau);
		Scalar hx = Xi * sqrt(-2 * (1 - E * E) * log(a_x)) * cosf(2 * 3.1415926535897 * b_x);
		Scalar hy = Xi * sqrt(-2 * (1 - E * E) * log(a_y)) * cosf(2 * 3.1415926535897 * b_y);
	        Scalar hz = Xi * sqrt(-2 * (1 - E * E) * log(a_z)) * cosf(2 * 3.1415926535897 * b_z);
		
		if (hx > Xi * sqrt(-2 * log(0.001)))
		    {
		    hx = Xi * sqrt(-2 * log(0.001));
		    }
		else if (hx <- Xi * sqrt(-2 * log(0.001)))
		    {
		    hx = -Xi * sqrt(-2 * log(0.001));
		    }
		if (hy > Xi * sqrt(-2 * log(0.001)))
		    {
		    hy = Xi * sqrt(-2 * log(0.001));
		    }
		else if (hy <- Xi * sqrt(-2 * log(0.001)))
		    {
		    hy = -Xi * sqrt(-2 * log(0.001));
		    }
		if (hz > Xi * sqrt(-2 * log(0.001)))
		    {
		    hz = Xi * sqrt(-2 * log(0.001));
		    }
		else if (hz <- Xi * sqrt(-2 * log(0.001)))
		    {
		    hz= -Xi * sqrt(-2 * log(0.001));
		    }
	        if (idx == 70 && timestep <10)
		    {
		    Scalar carryx = devCarryover[(number) * 6 + 0 + counter * 3];
		    Scalar carryy = devCarryover[(number) * 6 + 1 + counter * 3];
		    Scalar carryz = devCarryover[(number) * 6 + 2 + counter * 3];
		    //printf("timestep = %i magx=%f carryover=%f counter=%i, hx=%f, R1=%f, R2=%f\n",timestep,magx,carryx,counter,hx,R1,R2);
		    //printf("timestep = %i magy=%f carryover=%f counter=%i, hy=%f, R3=%f, R4=%f\n",timestep,magy,carryy,counter,hy,R3,R4);
		    //printf("timestep = %i magz=%f carryover=%f counter=%i, hz=%f, R5=%f, R6=%f\n",timestep,magz,carryz,counter,hz,R5,R6);
		    printf("forcex=%f forcey=%f forcez=%f counter=%i\n",force_idx.x,force_idx.y,force_idx.z,counter);
		    } 
					
		devCarryover[(number) * 6 + counter * 3] = E * magx + hx;
		devCarryover[(number) * 6 + 1 + counter * 3] = E * magy + hy;
		devCarryover[(number) * 6 + 2 + counter * 3] = E * magz + hz;
		force_idx.x += devCarryover[(number) * 6 + counter * 3];
		force_idx.y += devCarryover[(number) * 6 + 1 + counter * 3];
		force_idx.z += devCarryover[(number) * 6 + 2 + counter * 3];
		}
	    }
        }
        force_idx.w += sqrt(force_idx.x * force_idx.x + force_idx.y * force_idx.y + force_idx.z * force_idx.z);
        d_force[idx] = force_idx;
    }
    

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param atable List of angles stored on the GPU
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
    \param d_params K and t_0 params packed as Scalar2 variables
    \param d_PCNDsr sigma, and rcut packed as Scalar2 variables
    \param d_PCNDepow epsilon, pow1, pow2, and prefactor packed as Scalar4 variables
    \param n_angle_types Number of angle types in d_params
    \param block_size Block size to use when performing calculations
    \param compute_capability Compute capability of the device (200, 300, 350, ...)

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one Scalar2 element per angle type. The x component contains K
    the spring constant and the y component contains t_0 the equilibrium angle.
*/
hipError_t gpu_compute_PCND_angle_forces(Scalar4* d_force,
                                         Scalar* d_virial,
                                         const size_t virial_pitch,
					 const unsigned int* d_tag,
                                         const unsigned int N,
                                         const Scalar4* d_pos,
                                         const BoxDim& box,
                                         const group_storage<3>* atable,
                                         const unsigned int* apos_list,
                                         const unsigned int pitch,
                                         const unsigned int* n_angles_list,
                                         Scalar2* d_params,
                                         Scalar2* d_PCNDsr,
                                         uint16_t* d_PCNDepow,
                                         unsigned int n_angle_types,
                                         int block_size,
                                         //const unsigned int compute_capability,
                                         uint64_t timestep,
                                         //float* devData,
                                         uint64_t PCNDtimestep,
                                         float* devCarryover)
    {
    assert(d_params);
    assert(d_PCNDsr);
    assert(d_PCNDepow);
    
    //float *lookupArray;
    //hipMemcpyToSymbol(HIP_SYMBOL(lookupArray),(void*)hostData,100*sizeof(float),0);
    
    static unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_PCND_angle_forces_kernel);
    max_block_size = attr.maxThreadsPerBlock;
    
    unsigned int run_block_size = min(block_size, max_block_size);

    // setup the grid to run the kernel
    dim3 grid(N / run_block_size + 1, 1, 1);
    dim3 threads(run_block_size, 1, 1);

    // bind the textures on pre sm 35 arches
    /* if (compute_capability < 350)
        {
        hipError_t error = hipBindTexture(0, angle_params_tex, d_params, sizeof(Scalar2) * n_angle_types);
        if (error != hipSuccess)
            return error;

        error = hipBindTexture(0, angle_CGCMMsr_tex, d_CGCMMsr, sizeof(Scalar2) * n_angle_types);
        if (error != hipSuccess)
            return error;

        error = hipBindTexture(0, angle_CGCMMepow_tex, d_CGCMMepow, sizeof(Scalar4) * n_angle_types);
        if (error != hipSuccess)
            return error;
        }
    */
    // run the kernel
    hipLaunchKernelGGL((gpu_compute_PCND_angle_forces_kernel),
		        dim3(grid),
			dim3(threads),
			0,
			0,
			d_force,
                        d_virial,
			d_tag,
                        virial_pitch,
                        N,
                        d_pos,
                        box,
                        atable,
                        apos_list,
                        pitch,
                        n_angles_list,
                        d_params,
                        d_PCNDsr,
                        d_PCNDepow,
                        timestep,
                        //devData,
                        PCNDtimestep,
                        devCarryover);

    return hipSuccess;
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
