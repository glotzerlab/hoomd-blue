#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: joaander

#include "TwoStepBDGPU.cuh"
#include "hoomd/HOOMDMath.h"
#include "hoomd/VectorMath.h"

#include "hoomd/RNGIdentifiers.h"
#include "hoomd/RandomNumbers.h"
using namespace hoomd;

#include <assert.h>

/*! \file TwoSteBDGPU.cu
    \brief Defines GPU kernel code for Brownian integration on the GPU. Used by TwoStepBDGPU.
*/

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
//! Takes the second half-step forward in the Langevin integration on a group of particles with
/*! \param d_pos array of particle positions and types
    \param d_vel array of particle positions and masses
    \param d_image array of particle images
    \param box simulation box
    \param d_diameter array of particle diameters
    \param d_tag array of particle tags
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param nwork Number of group members to process on this GPU
    \param d_net_force Net force on each particle
    \param d_gamma_r List of per-type gamma_rs (rotational drag coeff.)
    \param d_orientation Device array of orientation quaternion
    \param d_torque Device array of net torque on each particle
    \param d_inertia Device array of moment of inertial of each particle
    \param d_angmom Device array of transformed angular momentum quaternion of each particle (see
   online documentation) \param d_gamma List of per-type gammas \param n_types Number of particle
   types in the simulation \param use_alpha If true, gamma = alpha * diameter \param alpha Scale
   factor to convert diameter to alpha (when use_alpha is true) \param timestep Current timestep of
   the simulation \param seed User chosen random number seed \param T Temperature set point \param
   aniso If set true, the system would go through rigid body updates for its orientation \param
   deltaT Amount of real time to step forward in one time step \param D Dimensionality of the system
    \param d_noiseless_t If set true, there will be no translational noise (random force)
    \param d_noiseless_r If set true, there will be no rotational noise (random torque)
    \param offset Offset of this GPU into group indices

    This kernel is implemented in a very similar manner to gpu_nve_step_one_kernel(), see it for
   design details.

    This kernel must be launched with enough dynamic shared memory per block to read in d_gamma
*/
__global__ void gpu_brownian_step_one_kernel(Scalar4* d_pos,
                                             Scalar4* d_vel,
                                             int3* d_image,
                                             const BoxDim box,
                                             const Scalar* d_diameter,
                                             const unsigned int* d_tag,
                                             const unsigned int* d_group_members,
                                             const unsigned int nwork,
                                             const Scalar4* d_net_force,
                                             const Scalar3* d_gamma_r,
                                             Scalar4* d_orientation,
                                             Scalar4* d_torque,
                                             const Scalar3* d_inertia,
                                             Scalar4* d_angmom,
                                             const Scalar* d_gamma,
                                             const unsigned int n_types,
                                             const bool use_alpha,
                                             const Scalar alpha,
                                             const uint64_t timestep,
                                             const uint16_t seed,
                                             const Scalar T,
                                             const bool aniso,
                                             const Scalar deltaT,
                                             unsigned int D,
                                             const bool d_noiseless_t,
                                             const bool d_noiseless_r,
                                             const unsigned int offset)
    {
    HIP_DYNAMIC_SHARED(char, s_data)

    Scalar3* s_gammas_r = (Scalar3*)s_data;
    Scalar* s_gammas = (Scalar*)(s_gammas_r + n_types);

    if (!use_alpha)
        {
        // read in the gamma (1 dimensional array), stored in s_gammas[0: n_type] (Pythonic
        // convention)
        for (int cur_offset = 0; cur_offset < n_types; cur_offset += blockDim.x)
            {
            if (cur_offset + threadIdx.x < n_types)
                s_gammas[cur_offset + threadIdx.x] = d_gamma[cur_offset + threadIdx.x];
            }
        __syncthreads();
        }

    // read in the gamma_r, stored in s_gammas_r[0: n_type], which is s_gamma_r[0:n_type]

    for (int cur_offset = 0; cur_offset < n_types; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < n_types)
            s_gammas_r[cur_offset + threadIdx.x] = d_gamma_r[cur_offset + threadIdx.x];
        }
    __syncthreads();

    // determine which particle this thread works on (MEM TRANSFER: 4 bytes)
    int local_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (local_idx < nwork)
        {
        const unsigned int group_idx = local_idx + offset;

        // determine the particle to work on
        unsigned int idx = d_group_members[group_idx];
        Scalar4 postype = d_pos[idx];
        Scalar4 vel = d_vel[idx];
        Scalar4 net_force = d_net_force[idx];
        int3 image = d_image[idx];

        // read in the tag of our particle.
        unsigned int ptag = d_tag[idx];

        // compute the random force
        RandomGenerator rng(hoomd::Seed(RNGIdentifier::TwoStepBD, timestep, seed),
                            hoomd::Counter(ptag));
        UniformDistribution<Scalar> uniform(Scalar(-1), Scalar(1));
        Scalar rx = uniform(rng);
        Scalar ry = uniform(rng);
        Scalar rz = uniform(rng);

        // calculate the magnitude of the random force
        Scalar gamma;
        if (use_alpha)
            {
            // determine gamma from diameter
            gamma = alpha * d_diameter[idx];
            }
        else
            {
            // determine gamma from type
            unsigned int typ = __scalar_as_int(postype.w);
            gamma = s_gammas[typ];
            }

        // compute the bd force (the extra factor of 3 is because <rx^2> is 1/3 in the uniform -1,1
        // distribution it is not the dimensionality of the system
        Scalar coeff = fast::sqrt(Scalar(3.0) * Scalar(2.0) * gamma * T / deltaT);
        if (d_noiseless_t)
            coeff = Scalar(0.0);
        Scalar Fr_x = rx * coeff;
        Scalar Fr_y = ry * coeff;
        Scalar Fr_z = rz * coeff;

        if (D < 3)
            Fr_z = Scalar(0.0);

        // update position
        postype.x += (net_force.x + Fr_x) * deltaT / gamma;
        postype.y += (net_force.y + Fr_y) * deltaT / gamma;
        postype.z += (net_force.z + Fr_z) * deltaT / gamma;

        // particles may have been moved slightly outside the box by the above steps, wrap them back
        // into place
        box.wrap(postype, image);

        if (d_noiseless_t)
            {
            vel.x = net_force.x / gamma;
            vel.y = net_force.y / gamma;
            if (D > 2)
                vel.z = net_force.z / gamma;
            else
                vel.z = 0;
            }
        else
            {
            // draw a new random velocity for particle j
            Scalar mass = vel.w;
            Scalar sigma = fast::sqrt(T / mass);
            NormalDistribution<Scalar> normal(sigma);
            vel.x = normal(rng);
            vel.y = normal(rng);
            if (D > 2)
                vel.z = normal(rng);
            else
                vel.z = 0;
            }

        // write out data
        d_pos[idx] = postype;
        d_vel[idx] = vel;
        d_image[idx] = image;

        // rotational random force and orientation quaternion updates
        if (aniso)
            {
            unsigned int type_r = __scalar_as_int(d_pos[idx].w);

            // gamma_r is stored in the second half of s_gammas a.k.a s_gammas_r
            Scalar3 gamma_r = s_gammas_r[type_r];
            if (gamma_r.x > 0 || gamma_r.y > 0 || gamma_r.z > 0)
                {
                vec3<Scalar> p_vec;
                quat<Scalar> q(d_orientation[idx]);
                vec3<Scalar> t(d_torque[idx]);
                vec3<Scalar> I(d_inertia[idx]);

                // check if the shape is degenerate
                bool x_zero, y_zero, z_zero;
                x_zero = (I.x < EPSILON);
                y_zero = (I.y < EPSILON);
                z_zero = (I.z < EPSILON);

                Scalar3 sigma_r = make_scalar3(fast::sqrt(Scalar(2.0) * gamma_r.x * T / deltaT),
                                               fast::sqrt(Scalar(2.0) * gamma_r.y * T / deltaT),
                                               fast::sqrt(Scalar(2.0) * gamma_r.z * T / deltaT));
                if (d_noiseless_r)
                    sigma_r = make_scalar3(0, 0, 0);

                // original Gaussian random torque
                // Gaussian random distribution is preferred in terms of preserving the exact math
                vec3<Scalar> bf_torque;
                bf_torque.x = NormalDistribution<Scalar>(sigma_r.x)(rng);
                bf_torque.y = NormalDistribution<Scalar>(sigma_r.y)(rng);
                bf_torque.z = NormalDistribution<Scalar>(sigma_r.z)(rng);

                if (x_zero)
                    bf_torque.x = 0;
                if (y_zero)
                    bf_torque.y = 0;
                if (z_zero)
                    bf_torque.z = 0;

                // use the damping by gamma_r and rotate back to lab frame
                // For Future Updates: take special care when have anisotropic gamma_r
                bf_torque = rotate(q, bf_torque);
                if (D < 3)
                    {
                    bf_torque.x = 0;
                    bf_torque.y = 0;
                    t.x = 0;
                    t.y = 0;
                    }

                // do the integration for quaternion
                q += Scalar(0.5) * deltaT * ((t + bf_torque) / vec3<Scalar>(gamma_r)) * q;
                q = q * (Scalar(1.0) / slow::sqrt(norm2(q)));
                d_orientation[idx] = quat_to_scalar4(q);

                if (d_noiseless_r)
                    {
                    p_vec.x = t.x / gamma_r.x;
                    p_vec.y = t.y / gamma_r.y;
                    p_vec.z = t.z / gamma_r.z;
                    }
                else
                    {
                    // draw a new random ang_mom for particle j in body frame
                    p_vec.x = NormalDistribution<Scalar>(fast::sqrt(T * I.x))(rng);
                    p_vec.y = NormalDistribution<Scalar>(fast::sqrt(T * I.y))(rng);
                    p_vec.z = NormalDistribution<Scalar>(fast::sqrt(T * I.z))(rng);
                    }

                if (x_zero)
                    p_vec.x = 0;
                if (y_zero)
                    p_vec.y = 0;
                if (z_zero)
                    p_vec.z = 0;

                // !! Note this ang_mom isn't well-behaving in 2D,
                // !! because may have effective non-zero ang_mom in x,y

                // store ang_mom quaternion
                quat<Scalar> p = Scalar(2.0) * q * p_vec;
                d_angmom[idx] = quat_to_scalar4(p);
                }
            }
        }
    }

/*! \param d_pos array of particle positions and types
    \param d_vel array of particle positions and masses
    \param d_image array of particle images
    \param box simulation box
    \param d_diameter array of particle diameters
    \param d_tag array of particle tags
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Net force on each particle
    \param d_gamma_r List of per-type gamma_rs (rotational drag coeff.)
    \param d_orientation Device array of orientation quaternion
    \param d_torque Device array of net torque on each particle
    \param d_inertia Device array of moment of inertial of each particle
    \param d_angmom Device array of transformed angular momentum quaternion of each particle (see
   online documentation) \param langevin_args Collected arguments for gpu_brownian_step_one_kernel()
    \param aniso If set true, the system would go through rigid body updates for its orientation
    \param deltaT Amount of real time to step forward in one time step
    \param D Dimensionality of the system
    \param d_noiseless_t If set true, there will be no translational noise (random force)
    \param d_noiseless_r If set true, there will be no rotational noise (random torque)

    This is just a driver for gpu_brownian_step_one_kernel(), see it for details.
*/
hipError_t gpu_brownian_step_one(Scalar4* d_pos,
                                 Scalar4* d_vel,
                                 int3* d_image,
                                 const BoxDim& box,
                                 const Scalar* d_diameter,
                                 const unsigned int* d_tag,
                                 const unsigned int* d_group_members,
                                 const unsigned int group_size,
                                 const Scalar4* d_net_force,
                                 const Scalar3* d_gamma_r,
                                 Scalar4* d_orientation,
                                 Scalar4* d_torque,
                                 const Scalar3* d_inertia,
                                 Scalar4* d_angmom,
                                 const langevin_step_two_args& langevin_args,
                                 const bool aniso,
                                 const Scalar deltaT,
                                 const unsigned int D,
                                 const bool d_noiseless_t,
                                 const bool d_noiseless_r,
                                 const GPUPartition& gpu_partition)
    {
    unsigned int run_block_size = 256;

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid((nwork / run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        hipLaunchKernelGGL((gpu_brownian_step_one_kernel),
                           dim3(grid),
                           dim3(threads),
                           (unsigned int)(sizeof(Scalar) * langevin_args.n_types
                                          + sizeof(Scalar3) * langevin_args.n_types),
                           0,
                           d_pos,
                           d_vel,
                           d_image,
                           box,
                           d_diameter,
                           d_tag,
                           d_group_members,
                           nwork,
                           d_net_force,
                           d_gamma_r,
                           d_orientation,
                           d_torque,
                           d_inertia,
                           d_angmom,
                           langevin_args.d_gamma,
                           langevin_args.n_types,
                           langevin_args.use_alpha,
                           langevin_args.alpha,
                           langevin_args.timestep,
                           langevin_args.seed,
                           langevin_args.T,
                           aniso,
                           deltaT,
                           D,
                           d_noiseless_t,
                           d_noiseless_r,
                           range.first);
        }

    return hipSuccess;
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
