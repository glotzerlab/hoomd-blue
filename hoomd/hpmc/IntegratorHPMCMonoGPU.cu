#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "IntegratorHPMCMonoGPUTypes.cuh"
#include "hoomd/GPUPartition.cuh"

namespace hoomd
    {
namespace hpmc
    {
namespace gpu
    {
namespace kernel
    {
//! Kernel to generate expanded cells
/*! \param d_excell_idx Output array to list the particle indices in the expanded cells
    \param d_excell_size Output array to list the number of particles in each expanded cell
    \param excli Indexer for the expanded cells
    \param d_cell_idx Particle indices in the normal cells
    \param d_cell_size Number of particles in each cell
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer
    \param cli Cell list indexer
    \param cadji Cell adjacency indexer
    \param ngpu Number of active devices

    gpu_hpmc_excell_kernel executes one thread per cell. It gathers the particle indices from all
   neighboring cells into the output expanded cell.
*/
__global__ void hpmc_excell(unsigned int* d_excell_idx,
                            unsigned int* d_excell_size,
                            const Index2D excli,
                            const unsigned int* d_cell_idx,
                            const unsigned int* d_cell_size,
                            const unsigned int* d_cell_adj,
                            const Index3D ci,
                            const Index2D cli,
                            const Index2D cadji,
                            const unsigned int ngpu)
    {
    // compute the output cell
    unsigned int my_cell = 0;
    my_cell = blockDim.x * blockIdx.x + threadIdx.x;

    if (my_cell >= ci.getNumElements())
        return;

    unsigned int my_cell_size = 0;

    // loop over neighboring cells and build up the expanded cell list
    for (unsigned int offset = 0; offset < cadji.getW(); offset++)
        {
        unsigned int neigh_cell = d_cell_adj[cadji(offset, my_cell)];

        // iterate over per-device cell lists
        for (unsigned int igpu = 0; igpu < ngpu; ++igpu)
            {
            unsigned int neigh_cell_size = d_cell_size[neigh_cell + igpu * ci.getNumElements()];

            for (unsigned int k = 0; k < neigh_cell_size; k++)
                {
                // read in the index of the new particle to add to our cell
                unsigned int new_idx = d_cell_idx[cli(k, neigh_cell) + igpu * cli.getNumElements()];
                d_excell_idx[excli(my_cell_size, my_cell)] = new_idx;
                my_cell_size++;
                }
            }
        }

    // write out the final size
    d_excell_size[my_cell] = my_cell_size;
    }

//! Kernel for grid shift
/*! \param d_postype postype of each particle
    \param d_image Image flags for each particle
    \param N number of particles
    \param box Simulation box
    \param shift Vector by which to translate the particles

    Shift all the particles by a given vector.

    \ingroup hpmc_kernels
*/
__global__ void hpmc_shift(Scalar4* d_postype,
                           int3* d_image,
                           const unsigned int N,
                           const BoxDim box,
                           const Scalar3 shift)
    {
    // identify the active cell that this thread handles
    unsigned int my_pidx = blockIdx.x * blockDim.x + threadIdx.x;

    // this thread is inactive if it indexes past the end of the particle list
    if (my_pidx >= N)
        return;

    // pull in the current position
    Scalar4 postype = d_postype[my_pidx];

    // shift the position
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    pos += shift;

    // wrap the particle back into the box
    int3 image = d_image[my_pidx];
    box.wrap(pos, image);

    // write out the new position and orientation
    d_postype[my_pidx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
    d_image[my_pidx] = image;
    }

//!< Kernel to evaluate convergence
__global__ void hpmc_check_convergence(const unsigned int* d_trial_move_type,
                                       const unsigned int* d_reject_out_of_cell,
                                       unsigned int* d_reject_in,
                                       unsigned int* d_reject_out,
                                       unsigned int* d_condition,
                                       const unsigned int nwork,
                                       const unsigned work_offset)
    {
    // the particle we are handling
    unsigned int work_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (work_idx >= nwork)
        return;
    unsigned int i = work_idx + work_offset;

    // is this particle considered?
    bool move_active = d_trial_move_type[i] > 0;

    // combine with reject flag from gen_moves for particles which are always rejected
    bool reject_out_of_cell = d_reject_out_of_cell[i];
    bool reject = d_reject_out[i];

    // did the answer change since the last iteration?
    if (move_active && !reject_out_of_cell && reject != d_reject_in[i])
        {
        // flag that we're not done yet (a trivial race condition upon write)
        *d_condition = 1;
        }

    // update the reject flags
    d_reject_out[i] = reject || reject_out_of_cell;

    // clear input
    d_reject_in[i] = reject_out_of_cell;
    }
    } // end namespace kernel

//! Driver for kernel::hpmc_excell()
void hpmc_excell(unsigned int* d_excell_idx,
                 unsigned int* d_excell_size,
                 const Index2D& excli,
                 const unsigned int* d_cell_idx,
                 const unsigned int* d_cell_size,
                 const unsigned int* d_cell_adj,
                 const Index3D& ci,
                 const Index2D& cli,
                 const Index2D& cadji,
                 const unsigned int ngpu,
                 const unsigned int block_size)
    {
    assert(d_excell_idx);
    assert(d_excell_size);
    assert(d_cell_idx);
    assert(d_cell_size);
    assert(d_cell_adj);

    // determine the maximum block size and clamp the input block size down
    int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_excell)));
    max_block_size = attr.maxThreadsPerBlock;

    // setup the grid to run the kernel
    unsigned int run_block_size = min(block_size, (unsigned int)max_block_size);
    dim3 threads(run_block_size, 1, 1);
    dim3 grid(ci.getNumElements() / run_block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_excell,
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       d_excell_idx,
                       d_excell_size,
                       excli,
                       d_cell_idx,
                       d_cell_size,
                       d_cell_adj,
                       ci,
                       cli,
                       cadji,
                       ngpu);
    }

//! Kernel driver for kernel::hpmc_shift()
void hpmc_shift(Scalar4* d_postype,
                int3* d_image,
                const unsigned int N,
                const BoxDim& box,
                const Scalar3 shift,
                const unsigned int block_size)
    {
    assert(d_postype);
    assert(d_image);

    // setup the grid to run the kernel
    dim3 threads_shift(block_size, 1, 1);
    dim3 grid_shift(N / block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_shift,
                       dim3(grid_shift),
                       dim3(threads_shift),
                       0,
                       0,
                       d_postype,
                       d_image,
                       N,
                       box,
                       shift);

    // after this kernel we return control of cuda managed memory to the host
    hipDeviceSynchronize();
    }

void hpmc_check_convergence(const unsigned int* d_trial_move_type,
                            const unsigned int* d_reject_out_of_cell,
                            unsigned int* d_reject_in,
                            unsigned int* d_reject_out,
                            unsigned int* d_condition,
                            const GPUPartition& gpu_partition,
                            const unsigned int block_size)
    {
    // determine the maximum block size and clamp the input block size down
    int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_check_convergence)));
    max_block_size = attr.maxThreadsPerBlock;

    // setup the grid to run the kernel
    unsigned int run_block_size = min(block_size, (unsigned int)max_block_size);

    dim3 threads(run_block_size, 1, 1);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;
        const unsigned int num_blocks = nwork / run_block_size + 1;
        dim3 grid(num_blocks, 1, 1);

        hipLaunchKernelGGL(kernel::hpmc_check_convergence,
                           grid,
                           threads,
                           0,
                           0,
                           d_trial_move_type,
                           d_reject_out_of_cell,
                           d_reject_in,
                           d_reject_out,
                           d_condition,
                           nwork,
                           range.first);
        }
    }

    } // end namespace gpu
    } // end namespace hpmc
    } // end namespace hoomd
