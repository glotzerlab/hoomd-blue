// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "EvaluatorExternalElectricField.h"
#include "EvaluatorExternalPeriodic.h"
#include "EvaluatorPairForceShiftedLJ.h"
#include "EvaluatorPairGauss.h"
#include "EvaluatorPairLJ.h"
#include "EvaluatorPairMie.h"
#include "EvaluatorPairMorse.h"
#include "EvaluatorPairSLJ.h"
#include "EvaluatorPairYukawa.h"
#include "EvaluatorWalls.h"
#include "PotentialExternalGPU.cuh"
#include "WallData.h"

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
// Instantiate external evaluator templates
//! Evaluator for External Periodic potentials.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorExternalPeriodic>(const external_potential_args_t& external_potential_args,
                                    const typename EvaluatorExternalPeriodic::param_type* d_params,
                                    const typename EvaluatorExternalPeriodic::field_type* d_field);
//! Evaluator for electric fields
template hipError_t __attribute__((visibility("default"))) gpu_cpef<EvaluatorExternalElectricField>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorExternalElectricField::param_type* d_params,
    const typename EvaluatorExternalElectricField::field_type* d_field);
//! Evaluator for Lennard-Jones pair potential.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorWalls<EvaluatorPairLJ>>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorWalls<EvaluatorPairLJ>::param_type* d_params,
    const typename EvaluatorWalls<EvaluatorPairLJ>::field_type* d_field);
//! Evaluator for Gaussian pair potential.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorWalls<EvaluatorPairGauss>>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorWalls<EvaluatorPairGauss>::param_type* d_params,
    const typename EvaluatorWalls<EvaluatorPairGauss>::field_type* d_field);
//! Evaluator for Yukawa pair potential.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorWalls<EvaluatorPairYukawa>>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorWalls<EvaluatorPairYukawa>::param_type* d_params,
    const typename EvaluatorWalls<EvaluatorPairYukawa>::field_type* d_field);
//! Evaluator for Shifted Lennard-Jones pair potential.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorWalls<EvaluatorPairSLJ>>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorWalls<EvaluatorPairSLJ>::param_type* d_params,
    const typename EvaluatorWalls<EvaluatorPairSLJ>::field_type* d_field);
//! Evaluator for Morse pair potential.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorWalls<EvaluatorPairMorse>>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorWalls<EvaluatorPairMorse>::param_type* d_params,
    const typename EvaluatorWalls<EvaluatorPairMorse>::field_type* d_field);
//! Evaluator for Force Shifted Lennard-Jones pair potential.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorWalls<EvaluatorPairForceShiftedLJ>>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorWalls<EvaluatorPairForceShiftedLJ>::param_type* d_params,
    const typename EvaluatorWalls<EvaluatorPairForceShiftedLJ>::field_type* d_field);
//! Evaluator for Mie pair potential.
template hipError_t __attribute__((visibility("default")))
gpu_cpef<EvaluatorWalls<EvaluatorPairMie>>(
    const external_potential_args_t& external_potential_args,
    const typename EvaluatorWalls<EvaluatorPairMie>::param_type* d_params,
    const typename EvaluatorWalls<EvaluatorPairMie>::field_type* d_field);

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
