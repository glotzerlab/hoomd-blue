// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file ReactionFieldDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "AllDriverPotentialPairGPU.cuh"
#include "EvaluatorPairReactionField.h"

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
hipError_t gpu_compute_reaction_field_forces(const pair_args_t& args,
                                             const EvaluatorPairReactionField::param_type* d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairReactionField>(args, d_params);
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
