#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "IntegratorHPMCMonoGPUDepletants.cuh"
#include "hoomd/CachedAllocator.h"
#include "hoomd/GPUPartition.cuh"
#include "hoomd/RNGIdentifiers.h"
#include "hoomd/RandomNumbers.h"

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

namespace hoomd
    {
namespace hpmc
    {
namespace gpu
    {
namespace kernel
    {
//! Generate number of depletants per particle
__global__ void generate_num_depletants(const uint16_t seed,
                                        const uint64_t timestep,
                                        const unsigned int select,
                                        const unsigned int rank,
                                        const unsigned int depletant_type_a,
                                        const unsigned int depletant_type_b,
                                        const Index2D depletant_idx,
                                        const unsigned int work_offset,
                                        const unsigned int nwork,
                                        const Scalar* d_lambda,
                                        const Scalar4* d_postype,
                                        unsigned int* d_n_depletants)
    {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= nwork)
        return;

    idx += work_offset;

    hoomd::RandomGenerator rng_poisson(
        hoomd::Seed(hoomd::RNGIdentifier::HPMCDepletantNum, timestep, seed),
        hoomd::Counter(idx,
                       rank,
                       depletant_idx(depletant_type_a, depletant_type_b),
                       static_cast<uint16_t>(select)));
    unsigned int type_i = __scalar_as_int(d_postype[idx].w);
    d_n_depletants[idx] = hoomd::PoissonDistribution<Scalar>(
        d_lambda[type_i * depletant_idx.getNumElements()
                 + depletant_idx(depletant_type_a, depletant_type_b)])(rng_poisson);
    }

//! Generate number of depletants per particle (ntrial version)
__global__ void generate_num_depletants_ntrial(const Scalar4* d_vel,
                                               const Scalar4* d_trial_vel,
                                               const unsigned int ntrial,
                                               const unsigned int depletant_type_a,
                                               const unsigned int depletant_type_b,
                                               const Index2D depletant_idx,
                                               const Scalar* d_lambda,
                                               const Scalar4* d_postype,
                                               unsigned int* d_n_depletants,
                                               const unsigned int N_local,
                                               const unsigned int work_offset,
                                               const unsigned int nwork)
    {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= nwork)
        return;

    unsigned int i = idx + work_offset;

    unsigned int i_trial_config = blockIdx.y;
    unsigned int i_trial = (i_trial_config >> 1) % ntrial;
    unsigned int new_config = i_trial_config & 1;

    if (i >= N_local && new_config)
        return; // ghosts only exist in the old config

    // draw a Poisson variate according to the seed stored in the auxillary variable (vel.x)
    unsigned int seed_i
        = new_config ? __scalar_as_int(d_trial_vel[i].x) : __scalar_as_int(d_vel[i].x);
    hoomd::RandomGenerator rng_num(
        hoomd::Seed(hoomd::RNGIdentifier::HPMCDepletantNum, 0, 0),
        hoomd::Counter(depletant_idx(depletant_type_a, depletant_type_b), seed_i, i_trial));

    unsigned int type_i = __scalar_as_int(d_postype[i].w);
    Scalar lambda = d_lambda[type_i * depletant_idx.getNumElements()
                             + depletant_idx(depletant_type_a, depletant_type_b)];
    unsigned int n = hoomd::PoissonDistribution<Scalar>(lambda)(rng_num);

    // store result
    d_n_depletants[i * 2 * ntrial + new_config * ntrial + i_trial] = n;
    }

__global__ void hpmc_reduce_counters(const unsigned int ngpu,
                                     const unsigned int pitch,
                                     const hpmc_counters_t* d_per_device_counters,
                                     hpmc_counters_t* d_counters,
                                     const unsigned int implicit_pitch,
                                     const Index2D depletant_idx,
                                     const hpmc_implicit_counters_t* d_per_device_implicit_counters,
                                     hpmc_implicit_counters_t* d_implicit_counters)
    {
    for (unsigned int igpu = 0; igpu < ngpu; ++igpu)
        {
        *d_counters = *d_counters + d_per_device_counters[igpu * pitch];

        for (unsigned int itype = 0; itype < depletant_idx.getNumElements(); ++itype)
            d_implicit_counters[itype]
                = d_implicit_counters[itype]
                  + d_per_device_implicit_counters[itype + igpu * implicit_pitch];
        }
    }

//! Kernel to perform the Metroplis-Hastings step for depletants
__global__ void hpmc_depletants_accept(const uint16_t seed,
                                       const uint64_t timestep,
                                       const unsigned int select,
                                       const unsigned int rank,
                                       const int* d_deltaF_int,
                                       const Index2D depletant_idx,
                                       const unsigned int deltaF_pitch,
                                       const Scalar* d_fugacity,
                                       const unsigned int* d_ntrial,
                                       unsigned int* d_reject_out,
                                       const unsigned int nwork,
                                       const unsigned work_offset)
    {
    // the particle we are handling
    unsigned int work_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (work_idx >= nwork)
        return;
    unsigned int i = work_idx + work_offset;

    // reduce free energy over depletant type pairs
    Scalar deltaF_i(0.0);
    for (unsigned int itype = 0; itype < depletant_idx.getW(); ++itype)
        for (unsigned int jtype = itype; jtype < depletant_idx.getH(); ++jtype)
            {
            // it is important that this loop is serial, to eliminate non-determism
            // in the acceptance loop (too much noise can make convergence difficult)
            unsigned int ntrial = d_ntrial[depletant_idx(itype, jtype)];
            Scalar fugacity = d_fugacity[depletant_idx(itype, jtype)];

            if (fugacity == 0.0 || ntrial == 0)
                continue;

            // rescale deltaF to units of kBT
            int dF_int_i = d_deltaF_int[deltaF_pitch * depletant_idx(itype, jtype) + i];
            deltaF_i += log(1 + 1 / (Scalar)ntrial) * dF_int_i;
            }

    hoomd::RandomGenerator rng_accept(
        hoomd::Seed(hoomd::RNGIdentifier::HPMCDepletantsAccept, timestep, seed),
        hoomd::Counter(i, rank, select));

    Scalar u = hoomd::UniformDistribution<Scalar>()(rng_accept);
    bool accept = u <= exp(deltaF_i);

    // update the reject flags
    if (!accept)
        atomicAdd(&d_reject_out[i], 1);
    }

    } // end namespace kernel

void generate_num_depletants(const uint16_t seed,
                             const uint64_t timestep,
                             const unsigned int select,
                             const unsigned int rank,
                             const unsigned int depletant_type_a,
                             const unsigned int depletant_type_b,
                             const Index2D depletant_idx,
                             const Scalar* d_lambda,
                             const Scalar4* d_postype,
                             unsigned int* d_n_depletants,
                             const unsigned int block_size,
                             const hipStream_t* streams,
                             const GPUPartition& gpu_partition)
    {
    // determine the maximum block size and clamp the input block size down
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::generate_num_depletants)));
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);
        unsigned int nwork = range.second - range.first;

        hipLaunchKernelGGL(kernel::generate_num_depletants,
                           nwork / run_block_size + 1,
                           run_block_size,
                           0,
                           streams[idev],
                           seed,
                           timestep,
                           select,
                           rank,
                           depletant_type_a,
                           depletant_type_b,
                           depletant_idx,
                           range.first,
                           nwork,
                           d_lambda,
                           d_postype,
                           d_n_depletants);
        }
    }

void generate_num_depletants_ntrial(const Scalar4* d_vel,
                                    const Scalar4* d_trial_vel,
                                    const unsigned int ntrial,
                                    const unsigned int depletant_type_a,
                                    const unsigned int depletant_type_b,
                                    const Index2D depletant_idx,
                                    const Scalar* d_lambda,
                                    const Scalar4* d_postype,
                                    unsigned int* d_n_depletants,
                                    const unsigned int N_local,
                                    const bool add_ghosts,
                                    const unsigned int n_ghosts,
                                    const GPUPartition& gpu_partition,
                                    const unsigned int block_size,
                                    const hipStream_t* streams)
    {
    // determine the maximum block size and clamp the input block size down
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,
                         reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::generate_num_depletants_ntrial)));
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // add ghosts to final range
        if (idev == (int)gpu_partition.getNumActiveGPUs() - 1 && add_ghosts)
            nwork += n_ghosts;

        if (!nwork)
            continue;

        dim3 grid(nwork / run_block_size + 1, 2 * ntrial, 1);
        dim3 threads(run_block_size, 1, 1);

        hipLaunchKernelGGL((kernel::generate_num_depletants_ntrial),
                           grid,
                           threads,
                           0,
                           streams[idev],
                           d_vel,
                           d_trial_vel,
                           ntrial,
                           depletant_type_a,
                           depletant_type_b,
                           depletant_idx,
                           d_lambda,
                           d_postype,
                           d_n_depletants,
                           N_local,
                           range.first,
                           nwork);
        }
    }

void get_max_num_depletants(unsigned int* d_n_depletants,
                            unsigned int* max_n_depletants,
                            const hipStream_t* streams,
                            const GPUPartition& gpu_partition,
                            CachedAllocator& alloc)
    {
    assert(d_n_depletants);
    thrust::device_ptr<unsigned int> n_depletants(d_n_depletants);
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

#ifdef __HIP_PLATFORM_HCC__
        max_n_depletants[idev] = thrust::reduce(thrust::hip::par(alloc).on(streams[idev]),
#else
        max_n_depletants[idev] = thrust::reduce(thrust::cuda::par(alloc).on(streams[idev]),
#endif
                                                n_depletants + range.first,
                                                n_depletants + range.second,
                                                0,
                                                thrust::maximum<unsigned int>());
        }
    }

//! Compute the max # of depletants per particle, trial insertion, and configuration
void get_max_num_depletants_ntrial(const unsigned int ntrial,
                                   unsigned int* d_n_depletants,
                                   unsigned int* max_n_depletants,
                                   const bool add_ghosts,
                                   const unsigned int n_ghosts,
                                   const hipStream_t* streams,
                                   const GPUPartition& gpu_partition,
                                   CachedAllocator& alloc)
    {
    assert(d_n_depletants);
    thrust::device_ptr<unsigned int> n_depletants(d_n_depletants);
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // add ghosts to final range
        if (idev == (int)gpu_partition.getNumActiveGPUs() - 1 && add_ghosts)
            nwork += n_ghosts;

#ifdef __HIP_PLATFORM_HCC__
        max_n_depletants[idev] = thrust::reduce(thrust::hip::par(alloc).on(streams[idev]),
#else
        max_n_depletants[idev] = thrust::reduce(thrust::cuda::par(alloc).on(streams[idev]),
#endif
                                                n_depletants + range.first * 2 * ntrial,
                                                n_depletants + (range.first + nwork) * 2 * ntrial,
                                                0,
                                                thrust::maximum<unsigned int>());
        }
    }

void reduce_counters(const unsigned int ngpu,
                     const unsigned int pitch,
                     const hpmc_counters_t* d_per_device_counters,
                     hpmc_counters_t* d_counters,
                     const unsigned int implicit_pitch,
                     const Index2D depletant_idx,
                     const hpmc_implicit_counters_t* d_per_device_implicit_counters,
                     hpmc_implicit_counters_t* d_implicit_counters)
    {
    hipLaunchKernelGGL(kernel::hpmc_reduce_counters,
                       1,
                       1,
                       0,
                       0,
                       ngpu,
                       pitch,
                       d_per_device_counters,
                       d_counters,
                       implicit_pitch,
                       depletant_idx,
                       d_per_device_implicit_counters,
                       d_implicit_counters);
    }

void hpmc_depletants_accept(const uint16_t seed,
                            const uint64_t timestep,
                            const unsigned int select,
                            const unsigned int rank,
                            const int* d_deltaF_int,
                            const Index2D depletant_idx,
                            const unsigned int deltaF_pitch,
                            const Scalar* d_fugacity,
                            const unsigned int* d_ntrial,
                            unsigned int* d_reject_out,
                            const GPUPartition& gpu_partition,
                            const unsigned int block_size)
    {
    // determine the maximum block size and clamp the input block size down
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_depletants_accept)));
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    assert(d_deltaF_int);
    assert(d_fugacity);
    assert(d_ntrial);
    assert(d_reject_out);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);
        unsigned int nwork = range.second - range.first;

        hipLaunchKernelGGL(kernel::hpmc_depletants_accept,
                           nwork / run_block_size + 1,
                           run_block_size,
                           0,
                           0,
                           seed,
                           timestep,
                           select,
                           rank,
                           d_deltaF_int,
                           depletant_idx,
                           deltaF_pitch,
                           d_fugacity,
                           d_ntrial,
                           d_reject_out,
                           nwork,
                           range.first);
        }
    }
    } // end namespace gpu
    } // end namespace hpmc
    } // end namespace hoomd
