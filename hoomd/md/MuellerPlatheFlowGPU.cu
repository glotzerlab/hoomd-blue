#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "MuellerPlatheFlowGPU.cuh"
#include "MuellerPlatheFlowGPU.h"
#include "hoomd/HOOMDMath.h"
#include <assert.h>

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wconversion"
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#pragma GCC diagnostic pop

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
struct vel_search_un_opt : public thrust::unary_function<const unsigned int, Scalar3>
    {
    vel_search_un_opt(const Scalar4* const d_vel,
                      const unsigned int* const d_tag,
                      flow_enum::Direction flow_direction)
        : m_vel(d_vel), m_tag(d_tag), m_flow_direction(flow_direction)
        {
        }
    const Scalar4* const m_vel;
    const unsigned int* const m_tag;
    const flow_enum::Direction m_flow_direction;
    __host__ __device__ Scalar3 operator()(const unsigned int idx) const
        {
        const unsigned int tag = m_tag[idx];
        Scalar vel;
        switch (m_flow_direction)
            {
        case flow_enum::X:
            vel = m_vel[idx].x;
            break;
        case flow_enum::Y:
            vel = m_vel[idx].y;
            break;
        case flow_enum::Z:
            vel = m_vel[idx].z;
            break;
            }
        const Scalar mass = m_vel[idx].w;
        vel *= mass;
        Scalar3 result;
        result.x = vel;
        result.y = mass;
        result.z = __int_as_scalar(tag);
        return result;
        }
    };
template<typename CMP>
struct vel_search_binary_opt : public thrust::binary_function<Scalar3, Scalar3, Scalar3>
    {
    vel_search_binary_opt(const unsigned int* const d_rtag,
                          const Scalar4* const d_pos,
                          const BoxDim gl_box,
                          const unsigned int Nslabs,
                          const unsigned int slab_index,
                          const Scalar3 invalid,
                          const flow_enum::Direction slab_direction)
        : m_rtag(d_rtag), m_pos(d_pos), m_gl_box(gl_box), m_Nslabs(Nslabs),
          m_slab_index(slab_index), m_invalid(invalid), m_slab_direction(slab_direction)
        {
        }
    const unsigned int* const m_rtag;
    const Scalar4* const m_pos;
    const BoxDim m_gl_box;
    const unsigned int m_Nslabs;
    const unsigned int m_slab_index;
    const Scalar3 m_invalid;
    const flow_enum::Direction m_slab_direction;

    __host__ __device__ Scalar3 operator()(const Scalar3& a, const Scalar3& b) const
        {
        Scalar3 result = m_invalid;
        // Early exit, if invalid args involved.
        if (a.z == m_invalid.z)
            return b;
        if (b.z == m_invalid.z)
            return a;

        const unsigned int idx_a = m_rtag[__scalar_as_int(a.z)];
        const unsigned int idx_b = m_rtag[__scalar_as_int(b.z)];

        unsigned int index_a, index_b;
        switch (m_slab_direction)
            {
        case flow_enum::X:
            index_a = (m_pos[idx_a].x / m_gl_box.getL().x + .5) * m_Nslabs;
            index_b = (m_pos[idx_b].x / m_gl_box.getL().x + .5) * m_Nslabs;
            break;
        case flow_enum::Y:
            index_a = (m_pos[idx_a].y / m_gl_box.getL().y + .5) * m_Nslabs;
            index_b = (m_pos[idx_b].y / m_gl_box.getL().y + .5) * m_Nslabs;
            break;
        case flow_enum::Z:
            index_a = (m_pos[idx_a].z / m_gl_box.getL().z + .5) * m_Nslabs;
            index_b = (m_pos[idx_b].z / m_gl_box.getL().z + .5) * m_Nslabs;
            break;
            }
        index_a %= m_Nslabs;
        index_b %= m_Nslabs;

        if (index_a == index_b)
            {
            if (index_a == m_slab_index)
                {
                CMP cmp;
                if (cmp(a.x, b.x))
                    result = a;
                else
                    result = b;
                }
            }
        else
            {
            if (index_a == m_slab_index)
                result = a;
            if (index_b == m_slab_index)
                result = b;
            }
        return result;
        }
    };

hipError_t gpu_search_min_max_velocity(const unsigned int group_size,
                                       const Scalar4* const d_vel,
                                       const Scalar4* const d_pos,
                                       const unsigned int* const d_tag,
                                       const unsigned int* const d_rtag,
                                       const unsigned int* const d_group_members,
                                       const BoxDim gl_box,
                                       const unsigned int Nslabs,
                                       const unsigned int max_slab,
                                       const unsigned int min_slab,
                                       Scalar3* const last_max_vel,
                                       Scalar3* const last_min_vel,
                                       const bool has_max_slab,
                                       const bool has_min_slab,
                                       const unsigned int blocksize,
                                       const flow_enum::Direction flow_direction,
                                       const flow_enum::Direction slab_direction)
    {
    thrust::device_ptr<const unsigned int> member_ptr(d_group_members);

    vel_search_un_opt un_opt(d_vel, d_tag, flow_direction);

    if (has_max_slab)
        {
        vel_search_binary_opt<thrust::greater<const Scalar>>
            max_bin_opt(d_rtag, d_pos, gl_box, Nslabs, max_slab, *last_max_vel, slab_direction);
        Scalar3 init = *last_max_vel;
        *last_max_vel = thrust::transform_reduce(member_ptr,
                                                 member_ptr + group_size,
                                                 un_opt,
                                                 init,
                                                 max_bin_opt);
        }

    if (has_min_slab)
        {
        vel_search_binary_opt<thrust::less<const Scalar>>
            min_bin_opt(d_rtag, d_pos, gl_box, Nslabs, min_slab, *last_min_vel, slab_direction);
        Scalar3 init = *last_min_vel;
        *last_min_vel = thrust::transform_reduce(member_ptr,
                                                 member_ptr + group_size,
                                                 un_opt,
                                                 init,
                                                 min_bin_opt);
        }

    return hipPeekAtLastError();
    }

void __global__ gpu_update_min_max_velocity_kernel(const unsigned int* const d_rtag,
                                                   Scalar4* const d_vel,
                                                   const unsigned int Ntotal,
                                                   const Scalar3 last_max_vel,
                                                   const Scalar3 last_min_vel,
                                                   const flow_enum::Direction flow_direction)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 1)
        return;
    const unsigned int min_tag = __scalar_as_int(last_min_vel.z);
    const unsigned int min_idx = d_rtag[min_tag];
    const unsigned int max_tag = __scalar_as_int(last_max_vel.z);
    const unsigned int max_idx = d_rtag[max_tag];
    // Is the particle local on the processor?
    // Swap the particles the new velocities.
    if (min_idx < Ntotal)
        {
        const Scalar new_min_vel = last_max_vel.x / last_min_vel.y;
        switch (flow_direction)
            {
        case flow_enum::X:
            d_vel[min_idx].x = new_min_vel;
            break;
        case flow_enum::Y:
            d_vel[min_idx].y = new_min_vel;
            break;
        case flow_enum::Z:
            d_vel[min_idx].z = new_min_vel;
            break;
            }
        }

    if (max_idx < Ntotal)
        {
        const Scalar new_max_vel = last_min_vel.x / last_max_vel.y;
        switch (flow_direction)
            {
        case flow_enum::X:
            d_vel[max_idx].x = new_max_vel;
            break;
        case flow_enum::Y:
            d_vel[max_idx].y = new_max_vel;
            break;
        case flow_enum::Z:
            d_vel[max_idx].z = new_max_vel;
            break;
            }
        }
    }

hipError_t gpu_update_min_max_velocity(const unsigned int* const d_rtag,
                                       Scalar4* const d_vel,
                                       const unsigned int Ntotal,
                                       const Scalar3 last_max_vel,
                                       const Scalar3 last_min_vel,
                                       const flow_enum::Direction flow_direction)
    {
    dim3 grid(1, 1, 1);
    dim3 threads(1, 1, 1);

    hipLaunchKernelGGL((gpu_update_min_max_velocity_kernel),
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       d_rtag,
                       d_vel,
                       Ntotal,
                       last_max_vel,
                       last_min_vel,
                       flow_direction);

    return hipPeekAtLastError();
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
