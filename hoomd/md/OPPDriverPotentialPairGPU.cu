/*! \file OPPDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing opp pair forces on the GPU
*/

#include "AllDriverPotentialPairGPU.cuh"
#include "EvaluatorPairOPP.h"

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
hipError_t gpu_compute_opp_forces(const pair_args_t& pair_args,
                                  const EvaluatorPairOPP::param_type* d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairOPP>(pair_args, d_params);
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
