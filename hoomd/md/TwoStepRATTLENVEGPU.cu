#include "hip/hip_runtime.h"
// Copyright (c) 2009-2024 The Regents of the University of Michigan.
// Part of HOOMD-blue, released under the BSD 3-Clause License.

#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "TwoStepRATTLENVEGPU.cuh"
#include "hoomd/VectorMath.h"

#include <assert.h>

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
//! Takes the first half-step forward in the velocity-verlet NVE integration on a group of particles
/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param box Box dimensions for periodic boundary condition handling
    \param deltaT timestep
    \param limit If \a limit is true, then the dynamics will be limited so that particles do not
   move a distance further than \a limit_val in one step. \param limit_val Length to limit particle
   distance movement to \param zero_force Set to true to always assign an acceleration of 0 to all
   particles in the group This kernel must be executed with a 1D grid of any block size such that
   the number of threads is greater than or equal to the number of members in the group. The
   kernel's implementation simply reads one particle in each thread and updates that particle.
    <b>Performance notes:</b>
    Particle properties are read via the texture cache to optimize the bandwidth obtained with
   sparse groups. The writes in sparse groups will not be coalesced. However, because ParticleGroup
   sorts the index list the writes will be as contiguous as possible leading to fewer memory
   transactions on compute 1.3 hardware and more cache hits on Fermi.
*/

__global__ void gpu_rattle_nve_step_one_kernel(Scalar4* d_pos,
                                               Scalar4* d_vel,
                                               const Scalar3* d_accel,
                                               int3* d_image,
                                               unsigned int* d_group_members,
                                               const unsigned int nwork,
                                               const unsigned int offset,
                                               BoxDim box,
                                               Scalar deltaT,
                                               bool limit,
                                               Scalar limit_val)
    {
    // determine which particle this thread works on (MEM TRANSFER: 4 bytes)
    int work_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (work_idx < nwork)
        {
        const unsigned int group_idx = work_idx + offset;
        unsigned int idx = d_group_members[group_idx];

        // do velocity verlet update
        // r(t+deltaT) = r(t) + v(t)*deltaT + (1/2)a(t)*deltaT^2
        // v(t+deltaT/2) = v(t) + (1/2)a*deltaT

        // read the particle's position (MEM TRANSFER: 16 bytes)
        Scalar4 postype = d_pos[idx];
        Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);

        // read the particle's velocity and acceleration (MEM TRANSFER: 32 bytes)
        Scalar4 velmass = d_vel[idx];
        Scalar3 vel = make_scalar3(velmass.x, velmass.y, velmass.z);

        Scalar3 accel = d_accel[idx];

        Scalar deltaT_half = Scalar(1.0 / 2.0) * deltaT;

        Scalar3 half_vel;
        half_vel = vel + deltaT_half * accel;

        // update the position (FLOPS: 15)
        Scalar3 dx = deltaT * half_vel;

        // limit the movement of the particles
        if (limit)
            {
            Scalar len = sqrtf(dot(dx, dx));
            if (len > limit_val)
                dx = dx / len * limit_val;
            }

        // FLOPS: 3
        pos += dx;

        // update the velocity (FLOPS: 3)
        vel = half_vel;

        // read in the particle's image (MEM TRANSFER: 16 bytes)
        int3 image = d_image[idx];

        // fix the periodic boundary conditions (FLOPS: 15)
        box.wrap(pos, image);

        // write out the results (MEM_TRANSFER: 48 bytes)
        d_pos[idx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
        d_vel[idx] = make_scalar4(vel.x, vel.y, vel.z, velmass.w);
        d_image[idx] = image;
        }
    }

/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param box Box dimensions for periodic boundary condition handling
    \param deltaT timestep
    \param limit If \a limit is true, then the dynamics will be limited so that particles do not
   move a distance further than \a limit_val in one step. \param limit_val Length to limit particle
   distance movement to \param zero_force Set to true to always assign an acceleration of 0 to all
   particles in the group See gpu_rattle_nve_step_one_kernel() for full documentation, this function
   is just a driver.
*/
hipError_t gpu_rattle_nve_step_one(Scalar4* d_pos,
                                   Scalar4* d_vel,
                                   const Scalar3* d_accel,
                                   int3* d_image,
                                   unsigned int* d_group_members,
                                   const GPUPartition& gpu_partition,
                                   const BoxDim& box,
                                   Scalar deltaT,
                                   bool limit,
                                   Scalar limit_val,
                                   unsigned int block_size)
    {
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_rattle_nve_step_one_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid((nwork / run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        hipLaunchKernelGGL((gpu_rattle_nve_step_one_kernel),
                           dim3(grid),
                           dim3(threads),
                           0,
                           0,
                           d_pos,
                           d_vel,
                           d_accel,
                           d_image,
                           d_group_members,
                           nwork,
                           range.first,
                           box,
                           deltaT,
                           limit,
                           limit_val);
        }

    return hipSuccess;
    }

//! NO_SQUISH angular part of the first half step
/*! \param d_orientation array of particle orientations
    \param d_angmom array of particle conjugate quaternions
    \param d_inertia array of moments of inertia
    \param d_net_torque array of net torques
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param deltaT timestep
*/
__global__ void gpu_rattle_nve_angular_step_one_kernel(Scalar4* d_orientation,
                                                       Scalar4* d_angmom,
                                                       const Scalar3* d_inertia,
                                                       const Scalar4* d_net_torque,
                                                       const unsigned int* d_group_members,
                                                       const unsigned int nwork,
                                                       const unsigned int offset,
                                                       Scalar deltaT,
                                                       Scalar scale)
    {
    // determine which particle this thread works on (MEM TRANSFER: 4 bytes)
    int work_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (work_idx < nwork)
        {
        const unsigned int group_idx = work_idx + offset;
        unsigned int idx = d_group_members[group_idx];

        // read the particle's orientation, conjugate quaternion, moment of inertia and net torque
        quat<Scalar> q(d_orientation[idx]);
        quat<Scalar> p(d_angmom[idx]);
        vec3<Scalar> t(d_net_torque[idx]);
        vec3<Scalar> I(d_inertia[idx]);

        // rotate torque into principal frame
        t = rotate(conj(q), t);

        // check for zero moment of inertia
        bool x_zero, y_zero, z_zero;
        x_zero = (I.x == 0);
        y_zero = (I.y == 0);
        z_zero = (I.z == 0);

        // ignore torque component along an axis for which the moment of inertia zero
        if (x_zero)
            t.x = Scalar(0.0);
        if (y_zero)
            t.y = Scalar(0.0);
        if (z_zero)
            t.z = Scalar(0.0);

        // advance p(t)->p(t+deltaT/2), q(t)->q(t+deltaT)
        p += deltaT * q * t;

        p = p * scale;

        quat<Scalar> p1, p2, p3; // permutated quaternions
        quat<Scalar> q1, q2, q3;
        Scalar phi1, cphi1, sphi1;
        Scalar phi2, cphi2, sphi2;
        Scalar phi3, cphi3, sphi3;

        if (!z_zero)
            {
            p3 = quat<Scalar>(-p.v.z, vec3<Scalar>(p.v.y, -p.v.x, p.s));
            q3 = quat<Scalar>(-q.v.z, vec3<Scalar>(q.v.y, -q.v.x, q.s));
            phi3 = Scalar(1. / 4.) / I.z * dot(p, q3);
            cphi3 = slow::cos(Scalar(1. / 2.) * deltaT * phi3);
            sphi3 = slow::sin(Scalar(1. / 2.) * deltaT * phi3);

            p = cphi3 * p + sphi3 * p3;
            q = cphi3 * q + sphi3 * q3;
            }

        if (!y_zero)
            {
            p2 = quat<Scalar>(-p.v.y, vec3<Scalar>(-p.v.z, p.s, p.v.x));
            q2 = quat<Scalar>(-q.v.y, vec3<Scalar>(-q.v.z, q.s, q.v.x));
            phi2 = Scalar(1. / 4.) / I.y * dot(p, q2);
            cphi2 = slow::cos(Scalar(1. / 2.) * deltaT * phi2);
            sphi2 = slow::sin(Scalar(1. / 2.) * deltaT * phi2);

            p = cphi2 * p + sphi2 * p2;
            q = cphi2 * q + sphi2 * q2;
            }

        if (!x_zero)
            {
            p1 = quat<Scalar>(-p.v.x, vec3<Scalar>(p.s, p.v.z, -p.v.y));
            q1 = quat<Scalar>(-q.v.x, vec3<Scalar>(q.s, q.v.z, -q.v.y));
            phi1 = Scalar(1. / 4.) / I.x * dot(p, q1);
            cphi1 = slow::cos(deltaT * phi1);
            sphi1 = slow::sin(deltaT * phi1);

            p = cphi1 * p + sphi1 * p1;
            q = cphi1 * q + sphi1 * q1;
            }

        if (!y_zero)
            {
            p2 = quat<Scalar>(-p.v.y, vec3<Scalar>(-p.v.z, p.s, p.v.x));
            q2 = quat<Scalar>(-q.v.y, vec3<Scalar>(-q.v.z, q.s, q.v.x));
            phi2 = Scalar(1. / 4.) / I.y * dot(p, q2);
            cphi2 = slow::cos(Scalar(1. / 2.) * deltaT * phi2);
            sphi2 = slow::sin(Scalar(1. / 2.) * deltaT * phi2);

            p = cphi2 * p + sphi2 * p2;
            q = cphi2 * q + sphi2 * q2;
            }

        if (!z_zero)
            {
            p3 = quat<Scalar>(-p.v.z, vec3<Scalar>(p.v.y, -p.v.x, p.s));
            q3 = quat<Scalar>(-q.v.z, vec3<Scalar>(q.v.y, -q.v.x, q.s));
            phi3 = Scalar(1. / 4.) / I.z * dot(p, q3);
            cphi3 = slow::cos(Scalar(1. / 2.) * deltaT * phi3);
            sphi3 = slow::sin(Scalar(1. / 2.) * deltaT * phi3);

            p = cphi3 * p + sphi3 * p3;
            q = cphi3 * q + sphi3 * q3;
            }

        // renormalize (improves stability)
        q = q * (Scalar(1.0) / slow::sqrt(norm2(q)));

        d_orientation[idx] = quat_to_scalar4(q);
        d_angmom[idx] = quat_to_scalar4(p);
        }
    }

/*! \param d_orientation array of particle orientations
    \param d_angmom array of particle conjugate quaternions
    \param d_inertia array of moments of inertia
    \param d_net_torque array of net torques
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param deltaT timestep
*/
hipError_t gpu_rattle_nve_angular_step_one(Scalar4* d_orientation,
                                           Scalar4* d_angmom,
                                           const Scalar3* d_inertia,
                                           const Scalar4* d_net_torque,
                                           unsigned int* d_group_members,
                                           const GPUPartition& gpu_partition,
                                           Scalar deltaT,
                                           Scalar scale,
                                           const unsigned int block_size)
    {
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_rattle_nve_angular_step_one_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid((nwork / run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        hipLaunchKernelGGL((gpu_rattle_nve_angular_step_one_kernel),
                           dim3(grid),
                           dim3(threads),
                           0,
                           0,
                           d_orientation,
                           d_angmom,
                           d_inertia,
                           d_net_torque,
                           d_group_members,
                           nwork,
                           range.first,
                           deltaT,
                           scale);
        }

    return hipSuccess;
    }

//! NO_SQUISH angular part of the second half step
/*! \param d_orientation array of particle orientations
    \param d_angmom array of particle conjugate quaternions
    \param d_inertia array of moments of inertia
    \param d_net_torque array of net torques
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param deltaT timestep
*/

__global__ void gpu_rattle_nve_angular_step_two_kernel(const Scalar4* d_orientation,
                                                       Scalar4* d_angmom,
                                                       const Scalar3* d_inertia,
                                                       const Scalar4* d_net_torque,
                                                       unsigned int* d_group_members,
                                                       const unsigned int nwork,
                                                       const unsigned int offset,
                                                       Scalar deltaT,
                                                       Scalar scale)
    {
    // determine which particle this thread works on (MEM TRANSFER: 4 bytes)
    int work_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (work_idx < nwork)
        {
        const unsigned int group_idx = work_idx + offset;
        unsigned int idx = d_group_members[group_idx];

        // read the particle's orientation, conjugate quaternion, moment of inertia and net torque
        quat<Scalar> q(d_orientation[idx]);
        quat<Scalar> p(d_angmom[idx]);
        vec3<Scalar> t(d_net_torque[idx]);
        vec3<Scalar> I(d_inertia[idx]);

        // rotate torque into principal frame
        t = rotate(conj(q), t);

        // check for zero moment of inertia
        bool x_zero, y_zero, z_zero;
        x_zero = (I.x == 0);
        y_zero = (I.y == 0);
        z_zero = (I.z == 0);

        // ignore torque component along an axis for which the moment of inertia zero
        if (x_zero)
            t.x = Scalar(0.0);
        if (y_zero)
            t.y = Scalar(0.0);
        if (z_zero)
            t.z = Scalar(0.0);

        // rescale
        p = p * scale;

        // advance p(t)->p(t+deltaT/2), q(t)->q(t+deltaT)
        p += deltaT * q * t;

        d_angmom[idx] = quat_to_scalar4(p);
        }
    }

/*! \param d_orientation array of particle orientations
    \param d_angmom array of particle conjugate quaternions
    \param d_inertia array of moments of inertia
    \param d_net_torque array of net torques
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param deltaT timestep
*/
hipError_t gpu_rattle_nve_angular_step_two(const Scalar4* d_orientation,
                                           Scalar4* d_angmom,
                                           const Scalar3* d_inertia,
                                           const Scalar4* d_net_torque,
                                           unsigned int* d_group_members,
                                           const GPUPartition& gpu_partition,
                                           Scalar deltaT,
                                           Scalar scale,
                                           const unsigned int block_size)
    {
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_rattle_nve_angular_step_two_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid((nwork / run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        hipLaunchKernelGGL((gpu_rattle_nve_angular_step_two_kernel),
                           dim3(grid),
                           dim3(threads),
                           0,
                           0,
                           d_orientation,
                           d_angmom,
                           d_inertia,
                           d_net_torque,
                           d_group_members,
                           nwork,
                           range.first,
                           deltaT,
                           scale);
        }

    return hipSuccess;
    }
    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
