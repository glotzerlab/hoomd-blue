#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: mphoward

/*!
 * \file mpcd/SRDCollisionMethodGPU.cu
 * \brief Defines GPU functions and kernels used by mpcd::SRDCollisionMethodGPU
 */

#include "SRDCollisionMethodGPU.cuh"
#include "hoomd/RandomNumbers.h"
#include "hoomd/RNGIdentifiers.h"

namespace mpcd
{
namespace gpu
{
namespace kernel
{
template<bool use_thermostat>
__global__ void srd_draw_vectors(double3 *d_rotvec,
                                 double *d_factors,
                                 const double3 *d_cell_energy,
                                 const Index3D ci,
                                 const int3 origin,
                                 const uint3 global_dim,
                                 const Index3D global_ci,
                                 const unsigned int timestep,
                                 const unsigned int seed,
                                 const Scalar T_set,
                                 const unsigned int n_dimensions,
                                 const unsigned int Ncell)
    {
    // one thread per cell
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= Ncell)
        return;

    // get local cell triple from 1d index
    const uint3 cell = ci.getTriple(idx);
    // shift local cell by local origin, and wrap through global boundaries
    int3 global_cell = make_int3(origin.x + (int)cell.x,
                                 origin.y + (int)cell.y,
                                 origin.z + (int)cell.z);
    if (global_cell.x >= (int)global_dim.x) global_cell.x -= global_dim.x;
    else if (global_cell.x < 0) global_cell.x += global_dim.x;

    if (global_cell.y >= (int)global_dim.y) global_cell.y -= global_dim.y;
    else if (global_cell.y < 0) global_cell.y += global_dim.y;

    if (global_cell.z >= (int)global_dim.z) global_cell.z -= global_dim.z;
    else if (global_cell.z < 0) global_cell.z += global_dim.z;

    // convert global triple to 1d global index
    const unsigned int global_idx = global_ci(global_cell.x, global_cell.y, global_cell.z);

    // Initialize the PRNG using the cell index, timestep, and seed for the hash
    hoomd::RandomGenerator rng(hoomd::RNGIdentifier::SRDCollisionMethod, seed, global_idx, timestep);

    // draw rotation vector off the surface of the sphere
    double3 rotvec;
    hoomd::SpherePointGenerator<double> sphgen;
    sphgen(rng, rotvec);
    d_rotvec[idx] = rotvec;

    if (use_thermostat)
        {
        const double3 cell_energy = d_cell_energy[idx];
        const unsigned int np = __double_as_int(cell_energy.z);
        double factor = 1.0;
        if (np > 1)
            {
            // the total number of degrees of freedom in the cell divided by 2
            const double alpha = n_dimensions*(np-1)/(double)2.;

            // draw a random kinetic energy for the cell at the set temperature
            hoomd::GammaDistribution<double> gamma_gen(alpha,T_set);
            const double rand_ke = gamma_gen(rng);

            // generate the scale factor from the current temperature
            // (don't use the kinetic energy of this cell, since this
            // is total not relative to COM)
            const double cur_ke = alpha * cell_energy.y;
            factor = (cur_ke > 0.) ? fast::sqrt(rand_ke/cur_ke) : 1.;
            }
        d_factors[idx] = factor;
        }
    }
__global__ void srd_rotate(Scalar4 *d_vel,
                           Scalar4 *d_vel_embed,
                           const unsigned int *d_embed_group,
                           const unsigned int *d_embed_cell_ids,
                           const double4 *d_cell_vel,
                           const double3 *d_rotvec,
                           const double cos_a,
                           const double one_minus_cos_a,
                           const double sin_a,
                           const double *d_factors,
                           const unsigned int N_mpcd,
                           const unsigned int N_tot)
    {
    // one thread per particle
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N_tot)
        return;

    // load particle data
    double3 vel;
    unsigned int cell;
    // these properties are needed for the embedded particles only
    unsigned int idx(0); double mass(0);
    if (tid < N_mpcd)
        {
        const Scalar4 vel_cell = d_vel[tid];
        vel = make_double3(vel_cell.x, vel_cell.y, vel_cell.z);
        cell = __scalar_as_int(vel_cell.w);
        }
    else
        {
        idx = d_embed_group[tid - N_mpcd];

        const Scalar4 vel_mass = d_vel_embed[idx];
        vel = make_double3(vel_mass.x, vel_mass.y, vel_mass.z);
        mass = vel_mass.w;
        cell = d_embed_cell_ids[tid - N_mpcd];
        }

    // subtract average velocity
    const double4 avg_vel = d_cell_vel[cell];
    vel.x -= avg_vel.x;
    vel.y -= avg_vel.y;
    vel.z -= avg_vel.z;

    // get rotation vector
    double3 rot_vec = d_rotvec[cell];

    // perform the rotation in double precision
    double3 new_vel;
    new_vel.x = (cos_a + rot_vec.x*rot_vec.x*one_minus_cos_a) * vel.x;
    new_vel.x += (rot_vec.x*rot_vec.y*one_minus_cos_a - sin_a*rot_vec.z) * vel.y;
    new_vel.x += (rot_vec.x*rot_vec.z*one_minus_cos_a + sin_a*rot_vec.y) * vel.z;

    new_vel.y = (cos_a + rot_vec.y*rot_vec.y*one_minus_cos_a) * vel.y;
    new_vel.y += (rot_vec.x*rot_vec.y*one_minus_cos_a + sin_a*rot_vec.z) * vel.x;
    new_vel.y += (rot_vec.y*rot_vec.z*one_minus_cos_a - sin_a*rot_vec.x) * vel.z;

    new_vel.z = (cos_a + rot_vec.z*rot_vec.z*one_minus_cos_a) * vel.z;
    new_vel.z += (rot_vec.x*rot_vec.z*one_minus_cos_a - sin_a*rot_vec.y) * vel.x;
    new_vel.z += (rot_vec.y*rot_vec.z*one_minus_cos_a + sin_a*rot_vec.x) * vel.y;

    // rescale the velocity if factor is available
    if (d_factors != NULL)
        {
        const double factor = d_factors[cell];
        new_vel.x *= factor; new_vel.y *= factor; new_vel.z *= factor;
        }

    new_vel.x += avg_vel.x;
    new_vel.y += avg_vel.y;
    new_vel.z += avg_vel.z;

    // set the new velocity
    if (tid < N_mpcd)
        {
        d_vel[tid] = make_scalar4(new_vel.x, new_vel.y, new_vel.z, __int_as_scalar(cell));
        }
    else
        {
        d_vel_embed[idx] = make_scalar4(new_vel.x, new_vel.y, new_vel.z, mass);
        }
    }
} // end namespace kernel

hipError_t srd_draw_vectors(double3 *d_rotvec,
                             double *d_factors,
                             const double3 *d_cell_energy,
                             const Index3D& ci,
                             const int3 origin,
                             const uint3 global_dim,
                             const Index3D& global_ci,
                             const unsigned int timestep,
                             const unsigned int seed,
                             const Scalar T_set,
                             const unsigned int n_dimensions,
                             const unsigned int block_size)
    {

    if (d_factors != NULL)
        {
        static unsigned int max_block_thermostat = UINT_MAX;
        if (max_block_thermostat == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::srd_draw_vectors<true>);
            max_block_thermostat = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_thermostat);

        const unsigned int Ncell = ci.getNumElements();
        dim3 grid(Ncell / run_block_size + 1);
        mpcd::gpu::kernel::srd_draw_vectors<true><<<grid, run_block_size>>>(d_rotvec,
                                                                            d_factors,
                                                                            d_cell_energy,
                                                                            ci,
                                                                            origin,
                                                                            global_dim,
                                                                            global_ci,
                                                                            timestep,
                                                                            seed,
                                                                            T_set,
                                                                            n_dimensions,
                                                                            Ncell);
        }
    else
        {
        static unsigned int max_block_nothermostat = UINT_MAX;
        if (max_block_nothermostat == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::srd_draw_vectors<false>);
            max_block_nothermostat = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_nothermostat);

        const unsigned int Ncell = ci.getNumElements();
        dim3 grid(Ncell / run_block_size + 1);
        mpcd::gpu::kernel::srd_draw_vectors<false><<<grid, run_block_size>>>(d_rotvec,
                                                                             d_factors,
                                                                             d_cell_energy,
                                                                             ci,
                                                                             origin,
                                                                             global_dim,
                                                                             global_ci,
                                                                             timestep,
                                                                             seed,
                                                                             T_set,
                                                                             n_dimensions,
                                                                             Ncell);
        }

    return hipSuccess;
    }

hipError_t srd_rotate(Scalar4 *d_vel,
                       Scalar4 *d_vel_embed,
                       const unsigned int *d_embed_group,
                       const unsigned int *d_embed_cell_ids,
                       const double4 *d_cell_vel,
                       const double3 *d_rotvec,
                       const double angle,
                       const double *d_factors,
                       const unsigned int N_mpcd,
                       const unsigned int N_tot,
                       const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))mpcd::gpu::kernel::srd_rotate);
        max_block_size = attr.maxThreadsPerBlock;
        }

    // precompute angles for rotation
    const double cos_a = slow::cos(angle);
    const double one_minus_cos_a = 1.0 - cos_a;
    const double sin_a = slow::sin(angle);

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(N_tot / run_block_size + 1);
    mpcd::gpu::kernel::srd_rotate<<<grid, run_block_size>>>(d_vel,
                                                            d_vel_embed,
                                                            d_embed_group,
                                                            d_embed_cell_ids,
                                                            d_cell_vel,
                                                            d_rotvec,
                                                            cos_a,
                                                            one_minus_cos_a,
                                                            sin_a,
                                                            d_factors,
                                                            N_mpcd,
                                                            N_tot);

    return hipSuccess;
    }

} // end namespace gpu
} // end namespace mpcd
