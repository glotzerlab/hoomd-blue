#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "hoomd/Index1D.h"
#include "hoomd/ParticleData.cuh"
#include "hoomd/VectorMath.h"

#include "ForceCompositeGPU.cuh"

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wconversion"
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#pragma GCC diagnostic pop

// Maintainer: jglaser

/*! \file ForceComposite.cu
    \brief Defines GPU kernel code for the composite particle integration on the GPU.
*/

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
//! Calculates the body forces and torques by summing the constituent particle forces using a fixed
//! sliding window size
/*  Compute the force and torque sum on all bodies in the system from their constituent particles.
   n_bodies_per_block bodies are handled within each block of execution on the GPU. The reason for
   this is to decrease over-parallelism and use the GPU cores more effectively when bodies are
   smaller than the block size. Otherwise, small bodies leave many threads in the block idle with
   nothing to do.

    On start, the properties common to each body are read in, computed, and stored in shared memory
   for all the threads working on that body to access. Then, the threads loop over all particles
   that are part of the body with a sliding window. Each loop of the window computes the force and
   torque for block_size/n_bodies_per_block particles in as many threads in parallel. These
   quantities are summed over enough windows to cover the whole body.

    The block_size/n_bodies_per_block partial sums are stored in shared memory. Then
   n_bodies_per_block partial reductions are performed in parallel using all threads to sum the
   total force and torque on each body. This looks just like a normal reduction, except that it
   terminates at a certain level in the tree. To make the math for the partial reduction work out,
   block_size must be a power of 2 as must n_bodies_per_block.

    Performance testing on GF100 with many different bodies of different sizes ranging from 4-256
   particles per body has found that the optimum block size for most bodies is 64 threads.
   Performance increases for all body sizes as n_bodies_per_block is increased, but only up to 8.
   n_bodies_per_block=16 slows performance significantly. Based on these performance results, this
   kernel is hardcoded to handle only 1,2,4,8 n_bodies_per_block with a power of 2 block size
   (hardcoded to 64 in the kernel launch).
*/
__global__ void gpu_rigid_force_sliding_kernel(Scalar4* d_force,
                                               Scalar4* d_torque,
                                               const unsigned int* d_molecule_len,
                                               const unsigned int* d_molecule_list,
                                               const unsigned int* d_molecule_idx,
                                               const unsigned int* d_rigid_center,
                                               Index2D molecule_indexer,
                                               const Scalar4* d_postype,
                                               const Scalar4* d_orientation,
                                               Index2D body_indexer,
                                               Scalar3* d_body_pos,
                                               Scalar4* d_body_orientation,
                                               const unsigned int* d_body_len,
                                               const unsigned int* d_body,
                                               const unsigned int* d_tag,
                                               uint2* d_flag,
                                               Scalar4* d_net_force,
                                               Scalar4* d_net_torque,
                                               unsigned int n_mol,
                                               unsigned int N,
                                               unsigned int window_size,
                                               unsigned int thread_mask,
                                               unsigned int n_bodies_per_block,
                                               bool zero_force,
                                               unsigned int first_body,
                                               unsigned int nwork)
    {
    extern __shared__ char sum[];

    // determine which body (0 ... n_bodies_per_block-1) this thread is working on
    // assign threads 0, 1, 2, ... to body 0, n, n+1, n+2, ... to body 1, and so on.
    unsigned int m = threadIdx.x / (blockDim.x / n_bodies_per_block);

    // arrays in shared memory
    Scalar4* body_force = (Scalar4*)sum;                 // blockDim.x elements
    Scalar4* body_orientation = body_force + blockDim.x; // n_bodies_per_block elements
    Scalar3* body_torque = (Scalar3*)(body_orientation + n_bodies_per_block); // blockDim.x elements
    unsigned int* body_type
        = (unsigned int*)(body_torque + blockDim.x);          // n_bodies_per_block elements
    unsigned int* mol_idx = body_type + n_bodies_per_block;   // n_bodies_per_block elements
    unsigned int* central_idx = mol_idx + n_bodies_per_block; // n_bodies_per_block elements

    // each thread makes partial sums of force and torque of all the particles that this thread
    // loops over
    Scalar4 sum_force = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
    Scalar3 sum_torque = make_scalar3(Scalar(0.0), Scalar(0.0), Scalar(0.0));

    // thread_mask is a bitmask that masks out the high bits in threadIdx.x.
    // threadIdx.x & thread_mask is an index from 0 to block_size/n_bodies_per_block-1 and
    // determines what offset this thread is to use when accessing the particles in the body
    if ((threadIdx.x & thread_mask) == 0)
        {
        // thread 0 for this body reads in the body id and orientation and stores them in shared
        // memory
        int group_idx = blockIdx.x * n_bodies_per_block + m;
        if (group_idx < nwork)
            {
            central_idx[m] = d_rigid_center[group_idx + first_body];
            mol_idx[m] = d_molecule_idx[central_idx[m]];

            if (d_tag[central_idx[m]] != d_body[central_idx[m]])
                {
                // this is not the central ptl, molecule is incomplete - mark as such
                body_type[m] = 0xffffffff;
                body_orientation[m] = make_scalar4(1, 0, 0, 0);
                }
            else
                {
                body_type[m] = __scalar_as_int(d_postype[central_idx[m]].w);
                body_orientation[m] = d_orientation[central_idx[m]];
                }
            }
        else
            {
            mol_idx[m] = NO_BODY;
            }
        }

    __syncthreads();

    if (mol_idx[m] < MIN_FLOPPY)
        {
        // compute the number of windows that we need to loop over
        unsigned int mol_len = d_molecule_len[mol_idx[m]];
        unsigned int n_windows = mol_len / window_size + 1;

        // slide the window throughout the block
        for (unsigned int start = 0; start < n_windows; start++)
            {
            // determine the index with this body that this particle should handle
            unsigned int k = start * window_size + (threadIdx.x & thread_mask);

            // if that index is in the body we are actually handling a real body
            if (k < mol_len)
                {
                // determine the particle idx of the particle
                unsigned int pidx = d_molecule_list[molecule_indexer(k, mol_idx[m])];

                // if this particle is not the central particle
                if (body_type[m] != 0xffffffff && pidx != central_idx[m])
                    {
                    Scalar4 fi = d_net_force[pidx];

                    // will likely need to rotate these components too
                    vec3<Scalar> ti(d_net_torque[pidx]);

                    // zero net torque on constituent particles
                    d_net_torque[pidx] = make_scalar4(0.0, 0.0, 0.0, 0.0);

                    // zero force only if we don't need it later
                    if (zero_force)
                        {
                        // zero net energy on constituent ptls to avoid double counting
                        // also zero net force for consistency
                        d_net_force[pidx] = make_scalar4(0.0, 0.0, 0.0, 0.0);
                        }

                    if (central_idx[m] < N)
                        {
                        // at this point, the molecule needs to be complete
                        if (mol_len != d_body_len[body_type[m]] + 1)
                            {
                            // incomplete molecule
                            atomicMax(&(d_flag->x), d_body[central_idx[m]] + 1);
                            }

                        // calculate body force and torques
                        vec3<Scalar> particle_pos(d_body_pos[body_indexer(body_type[m], k - 1)]);

                        // tally the force in the per thread counter
                        sum_force.x += fi.x;
                        sum_force.y += fi.y;
                        sum_force.z += fi.z;

                        // sum up energy
                        sum_force.w += fi.w;

                        vec3<Scalar> ri = rotate(quat<Scalar>(body_orientation[m]), particle_pos);

                        // torque = r x f
                        vec3<Scalar> del_torque(cross(ri, vec3<Scalar>(fi)));

                        // tally the torque in the per thread counter
                        sum_torque.x += ti.x + del_torque.x;
                        sum_torque.y += ti.y + del_torque.y;
                        sum_torque.z += ti.z + del_torque.z;
                        }
                    }
                }
            }
        }

    __syncthreads();

    // put the partial sums into shared memory
    body_force[threadIdx.x] = sum_force;
    body_torque[threadIdx.x] = sum_torque;

    __syncthreads();

    // perform a set of partial reductions. Each block_size/n_bodies_per_block threads performs a
    // sum reduction just within its own group
    unsigned int offset = window_size >> 1;
    while (offset > 0)
        {
        if ((threadIdx.x & thread_mask) < offset)
            {
            body_force[threadIdx.x].x += body_force[threadIdx.x + offset].x;
            body_force[threadIdx.x].y += body_force[threadIdx.x + offset].y;
            body_force[threadIdx.x].z += body_force[threadIdx.x + offset].z;
            body_force[threadIdx.x].w += body_force[threadIdx.x + offset].w;

            body_torque[threadIdx.x].x += body_torque[threadIdx.x + offset].x;
            body_torque[threadIdx.x].y += body_torque[threadIdx.x + offset].y;
            body_torque[threadIdx.x].z += body_torque[threadIdx.x + offset].z;
            }

        offset >>= 1;

        __syncthreads();
        }

    // thread 0 within this body writes out the total force and torque for the body
    if ((threadIdx.x & thread_mask) == 0 && mol_idx[m] < MIN_FLOPPY && central_idx[m] < N)
        {
        d_force[central_idx[m]] = body_force[threadIdx.x];
        d_torque[central_idx[m]] = make_scalar4(body_torque[threadIdx.x].x,
                                                body_torque[threadIdx.x].y,
                                                body_torque[threadIdx.x].z,
                                                0.0f);
        }
    }

__global__ void gpu_rigid_virial_sliding_kernel(Scalar* d_virial,
                                                const unsigned int* d_molecule_len,
                                                const unsigned int* d_molecule_list,
                                                const unsigned int* d_molecule_idx,
                                                const unsigned int* d_rigid_center,
                                                Index2D molecule_indexer,
                                                const Scalar4* d_postype,
                                                const Scalar4* d_orientation,
                                                Index2D body_indexer,
                                                Scalar3* d_body_pos,
                                                Scalar4* d_body_orientation,
                                                Scalar4* d_net_force,
                                                Scalar* d_net_virial,
                                                const unsigned int* d_body,
                                                const unsigned int* d_tag,
                                                unsigned int n_mol,
                                                unsigned int N,
                                                size_t net_virial_pitch,
                                                size_t virial_pitch,
                                                unsigned int window_size,
                                                unsigned int thread_mask,
                                                unsigned int n_bodies_per_block,
                                                unsigned int first_body,
                                                unsigned int nwork)
    {
    extern __shared__ char sum[];

    // determine which body (0 ... n_bodies_per_block-1) this thread is working on
    // assign threads 0, 1, 2, ... to body 0, n, n+1, n+2, ... to body 1, and so on.
    unsigned int m = threadIdx.x / (blockDim.x / n_bodies_per_block);

    // arrays in shared memory
    Scalar4* body_orientation = (Scalar4*)sum; // n_bodies_per_block elements
    Scalar* sum_virial = (Scalar*)(body_orientation + n_bodies_per_block); // 6*blockDim.x elements
    unsigned int* body_type
        = (unsigned int*)(sum_virial + 6 * blockDim.x);       // n_bodies_per_block elements
    unsigned int* mol_idx = body_type + n_bodies_per_block;   // n_bodies_per_block elements
    unsigned int* central_idx = mol_idx + n_bodies_per_block; // n_bodies_per_block elements

    // body_force and body_torque are each shared memory arrays with 1 element per threads
    Scalar* body_virial_xx = sum_virial;
    Scalar* body_virial_xy = &sum_virial[1 * blockDim.x];
    Scalar* body_virial_xz = &sum_virial[2 * blockDim.x];
    Scalar* body_virial_yy = &sum_virial[3 * blockDim.x];
    Scalar* body_virial_yz = &sum_virial[4 * blockDim.x];
    Scalar* body_virial_zz = &sum_virial[5 * blockDim.x];

    // each thread makes partial sums of the virial of all the particles that this thread loops over
    Scalar sum_virial_xx(0.0);
    Scalar sum_virial_xy(0.0);
    Scalar sum_virial_xz(0.0);
    Scalar sum_virial_yy(0.0);
    Scalar sum_virial_yz(0.0);
    Scalar sum_virial_zz(0.0);

    // thread_mask is a bitmask that masks out the high bits in threadIdx.x.
    // threadIdx.x & thread_mask is an index from 0 to block_size/n_bodies_per_block-1 and
    // determines what offset this thread is to use when accessing the particles in the body
    if ((threadIdx.x & thread_mask) == 0)
        {
        // thread 0 for this body reads in the body id and orientation and stores them in shared
        // memory
        int group_idx = blockIdx.x * n_bodies_per_block + m;
        if (group_idx < nwork)
            {
            central_idx[m] = d_rigid_center[group_idx + first_body];
            mol_idx[m] = d_molecule_idx[central_idx[m]];

            if (d_tag[central_idx[m]] != d_body[central_idx[m]])
                {
                // this is not the central ptl, molecule is incomplete - mark as such
                body_type[m] = NO_BODY;
                body_orientation[m] = make_scalar4(1, 0, 0, 0);
                }
            else
                {
                body_type[m] = __scalar_as_int(d_postype[central_idx[m]].w);
                body_orientation[m] = d_orientation[central_idx[m]];
                }
            }
        else
            {
            mol_idx[m] = NO_BODY;
            }
        }

    __syncthreads();

    if (mol_idx[m] < MIN_FLOPPY)
        {
        // compute the number of windows that we need to loop over
        unsigned int mol_len = d_molecule_len[mol_idx[m]];
        unsigned int n_windows = mol_len / window_size + 1;

        // slide the window throughout the block
        for (unsigned int start = 0; start < n_windows; start++)
            {
            // determine the index with this body that this particle should handle
            unsigned int k = start * window_size + (threadIdx.x & thread_mask);

            // if that index is in the body we are actually handling a real body
            if (k < mol_len)
                {
                // determine the particle idx of the particle
                unsigned int pidx = d_molecule_list[molecule_indexer(k, mol_idx[m])];

                if (body_type[m] < MIN_FLOPPY && pidx != central_idx[m])
                    {
                    // calculate body force and torques
                    Scalar4 fi = d_net_force[pidx];

                    // sum up virial
                    Scalar virialxx = d_net_virial[0 * net_virial_pitch + pidx];
                    Scalar virialxy = d_net_virial[1 * net_virial_pitch + pidx];
                    Scalar virialxz = d_net_virial[2 * net_virial_pitch + pidx];
                    Scalar virialyy = d_net_virial[3 * net_virial_pitch + pidx];
                    Scalar virialyz = d_net_virial[4 * net_virial_pitch + pidx];
                    Scalar virialzz = d_net_virial[5 * net_virial_pitch + pidx];

                    // zero force and virial on constituent particles
                    d_net_force[pidx] = make_scalar4(0.0, 0.0, 0.0, 0.0);

                    d_net_virial[0 * net_virial_pitch + pidx] = Scalar(0.0);
                    d_net_virial[1 * net_virial_pitch + pidx] = Scalar(0.0);
                    d_net_virial[2 * net_virial_pitch + pidx] = Scalar(0.0);
                    d_net_virial[3 * net_virial_pitch + pidx] = Scalar(0.0);
                    d_net_virial[4 * net_virial_pitch + pidx] = Scalar(0.0);
                    d_net_virial[5 * net_virial_pitch + pidx] = Scalar(0.0);

                    // if this particle is not the central particle (incomplete molecules can't have
                    // local members)
                    if (central_idx[m] < N)
                        {
                        vec3<Scalar> particle_pos(d_body_pos[body_indexer(body_type[m], k - 1)]);
                        vec3<Scalar> ri = rotate(quat<Scalar>(body_orientation[m]), particle_pos);

                        // subtract intra-body virial prt
                        sum_virial_xx += virialxx - fi.x * ri.x;
                        sum_virial_xy += virialxy - fi.x * ri.y;
                        sum_virial_xz += virialxz - fi.x * ri.z;
                        sum_virial_yy += virialyy - fi.y * ri.y;
                        sum_virial_yz += virialyz - fi.y * ri.z;
                        sum_virial_zz += virialzz - fi.z * ri.z;
                        }
                    }
                }
            }
        }

    __syncthreads();

    // put the partial sums into shared memory
    body_virial_xx[threadIdx.x] = sum_virial_xx;
    body_virial_xy[threadIdx.x] = sum_virial_xy;
    body_virial_xz[threadIdx.x] = sum_virial_xz;
    body_virial_yy[threadIdx.x] = sum_virial_yy;
    body_virial_yz[threadIdx.x] = sum_virial_yz;
    body_virial_zz[threadIdx.x] = sum_virial_zz;

    __syncthreads();

    // perform a set of partial reductions. Each block_size/n_bodies_per_block threads performs a
    // sum reduction just within its own group
    unsigned int offset = window_size >> 1;
    while (offset > 0)
        {
        if ((threadIdx.x & thread_mask) < offset)
            {
            body_virial_xx[threadIdx.x] += body_virial_xx[threadIdx.x + offset];
            body_virial_xy[threadIdx.x] += body_virial_xy[threadIdx.x + offset];
            body_virial_xz[threadIdx.x] += body_virial_xz[threadIdx.x + offset];
            body_virial_yy[threadIdx.x] += body_virial_yy[threadIdx.x + offset];
            body_virial_yz[threadIdx.x] += body_virial_yz[threadIdx.x + offset];
            body_virial_zz[threadIdx.x] += body_virial_zz[threadIdx.x + offset];
            }

        offset >>= 1;

        __syncthreads();
        }

    // thread 0 within this body writes out the total virial for the body
    if ((threadIdx.x & thread_mask) == 0 && mol_idx[m] < MIN_FLOPPY && central_idx[m] < N)
        {
        d_virial[0 * virial_pitch + central_idx[m]] = body_virial_xx[threadIdx.x];
        d_virial[1 * virial_pitch + central_idx[m]] = body_virial_xy[threadIdx.x];
        d_virial[2 * virial_pitch + central_idx[m]] = body_virial_xz[threadIdx.x];
        d_virial[3 * virial_pitch + central_idx[m]] = body_virial_yy[threadIdx.x];
        d_virial[4 * virial_pitch + central_idx[m]] = body_virial_yz[threadIdx.x];
        d_virial[5 * virial_pitch + central_idx[m]] = body_virial_zz[threadIdx.x];
        }
    }

/*!
 */
hipError_t gpu_rigid_force(Scalar4* d_force,
                           Scalar4* d_torque,
                           const unsigned int* d_molecule_len,
                           const unsigned int* d_molecule_list,
                           const unsigned int* d_molecule_idx,
                           const unsigned int* d_rigid_center,
                           Index2D molecule_indexer,
                           const Scalar4* d_postype,
                           const Scalar4* d_orientation,
                           Index2D body_indexer,
                           Scalar3* d_body_pos,
                           Scalar4* d_body_orientation,
                           const unsigned int* d_body_len,
                           const unsigned int* d_body,
                           const unsigned int* d_tag,
                           uint2* d_flag,
                           Scalar4* d_net_force,
                           Scalar4* d_net_torque,
                           unsigned int n_mol,
                           unsigned int N,
                           unsigned int n_bodies_per_block,
                           unsigned int block_size,
                           const hipDeviceProp_t& dev_prop,
                           bool zero_force,
                           const GPUPartition& gpu_partition)
    {
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        dim3 force_grid(nwork / n_bodies_per_block + 1, 1, 1);

        unsigned int max_block_size;
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_rigid_force_sliding_kernel);
        max_block_size = attr.maxThreadsPerBlock;

        unsigned int run_block_size = max_block_size < block_size ? max_block_size : block_size;

        // round down to nearest power of two
        unsigned int b = 1;
        while (b * 2 <= run_block_size)
            {
            b *= 2;
            }
        run_block_size = b;

        unsigned int window_size = run_block_size / n_bodies_per_block;
        unsigned int thread_mask = window_size - 1;

        size_t shared_bytes = run_block_size * (sizeof(Scalar4) + sizeof(Scalar3))
                              + n_bodies_per_block * (sizeof(Scalar4) + 3 * sizeof(unsigned int));

        while (shared_bytes + attr.sharedSizeBytes >= dev_prop.sharedMemPerBlock)
            {
            // block size is power of two
            run_block_size /= 2;

            shared_bytes = run_block_size * (sizeof(Scalar4) + sizeof(Scalar3))
                           + n_bodies_per_block * (sizeof(Scalar4) + 3 * sizeof(unsigned int));

            window_size = run_block_size / n_bodies_per_block;
            thread_mask = window_size - 1;
            }

        hipLaunchKernelGGL((gpu_rigid_force_sliding_kernel),
                           dim3(force_grid),
                           dim3(run_block_size),
                           shared_bytes,
                           0,
                           d_force,
                           d_torque,
                           d_molecule_len,
                           d_molecule_list,
                           d_molecule_idx,
                           d_rigid_center,
                           molecule_indexer,
                           d_postype,
                           d_orientation,
                           body_indexer,
                           d_body_pos,
                           d_body_orientation,
                           d_body_len,
                           d_body,
                           d_tag,
                           d_flag,
                           d_net_force,
                           d_net_torque,
                           n_mol,
                           N,
                           window_size,
                           thread_mask,
                           n_bodies_per_block,
                           zero_force,
                           range.first,
                           nwork);
        }
    return hipSuccess;
    }

hipError_t gpu_rigid_virial(Scalar* d_virial,
                            const unsigned int* d_molecule_len,
                            const unsigned int* d_molecule_list,
                            const unsigned int* d_molecule_idx,
                            const unsigned int* d_rigid_center,
                            Index2D molecule_indexer,
                            const Scalar4* d_postype,
                            const Scalar4* d_orientation,
                            Index2D body_indexer,
                            Scalar3* d_body_pos,
                            Scalar4* d_body_orientation,
                            Scalar4* d_net_force,
                            Scalar* d_net_virial,
                            const unsigned int* d_body,
                            const unsigned int* d_tag,
                            unsigned int n_mol,
                            unsigned int N,
                            unsigned int n_bodies_per_block,
                            size_t net_virial_pitch,
                            size_t virial_pitch,
                            unsigned int block_size,
                            const hipDeviceProp_t& dev_prop,
                            const GPUPartition& gpu_partition)
    {
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        dim3 force_grid(nwork / n_bodies_per_block + 1, 1, 1);

        unsigned int max_block_size;
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_rigid_virial_sliding_kernel);
        max_block_size = attr.maxThreadsPerBlock;

        unsigned int run_block_size = max_block_size < block_size ? max_block_size : block_size;

        // round down to nearest power of two
        unsigned int b = 1;
        while (b * 2 <= run_block_size)
            {
            b *= 2;
            }
        run_block_size = b;

        unsigned int window_size = run_block_size / n_bodies_per_block;
        unsigned int thread_mask = window_size - 1;

        size_t shared_bytes = 6 * run_block_size * sizeof(Scalar)
                              + n_bodies_per_block * (sizeof(Scalar4) + 3 * sizeof(unsigned int));

        while (shared_bytes + attr.sharedSizeBytes >= dev_prop.sharedMemPerBlock)
            {
            // block size is power of two
            run_block_size /= 2;

            shared_bytes = 6 * run_block_size * sizeof(Scalar)
                           + n_bodies_per_block * (sizeof(Scalar4) + 3 * sizeof(unsigned int));

            window_size = run_block_size / n_bodies_per_block;
            thread_mask = window_size - 1;
            }

        hipLaunchKernelGGL((gpu_rigid_virial_sliding_kernel),
                           dim3(force_grid),
                           dim3(run_block_size),
                           shared_bytes,
                           0,
                           d_virial,
                           d_molecule_len,
                           d_molecule_list,
                           d_molecule_idx,
                           d_rigid_center,
                           molecule_indexer,
                           d_postype,
                           d_orientation,
                           body_indexer,
                           d_body_pos,
                           d_body_orientation,
                           d_net_force,
                           d_net_virial,
                           d_body,
                           d_tag,
                           n_mol,
                           N,
                           net_virial_pitch,
                           virial_pitch,
                           window_size,
                           thread_mask,
                           n_bodies_per_block,
                           range.first,
                           nwork);
        }

    return hipSuccess;
    }

__global__ void gpu_update_composite_kernel(unsigned int N,
                                            unsigned int nwork,
                                            unsigned int offset,
                                            unsigned int n_ghost,
                                            const unsigned int* d_lookup_center,
                                            Scalar4* d_postype,
                                            Scalar4* d_orientation,
                                            Index2D body_indexer,
                                            const Scalar3* d_body_pos,
                                            const Scalar4* d_body_orientation,
                                            const unsigned int* d_body_len,
                                            const unsigned int* d_molecule_order,
                                            const unsigned int* d_molecule_len,
                                            const unsigned int* d_molecule_idx,
                                            int3* d_image,
                                            const BoxDim box,
                                            const BoxDim global_box,
                                            uint2* d_flag)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= nwork)
        return;

    idx += offset;

    unsigned int central_idx = d_lookup_center[idx];
    if (central_idx == NO_BODY)
        return;

    if (central_idx >= N + n_ghost)
        {
        // if a molecule with a local member has no central particle, error out
        if (idx < N)
            {
            atomicMax(&(d_flag->x), idx + 1);
            }

        // otherwise, ignore
        return;
        }

    // do not overwrite central ptl
    if (idx == central_idx)
        return;

    Scalar4 postype = d_postype[central_idx];
    vec3<Scalar> pos(postype);
    quat<Scalar> orientation(d_orientation[central_idx]);

    unsigned int body_type = __scalar_as_int(postype.w);

    unsigned int body_len = d_body_len[body_type];
    unsigned int mol_idx = d_molecule_idx[idx];

    if (body_len != d_molecule_len[mol_idx] - 1)
        {
        // if a molecule with a local member is incomplete, this is an error
        if (idx < N)
            {
            atomicMax(&(d_flag->y), idx + 1);
            }

        // otherwise, ignore
        return;
        }

    int3 img = d_image[central_idx];

    unsigned int idx_in_body = d_molecule_order[idx] - 1;

    vec3<Scalar> local_pos(d_body_pos[body_indexer(body_type, idx_in_body)]);
    vec3<Scalar> dr_space = rotate(orientation, local_pos);

    vec3<Scalar> updated_pos(pos);
    updated_pos += dr_space;

    quat<Scalar> local_orientation(d_body_orientation[body_indexer(body_type, idx_in_body)]);
    quat<Scalar> updated_orientation = orientation * local_orientation;

    // this runs before the ForceComputes,
    // wrap into box, allowing rigid bodies to span multiple images
    int3 imgi = box.getImage(vec_to_scalar3(updated_pos));
    int3 negimgi = make_int3(-imgi.x, -imgi.y, -imgi.z);
    updated_pos = global_box.shift(updated_pos, negimgi);

    unsigned int type = __scalar_as_int(d_postype[idx].w);

    d_postype[idx]
        = make_scalar4(updated_pos.x, updated_pos.y, updated_pos.z, __int_as_scalar(type));
    d_orientation[idx] = quat_to_scalar4(updated_orientation);
    d_image[idx] = img + imgi;
    }

void gpu_update_composite(unsigned int N,
                          unsigned int n_ghost,
                          Scalar4* d_postype,
                          Scalar4* d_orientation,
                          Index2D body_indexer,
                          const unsigned int* d_lookup_center,
                          const Scalar3* d_body_pos,
                          const Scalar4* d_body_orientation,
                          const unsigned int* d_body_len,
                          const unsigned int* d_molecule_order,
                          const unsigned int* d_molecule_len,
                          const unsigned int* d_molecule_idx,
                          int3* d_image,
                          const BoxDim box,
                          const BoxDim global_box,
                          unsigned int block_size,
                          uint2* d_flag,
                          const GPUPartition& gpu_partition)
    {
    unsigned int run_block_size = block_size;

    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_update_composite_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    if (max_block_size <= run_block_size)
        {
        run_block_size = max_block_size;
        }

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // process ghosts in final range
        if (idev == (int)gpu_partition.getNumActiveGPUs() - 1)
            nwork += n_ghost;

        unsigned int n_blocks = nwork / run_block_size + 1;
        hipLaunchKernelGGL((gpu_update_composite_kernel),
                           dim3(n_blocks),
                           dim3(run_block_size),
                           0,
                           0,
                           N,
                           nwork,
                           range.first,
                           n_ghost,
                           d_lookup_center,
                           d_postype,
                           d_orientation,
                           body_indexer,
                           d_body_pos,
                           d_body_orientation,
                           d_body_len,
                           d_molecule_order,
                           d_molecule_len,
                           d_molecule_idx,
                           d_image,
                           box,
                           global_box,
                           d_flag);
        }
    }

struct is_center
    {
    __host__ __device__ bool operator()(const thrust::tuple<unsigned int, unsigned int>& t)
        {
        return t.get<0>() == t.get<1>();
        }
    };

// create a lookup table ptl idx -> center idx
struct lookup_op : thrust::unary_function<unsigned int, unsigned int>
    {
    __host__ __device__ lookup_op(const unsigned int* _d_rtag) : d_rtag(_d_rtag) { }

    __device__ unsigned int operator()(const unsigned int& body)
        {
        return (body < MIN_FLOPPY) ? d_rtag[body] : NO_BODY;
        }

    const unsigned int* d_rtag;
    };

hipError_t gpu_find_rigid_centers(const unsigned int* d_body,
                                  const unsigned int* d_tag,
                                  const unsigned int* d_rtag,
                                  const unsigned int N,
                                  const unsigned int nghost,
                                  unsigned int* d_rigid_center,
                                  unsigned int* d_lookup_center,
                                  unsigned int& n_rigid)
    {
    thrust::device_ptr<const unsigned int> body(d_body);
    thrust::device_ptr<const unsigned int> tag(d_tag);
    thrust::device_ptr<unsigned int> rigid_center(d_rigid_center);
    thrust::counting_iterator<unsigned int> count(0);

    // create a contiguos list of rigid center indicies
    auto it = thrust::copy_if(count,
                              count + N + nghost,
                              thrust::make_zip_iterator(thrust::make_tuple(body, tag)),
                              rigid_center,
                              is_center());

    n_rigid = (unsigned int)(it - rigid_center);

    thrust::device_ptr<unsigned int> lookup_center(d_lookup_center);
    thrust::transform(body, body + N + nghost, lookup_center, lookup_op(d_rtag));

    return hipSuccess;
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
