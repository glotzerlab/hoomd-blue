// Copyright (c) 2009-2024 The Regents of the University of Michigan.
// Part of HOOMD-blue, released under the BSD 3-Clause License.

/*!
 * \file mpcd/ConfinedStreamingMethodGPU.cu
 * \brief Defines GPU functions and kernels used by mpcd::ConfinedStreamingMethodGPU
 *
 * \warning
 * This file needs separable compilation with ExternalFields.cu. Any plugins extending
 * the ConfinedStreamingGeometryGPU will also need to do separable compilation with
 * ExternalFields.cu.
 */

#include "ConfinedStreamingMethodGPU.cuh"
#include "StreamingGeometry.h"

#include "ExternalField.h"
#include "hoomd/GPUPolymorph.cuh"

namespace hoomd
    {
namespace mpcd
    {
namespace gpu
    {
//! Template instantiation of bulk geometry streaming
template hipError_t __attribute__((visibility("default")))
confined_stream<mpcd::detail::BulkGeometry>(const stream_args_t& args,
                                            const mpcd::detail::BulkGeometry& geom);

//! Template instantiation of slit geometry streaming
template hipError_t __attribute__((visibility("default")))
confined_stream<mpcd::detail::SlitGeometry>(const stream_args_t& args,
                                            const mpcd::detail::SlitGeometry& geom);

//! Template instantiation of slit geometry streaming
template hipError_t
confined_stream<mpcd::detail::SlitPoreGeometry>(const stream_args_t& args,
                                                const mpcd::detail::SlitPoreGeometry& geom);

    } // end namespace gpu
    } // end namespace mpcd
    } // end namespace hoomd
