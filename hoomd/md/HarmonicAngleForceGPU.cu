#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: dnlebard

#include "HarmonicAngleForceGPU.cuh"
#include "hoomd/TextureTools.h"

#include <assert.h>

// SMALL a relatively small number
#define SMALL Scalar(0.001)

/*! \file HarmonicAngleForceGPU.cu
    \brief Defines GPU kernel code for calculating the harmonic angle forces. Used by
   HarmonicAngleForceComputeGPU.
*/

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
//! Kernel for calculating harmonic angle forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch Pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param d_params Parameters for the angle force
    \param box Box dimensions for periodic boundary condition handling
    \param alist Angle data to use in calculating the forces
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
*/
__global__ void gpu_compute_harmonic_angle_forces_kernel(Scalar4* d_force,
                                                         Scalar* d_virial,
                                                         const size_t virial_pitch,
                                                         const unsigned int N,
                                                         const Scalar4* d_pos,
                                                         const Scalar2* d_params,
                                                         BoxDim box,
                                                         const group_storage<3>* alist,
                                                         const unsigned int* apos_list,
                                                         const unsigned int pitch,
                                                         const unsigned int* n_angles_list)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_angles = n_angles_list[idx];

    // read in the position of our b-particle from the a-b-c triplet. (MEM TRANSFER: 16 bytes)
    Scalar4 idx_postype = d_pos[idx]; // we can be either a, b, or c in the a-b-c triplet
    Scalar3 idx_pos = make_scalar3(idx_postype.x, idx_postype.y, idx_postype.z);
    Scalar3 a_pos, b_pos, c_pos; // allocate space for the a,b, and c atom in the a-b-c triplet

    // initialize the force to 0
    Scalar4 force_idx = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));

    Scalar fab[3], fcb[3];

    // initialize the virial to 0
    Scalar virial[6];
    for (int i = 0; i < 6; i++)
        virial[i] = Scalar(0.0);

    // loop over all angles
    for (int angle_idx = 0; angle_idx < n_angles; angle_idx++)
        {
        group_storage<3> cur_angle = alist[pitch * angle_idx + idx];

        int cur_angle_x_idx = cur_angle.idx[0];
        int cur_angle_y_idx = cur_angle.idx[1];
        int cur_angle_type = cur_angle.idx[2];

        int cur_angle_abc = apos_list[pitch * angle_idx + idx];

        // get the a-particle's position (MEM TRANSFER: 16 bytes)
        Scalar4 x_postype = d_pos[cur_angle_x_idx];
        Scalar3 x_pos = make_scalar3(x_postype.x, x_postype.y, x_postype.z);
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        Scalar4 y_postype = d_pos[cur_angle_y_idx];
        Scalar3 y_pos = make_scalar3(y_postype.x, y_postype.y, y_postype.z);

        if (cur_angle_abc == 0)
            {
            a_pos = idx_pos;
            b_pos = x_pos;
            c_pos = y_pos;
            }
        if (cur_angle_abc == 1)
            {
            b_pos = idx_pos;
            a_pos = x_pos;
            c_pos = y_pos;
            }
        if (cur_angle_abc == 2)
            {
            c_pos = idx_pos;
            a_pos = x_pos;
            b_pos = y_pos;
            }

        // calculate dr for a-b,c-b,and a-c
        Scalar3 dab = a_pos - b_pos;
        Scalar3 dcb = c_pos - b_pos;
        Scalar3 dac = a_pos - c_pos;

        // apply periodic boundary conditions
        dab = box.minImage(dab);
        dcb = box.minImage(dcb);
        dac = box.minImage(dac);

        // get the angle parameters (MEM TRANSFER: 8 bytes)
        Scalar2 params = __ldg(d_params + cur_angle_type);
        Scalar K = params.x;
        Scalar t_0 = params.y;

        Scalar rsqab = dot(dab, dab);
        Scalar rab = sqrtf(rsqab);
        Scalar rsqcb = dot(dcb, dcb);
        Scalar rcb = sqrtf(rsqcb);

        Scalar c_abbc = dot(dab, dcb);
        c_abbc /= rab * rcb;

        if (c_abbc > Scalar(1.0))
            c_abbc = Scalar(1.0);
        if (c_abbc < -Scalar(1.0))
            c_abbc = -Scalar(1.0);

        Scalar s_abbc = sqrtf(Scalar(1.0) - c_abbc * c_abbc);
        if (s_abbc < SMALL)
            s_abbc = SMALL;
        s_abbc = Scalar(1.0) / s_abbc;

        // actually calculate the force
        Scalar dth = fast::acos(c_abbc) - t_0;
        Scalar tk = K * dth;

        Scalar a = -Scalar(1.0) * tk * s_abbc;
        Scalar a11 = a * c_abbc / rsqab;
        Scalar a12 = -a / (rab * rcb);
        Scalar a22 = a * c_abbc / rsqcb;

        fab[0] = a11 * dab.x + a12 * dcb.x;
        fab[1] = a11 * dab.y + a12 * dcb.y;
        fab[2] = a11 * dab.z + a12 * dcb.z;

        fcb[0] = a22 * dcb.x + a12 * dab.x;
        fcb[1] = a22 * dcb.y + a12 * dab.y;
        fcb[2] = a22 * dcb.z + a12 * dab.z;

        // compute 1/3 of the energy, 1/3 for each atom in the angle
        Scalar angle_eng = tk * dth * Scalar(Scalar(1.0) / Scalar(6.0));

        // upper triangular version of virial tensor
        Scalar angle_virial[6];
        angle_virial[0] = Scalar(1. / 3.) * (dab.x * fab[0] + dcb.x * fcb[0]);
        angle_virial[1] = Scalar(1. / 3.) * (dab.y * fab[0] + dcb.y * fcb[0]);
        angle_virial[2] = Scalar(1. / 3.) * (dab.z * fab[0] + dcb.z * fcb[0]);
        angle_virial[3] = Scalar(1. / 3.) * (dab.y * fab[1] + dcb.y * fcb[1]);
        angle_virial[4] = Scalar(1. / 3.) * (dab.z * fab[1] + dcb.z * fcb[1]);
        angle_virial[5] = Scalar(1. / 3.) * (dab.z * fab[2] + dcb.z * fcb[2]);

        if (cur_angle_abc == 0)
            {
            force_idx.x += fab[0];
            force_idx.y += fab[1];
            force_idx.z += fab[2];
            }
        if (cur_angle_abc == 1)
            {
            force_idx.x -= fab[0] + fcb[0];
            force_idx.y -= fab[1] + fcb[1];
            force_idx.z -= fab[2] + fcb[2];
            }
        if (cur_angle_abc == 2)
            {
            force_idx.x += fcb[0];
            force_idx.y += fcb[1];
            force_idx.z += fcb[2];
            }

        force_idx.w += angle_eng;

        for (int i = 0; i < 6; i++)
            virial[i] += angle_virial[i];
        }

    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    d_force[idx] = force_idx;
    for (int i = 0; i < 6; i++)
        d_virial[i * virial_pitch + idx] = virial[i];
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param atable List of angles stored on the GPU
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
    \param d_params K and t_0 params packed as Scalar2 variables
    \param n_angle_types Number of angle types in d_params
    \param block_size Block size to use when performing calculations
    \param compute_capability Device compute capability (200, 300, 350, ...)

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one Scalar2 element per angle type. The x component contains K the
   spring constant and the y component contains t_0 the equilibrium angle.
*/
hipError_t gpu_compute_harmonic_angle_forces(Scalar4* d_force,
                                             Scalar* d_virial,
                                             const size_t virial_pitch,
                                             const unsigned int N,
                                             const Scalar4* d_pos,
                                             const BoxDim& box,
                                             const group_storage<3>* atable,
                                             const unsigned int* apos_list,
                                             const unsigned int pitch,
                                             const unsigned int* n_angles_list,
                                             Scalar2* d_params,
                                             unsigned int n_angle_types,
                                             int block_size)
    {
    assert(d_params);

    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_harmonic_angle_forces_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    // setup the grid to run the kernel
    dim3 grid(N / run_block_size + 1, 1, 1);
    dim3 threads(run_block_size, 1, 1);

    // run the kernel
    hipLaunchKernelGGL((gpu_compute_harmonic_angle_forces_kernel),
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       d_force,
                       d_virial,
                       virial_pitch,
                       N,
                       d_pos,
                       d_params,
                       box,
                       atable,
                       apos_list,
                       pitch,
                       n_angles_list);

    return hipSuccess;
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
