#include "hip/hip_runtime.h"
// Copyright (c) 2009-2024 The Regents of the University of Michigan.
// Part of HOOMD-blue, released under the BSD 3-Clause License.

#include "PPPMForceComputeGPU.cuh"
#include "hoomd/TextureTools.h"

// __scalar2int_rd is __float2int_rd in single, __double2int_rd in double
#if HOOMD_LONGREAL_SIZE == 32
#define __scalar2int_rd __float2int_rd
#else
#define __scalar2int_rd __double2int_rd
#endif

#define GPU_PPPM_MAX_ORDER 7

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
// workaround for HIP bug
#ifdef __HIP_PLATFORM_HCC__
inline __device__ float myAtomicAdd(float* address, float val)
    {
    unsigned int* address_as_uint = (unsigned int*)address;
    unsigned int old = *address_as_uint, assumed;

    do
        {
        assumed = old;
        old = atomicCAS(address_as_uint, assumed, __float_as_uint(val + __uint_as_float(assumed)));
        } while (assumed != old);

    return __uint_as_float(old);
    }
#else
inline __device__ float myAtomicAdd(float* address, float val)
    {
    return atomicAdd(address, val);
    }
#endif

//! GPU implementation of sinc(x)==sin(x)/x
__device__ Scalar gpu_sinc(Scalar x)
    {
    Scalar sinc = 0;

    //! Coefficients of a power expansion of sin(x)/x
    const Scalar sinc_coeff[] = {Scalar(1.0),
                                 Scalar(-1.0 / 6.0),
                                 Scalar(1.0 / 120.0),
                                 Scalar(-1.0 / 5040.0),
                                 Scalar(1.0 / 362880.0),
                                 Scalar(-1.0 / 39916800.0)};

    if (x * x <= Scalar(1.0))
        {
        Scalar term = Scalar(1.0);
        for (unsigned int i = 0; i < 6; ++i)
            {
            sinc += sinc_coeff[i] * term;
            term *= x * x;
            }
        }
    else
        {
        sinc = fast::sin(x) / x;
        }

    return sinc;
    }

__device__ int3 find_cell(const Scalar3& pos,
                          const unsigned int& inner_nx,
                          const unsigned int& inner_ny,
                          const unsigned int& inner_nz,
                          const uint3& n_ghost_cells,
                          const BoxDim& box,
                          int order,
                          Scalar3& dr)
    {
    // compute coordinates in units of the mesh size
    Scalar3 f = box.makeFraction(pos);
    uchar3 periodic = box.getPeriodic();

    Scalar3 reduced_pos
        = make_scalar3(f.x * (Scalar)inner_nx, f.y * (Scalar)inner_ny, f.z * (Scalar)inner_nz);

    reduced_pos += make_scalar3(n_ghost_cells.x, n_ghost_cells.y, n_ghost_cells.z);

    Scalar shift, shiftone;
    if (order % 2)
        {
        shift = Scalar(0.5);
        shiftone = Scalar(0.0);
        }
    else
        {
        shift = Scalar(0.0);
        shiftone = Scalar(0.5);
        }

    int ix = __scalar2int_rd(reduced_pos.x + shift);
    int iy = __scalar2int_rd(reduced_pos.y + shift);
    int iz = __scalar2int_rd(reduced_pos.z + shift);

    // set distance to cell center
    dr.x = shiftone + (Scalar)ix - reduced_pos.x;
    dr.y = shiftone + (Scalar)iy - reduced_pos.y;
    dr.z = shiftone + (Scalar)iz - reduced_pos.z;

    // handle particles on the boundary
    if (periodic.x && ix == (int)inner_nx)
        ix = 0;
    if (periodic.y && iy == (int)inner_ny)
        iy = 0;
    if (periodic.z && iz == (int)inner_nz)
        iz = 0;

    return make_int3(ix, iy, iz);
    }

__global__ void gpu_assign_particles_kernel(const uint3 mesh_dim,
                                            const uint3 n_ghost_bins,
                                            unsigned int work_size,
                                            const unsigned int* d_index_array,
                                            const Scalar4* d_postype,
                                            const Scalar* d_charge,
                                            hipfftComplex* d_mesh,
                                            Scalar V_cell,
                                            int order,
                                            unsigned int offset,
                                            BoxDim box,
                                            const Scalar* d_rho_coeff)
    {
    extern __shared__ Scalar s_coeff[];

    // load in interpolation coefficients
    unsigned int ncoeffs = order * (2 * order + 1);
    for (unsigned int cur_offset = 0; cur_offset < ncoeffs; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < ncoeffs)
            {
            s_coeff[cur_offset + threadIdx.x] = d_rho_coeff[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();

    unsigned int work_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (work_idx >= work_size)
        return;

    unsigned int group_idx = work_idx + offset;

    int3 bin_dim = make_int3(mesh_dim.x + 2 * n_ghost_bins.x,
                             mesh_dim.y + 2 * n_ghost_bins.y,
                             mesh_dim.z + 2 * n_ghost_bins.z);

    // grid coordinates of bin (column-major)
    unsigned int idx = d_index_array[group_idx];

    Scalar4 postype = d_postype[idx];

    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    Scalar qi = d_charge[idx];

    // compute coordinates in units of the cell size
    Scalar3 dr = make_scalar3(0, 0, 0);
    int3 bin_coord
        = find_cell(pos, mesh_dim.x, mesh_dim.y, mesh_dim.z, n_ghost_bins, box, order, dr);

    // ignore particles that are not within our domain (the error should be caught by HOOMD's cell
    // list)
    if (bin_coord.x < 0 || bin_coord.x >= bin_dim.x || bin_coord.y < 0 || bin_coord.y >= bin_dim.y
        || bin_coord.z < 0 || bin_coord.z >= bin_dim.z)
        {
        return;
        }

    int i = bin_coord.x;
    int j = bin_coord.y;
    int k = bin_coord.z;

    int nlower = -(order - 1) / 2;
    int nupper = order / 2;

    Scalar result;

    int mult_fact = 2 * order + 1;

    Scalar x0 = qi;

    bool ignore_x = false;
    bool ignore_y = false;
    bool ignore_z = false;

    // loop over neighboring bins
    for (int l = nlower; l <= nupper; ++l)
        {
        // precalculate assignment factor
        result = Scalar(0.0);
        for (int iorder = order - 1; iorder >= 0; iorder--)
            {
            result = s_coeff[l - nlower + iorder * mult_fact] + result * dr.x;
            }
        Scalar y0 = x0 * result;

        int neighi = i + l;
        if (neighi >= (int)bin_dim.x)
            {
            if (!n_ghost_bins.x)
                neighi -= (int)bin_dim.x;
            else
                ignore_x = true;
            }
        else if (neighi < 0)
            {
            if (!n_ghost_bins.x)
                neighi += (int)bin_dim.x;
            else
                ignore_x = true;
            }

        for (int m = nlower; m <= nupper; ++m)
            {
            result = Scalar(0.0);
            for (int iorder = order - 1; iorder >= 0; iorder--)
                {
                result = s_coeff[m - nlower + iorder * mult_fact] + result * dr.y;
                }
            Scalar z0 = y0 * result;

            int neighj = j + m;
            if (neighj >= (int)bin_dim.y)
                {
                if (!n_ghost_bins.y)
                    neighj -= (int)bin_dim.y;
                else
                    ignore_y = true;
                }
            else if (neighj < 0)
                {
                if (!n_ghost_bins.y)
                    neighj += (int)bin_dim.y;
                else
                    ignore_y = true;
                }

            for (int n = nlower; n <= nupper; ++n)
                {
                result = Scalar(0.0);
                for (int iorder = order - 1; iorder >= 0; iorder--)
                    {
                    result = s_coeff[n - nlower + iorder * mult_fact] + result * dr.z;
                    }

                int neighk = k + n;

                if (neighk >= (int)bin_dim.z)
                    {
                    if (!n_ghost_bins.z)
                        neighk -= (int)bin_dim.z;
                    else
                        ignore_z = true;
                    }
                else if (neighk < 0)
                    {
                    if (!n_ghost_bins.z)
                        neighk += (int)bin_dim.z;
                    else
                        ignore_z = true;
                    }

                if (!ignore_x && !ignore_y && !ignore_z)
                    {
                    // write out to global memory using row-major
                    unsigned int cell_idx = neighi + bin_dim.x * (neighj + bin_dim.y * neighk);

                    // compute fraction of particle density assigned to cell
                    // from particles in this bin
                    myAtomicAdd(&d_mesh[cell_idx].x, z0 * result / V_cell);
                    }

                ignore_z = false;
                }
            ignore_y = false;
            }
        ignore_x = false;
        } // end of loop over neighboring bins
    }

__global__ void gpu_reduce_meshes(const unsigned int mesh_elements,
                                  const hipfftComplex* d_mesh_scratch,
                                  hipfftComplex* d_mesh,
                                  unsigned int ngpu)
    {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= mesh_elements)
        return;

    hipfftComplex res;
    res.x = 0;
    res.y = 0;

    // reduce over all temporary meshes
    for (unsigned int igpu = 0; igpu < ngpu; ++igpu)
        {
        hipfftComplex m = d_mesh_scratch[idx + igpu * mesh_elements];
        res.x += m.x;
        res.y += m.y;
        }
    d_mesh[idx] = res;
    }

void gpu_assign_particles(const uint3 mesh_dim,
                          const uint3 n_ghost_bins,
                          const uint3 grid_dim,
                          unsigned int group_size,
                          const unsigned int* d_index_array,
                          const Scalar4* d_postype,
                          const Scalar* d_charge,
                          hipfftComplex* d_mesh,
                          hipfftComplex* d_mesh_scratch,
                          const unsigned int mesh_elements,
                          int order,
                          const BoxDim& box,
                          unsigned int block_size,
                          const Scalar* d_rho_coeff,
                          const hipDeviceProp_t& dev_prop,
                          const GPUPartition& gpu_partition)
    {
    hipMemsetAsync(d_mesh, 0, sizeof(hipfftComplex) * grid_dim.x * grid_dim.y * grid_dim.z);
    Scalar V_cell = box.getVolume() / (Scalar)(mesh_dim.x * mesh_dim.y * mesh_dim.z);

    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_assign_particles_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(max_block_size, block_size);

    while (attr.sharedSizeBytes >= dev_prop.sharedMemPerBlock)
        {
        run_block_size -= dev_prop.warpSize;
        }

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    unsigned int ngpu = gpu_partition.getNumActiveGPUs();
    for (int idev = ngpu - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        if (ngpu > 1)
            {
            // zero the temporary mesh array
            hipMemsetAsync(d_mesh_scratch + idev * mesh_elements,
                           0,
                           sizeof(hipfftComplex) * mesh_elements);
            }

        unsigned int nwork = range.second - range.first;
        unsigned int n_blocks = nwork / run_block_size + 1;
        const size_t shared_bytes = order * (2 * order + 1) * sizeof(Scalar);

        hipLaunchKernelGGL((gpu_assign_particles_kernel),
                           dim3(n_blocks),
                           dim3(run_block_size),
                           shared_bytes,
                           0,
                           mesh_dim,
                           n_ghost_bins,
                           nwork,
                           d_index_array,
                           d_postype,
                           d_charge,
                           ngpu > 1 ? d_mesh_scratch + idev * mesh_elements : d_mesh,
                           V_cell,
                           order,
                           range.first,
                           box,
                           d_rho_coeff);
        }
    }

//! Reduce temporary arrays for every GPU
void gpu_reduce_meshes(const unsigned int mesh_elements,
                       const hipfftComplex* d_mesh_scratch,
                       hipfftComplex* d_mesh,
                       const unsigned int ngpu,
                       const unsigned int block_size)
    {
    // reduce meshes on GPU 0
    hipLaunchKernelGGL((gpu_reduce_meshes),
                       dim3(mesh_elements / block_size + 1),
                       dim3(block_size),
                       0,
                       0,
                       mesh_elements,
                       d_mesh_scratch,
                       d_mesh,
                       ngpu);
    }

__global__ void gpu_compute_mesh_virial_kernel(const unsigned int n_wave_vectors,
                                               hipfftComplex* d_fourier_mesh,
                                               Scalar* d_inf_f,
                                               Scalar* d_virial_mesh,
                                               const Scalar3* d_k,
                                               const bool exclude_dc,
                                               Scalar kappa)
    {
    unsigned int idx;

    idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= n_wave_vectors)
        return;

    if (!exclude_dc || idx != 0)
        {
        // non-zero wave vector
        hipfftComplex fourier = d_fourier_mesh[idx];

        Scalar3 k = d_k[idx];

        Scalar rhog = (fourier.x * fourier.x + fourier.y * fourier.y) * d_inf_f[idx];
        Scalar vterm = -Scalar(2.0) * (Scalar(1.0) / dot(k, k) + Scalar(0.25) / (kappa * kappa));

        d_virial_mesh[0 * n_wave_vectors + idx] = rhog * (Scalar(1.0) + vterm * k.x * k.x); // xx
        d_virial_mesh[1 * n_wave_vectors + idx] = rhog * (vterm * k.x * k.y);               // xy
        d_virial_mesh[2 * n_wave_vectors + idx] = rhog * (vterm * k.x * k.z);               // xz
        d_virial_mesh[3 * n_wave_vectors + idx] = rhog * (Scalar(1.0) + vterm * k.y * k.y); // yy
        d_virial_mesh[4 * n_wave_vectors + idx] = rhog * (vterm * k.y * k.z);               // yz
        d_virial_mesh[5 * n_wave_vectors + idx] = rhog * (Scalar(1.0) + vterm * k.z * k.z); // zz
        }
    else
        {
        d_virial_mesh[0 * n_wave_vectors + idx] = Scalar(0.0);
        d_virial_mesh[1 * n_wave_vectors + idx] = Scalar(0.0);
        d_virial_mesh[2 * n_wave_vectors + idx] = Scalar(0.0);
        d_virial_mesh[3 * n_wave_vectors + idx] = Scalar(0.0);
        d_virial_mesh[4 * n_wave_vectors + idx] = Scalar(0.0);
        d_virial_mesh[5 * n_wave_vectors + idx] = Scalar(0.0);
        }
    }

void gpu_compute_mesh_virial(const unsigned int n_wave_vectors,
                             hipfftComplex* d_fourier_mesh,
                             Scalar* d_inf_f,
                             Scalar* d_virial_mesh,
                             const Scalar3* d_k,
                             const bool exclude_dc,
                             Scalar kappa)

    {
    const unsigned int block_size = 256;

    dim3 grid(n_wave_vectors / block_size + 1, 1, 1);

    hipLaunchKernelGGL((gpu_compute_mesh_virial_kernel),
                       dim3(grid),
                       dim3(block_size),
                       0,
                       0,
                       n_wave_vectors,
                       d_fourier_mesh,
                       d_inf_f,
                       d_virial_mesh,
                       d_k,
                       exclude_dc,
                       kappa);
    }

__global__ void gpu_update_meshes_kernel(const unsigned int n_wave_vectors,
                                         hipfftComplex* d_fourier_mesh,
                                         hipfftComplex* d_fourier_mesh_G_x,
                                         hipfftComplex* d_fourier_mesh_G_y,
                                         hipfftComplex* d_fourier_mesh_G_z,
                                         const Scalar* d_inf_f,
                                         const Scalar3* d_k,
                                         unsigned int NNN)
    {
    unsigned int k;

    k = blockDim.x * blockIdx.x + threadIdx.x;

    if (k >= n_wave_vectors)
        return;

    hipfftComplex f = d_fourier_mesh[k];

    Scalar scaled_inf_f = d_inf_f[k] / ((Scalar)NNN);

    Scalar3 kvec = d_k[k];

    // Normalization
    hipfftComplex fourier_G_x;
    fourier_G_x.x = f.y * kvec.x * scaled_inf_f;
    fourier_G_x.y = -f.x * kvec.x * scaled_inf_f;

    hipfftComplex fourier_G_y;
    fourier_G_y.x = f.y * kvec.y * scaled_inf_f;
    fourier_G_y.y = -f.x * kvec.y * scaled_inf_f;

    hipfftComplex fourier_G_z;
    fourier_G_z.x = f.y * kvec.z * scaled_inf_f;
    fourier_G_z.y = -f.x * kvec.z * scaled_inf_f;

    // store in global memory
    d_fourier_mesh_G_x[k] = fourier_G_x;
    d_fourier_mesh_G_y[k] = fourier_G_y;
    d_fourier_mesh_G_z[k] = fourier_G_z;
    }

void gpu_update_meshes(const unsigned int n_wave_vectors,
                       hipfftComplex* d_fourier_mesh,
                       hipfftComplex* d_fourier_mesh_G_x,
                       hipfftComplex* d_fourier_mesh_G_y,
                       hipfftComplex* d_fourier_mesh_G_z,
                       const Scalar* d_inf_f,
                       const Scalar3* d_k,
                       unsigned int NNN,
                       unsigned int block_size)

    {
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_update_meshes_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(max_block_size, block_size);
    dim3 grid(n_wave_vectors / run_block_size + 1, 1, 1);

    hipLaunchKernelGGL((gpu_update_meshes_kernel),
                       dim3(grid),
                       dim3(run_block_size),
                       0,
                       0,
                       n_wave_vectors,
                       d_fourier_mesh,
                       d_fourier_mesh_G_x,
                       d_fourier_mesh_G_y,
                       d_fourier_mesh_G_z,
                       d_inf_f,
                       d_k,
                       NNN);
    }

__global__ void gpu_compute_forces_kernel(const unsigned int work_size,
                                          const Scalar4* d_postype,
                                          Scalar4* d_force,
                                          const uint3 grid_dim,
                                          const uint3 n_ghost_cells,
                                          const Scalar* d_charge,
                                          const BoxDim box,
                                          int order,
                                          const unsigned int* d_index_array,
                                          const hipfftComplex* inv_fourier_mesh_x,
                                          const hipfftComplex* inv_fourier_mesh_y,
                                          const hipfftComplex* inv_fourier_mesh_z,
                                          const Scalar* d_rho_coeff,
                                          const unsigned int offset)
    {
    extern __shared__ Scalar s_coeff[];

    // load in interpolation coefficients
    unsigned int ncoeffs = order * (2 * order + 1);
    for (unsigned int cur_offset = 0; cur_offset < ncoeffs; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < ncoeffs)
            {
            s_coeff[cur_offset + threadIdx.x] = d_rho_coeff[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();

    unsigned int work_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (work_idx >= work_size)
        return;

    unsigned int group_idx = work_idx + offset;

    unsigned int idx = d_index_array[group_idx];

    int3 inner_dim = make_int3(grid_dim.x - 2 * n_ghost_cells.x,
                               grid_dim.y - 2 * n_ghost_cells.y,
                               grid_dim.z - 2 * n_ghost_cells.z);

    Scalar4 postype = d_postype[idx];

    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int type = __scalar_as_int(postype.w);
    Scalar qi = d_charge[idx];

    Scalar3 dr = make_scalar3(0, 0, 0);

    // find cell the particle is in
    int3 cell_coord
        = find_cell(pos, inner_dim.x, inner_dim.y, inner_dim.z, n_ghost_cells, box, order, dr);

    // ignore particles that are not within our domain (the error should be caught by HOOMD's cell
    // list)
    if (cell_coord.x < 0 || cell_coord.x >= (int)grid_dim.x || cell_coord.y < 0
        || cell_coord.y >= (int)grid_dim.y || cell_coord.z < 0 || cell_coord.z >= (int)grid_dim.z)
        {
        return;
        }

    Scalar3 force = make_scalar3(0.0, 0.0, 0.0);

    int nlower = -(order - 1) / 2;
    int nupper = order / 2;

    Scalar result;
    int mult_fact = 2 * order + 1;

    // back-interpolate forces from neighboring mesh points
    for (int l = nlower; l <= nupper; ++l)
        {
        result = Scalar(0.0);
        for (int k = order - 1; k >= 0; k--)
            {
            result = s_coeff[l - nlower + k * mult_fact] + result * dr.x;
            }
        Scalar x0 = result;

        for (int m = nlower; m <= nupper; ++m)
            {
            result = Scalar(0.0);
            for (int k = order - 1; k >= 0; k--)
                {
                result = s_coeff[m - nlower + k * mult_fact] + result * dr.y;
                }
            Scalar y0 = x0 * result;

            for (int n = nlower; n <= nupper; ++n)
                {
                result = Scalar(0.0);
                for (int k = order - 1; k >= 0; k--)
                    {
                    result = s_coeff[n - nlower + k * mult_fact] + result * dr.z;
                    }
                Scalar z0 = y0 * result;

                int neighl = (int)cell_coord.x + l;
                int neighm = (int)cell_coord.y + m;
                int neighn = (int)cell_coord.z + n;

                if (!n_ghost_cells.x)
                    {
                    if (neighl >= (int)grid_dim.x)
                        neighl -= grid_dim.x;
                    else if (neighl < 0)
                        neighl += grid_dim.x;
                    }

                if (!n_ghost_cells.y)
                    {
                    if (neighm >= (int)grid_dim.y)
                        neighm -= grid_dim.y;
                    else if (neighm < 0)
                        neighm += grid_dim.y;
                    }

                if (!n_ghost_cells.z)
                    {
                    if (neighn >= (int)grid_dim.z)
                        neighn -= grid_dim.z;
                    else if (neighn < 0)
                        neighn += grid_dim.z;
                    }

                // use column-major layout
                unsigned int cell_idx = neighl + grid_dim.x * (neighm + grid_dim.y * neighn);

                hipfftComplex inv_mesh_x = inv_fourier_mesh_x[cell_idx];
                hipfftComplex inv_mesh_y = inv_fourier_mesh_y[cell_idx];
                hipfftComplex inv_mesh_z = inv_fourier_mesh_z[cell_idx];

                force.x += qi * z0 * inv_mesh_x.x;
                force.y += qi * z0 * inv_mesh_y.x;
                force.z += qi * z0 * inv_mesh_z.x;
                }
            }
        } // end neighbor cells loop

    d_force[idx] = make_scalar4(force.x, force.y, force.z, 0.0);
    }

void gpu_compute_forces(const unsigned int N,
                        const Scalar4* d_postype,
                        Scalar4* d_force,
                        const hipfftComplex* d_inv_fourier_mesh_x,
                        const hipfftComplex* d_inv_fourier_mesh_y,
                        const hipfftComplex* d_inv_fourier_mesh_z,
                        const uint3 grid_dim,
                        const uint3 n_ghost_cells,
                        const Scalar* d_charge,
                        const BoxDim& box,
                        int order,
                        const unsigned int* d_index_array,
                        const GPUPartition& gpu_partition,
                        const GPUPartition& all_gpu_partition,
                        const Scalar* d_rho_coeff,
                        unsigned int block_size,
                        bool local_fft,
                        unsigned int inv_mesh_elements)
    {
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_forces_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(max_block_size, block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = all_gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = all_gpu_partition.getRangeAndSetGPU(idev);

        // reset force array for ALL particles
        hipMemsetAsync(d_force + range.first, 0, sizeof(Scalar4) * (range.second - range.first));
        }

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;
        unsigned int n_blocks = nwork / run_block_size + 1;
        const size_t shared_bytes = order * (2 * order + 1) * sizeof(Scalar);

        hipLaunchKernelGGL(
            (gpu_compute_forces_kernel),
            dim3(n_blocks),
            dim3(run_block_size),
            shared_bytes,
            0,
            nwork,
            d_postype,
            d_force,
            grid_dim,
            n_ghost_cells,
            d_charge,
            box,
            order,
            d_index_array,
            local_fft ? d_inv_fourier_mesh_x + idev * inv_mesh_elements : d_inv_fourier_mesh_x,
            local_fft ? d_inv_fourier_mesh_y + idev * inv_mesh_elements : d_inv_fourier_mesh_y,
            local_fft ? d_inv_fourier_mesh_z + idev * inv_mesh_elements : d_inv_fourier_mesh_z,
            d_rho_coeff,
            range.first);
        }
    }

__global__ void kernel_calculate_pe_partial(int n_wave_vectors,
                                            Scalar* sum_partial,
                                            const hipfftComplex* d_fourier_mesh,
                                            const Scalar* d_inf_f,
                                            const bool exclude_dc)
    {
    HIP_DYNAMIC_SHARED(Scalar, sdata)

    unsigned int tidx = threadIdx.x;

    unsigned int j;

    j = blockDim.x * blockIdx.x + threadIdx.x;

    Scalar mySum = Scalar(0.0);

    if (j < n_wave_vectors)
        {
        if (!exclude_dc || j != 0)
            {
            mySum = d_fourier_mesh[j].x * d_fourier_mesh[j].x
                    + d_fourier_mesh[j].y * d_fourier_mesh[j].y;
            mySum *= d_inf_f[j];
            }
        }

    sdata[tidx] = mySum;

    __syncthreads();

    // reduce the sum
    int offs = blockDim.x >> 1;
    while (offs > 0)
        {
        if (tidx < offs)
            {
            sdata[tidx] += sdata[tidx + offs];
            }
        offs >>= 1;
        __syncthreads();
        }

    // write result to global memory
    if (tidx == 0)
        sum_partial[blockIdx.x] = sdata[0];
    }

__global__ void kernel_final_reduce_pe(Scalar* sum_partial, unsigned int nblocks, Scalar* sum)
    {
    HIP_DYNAMIC_SHARED(Scalar, smem)

    if (threadIdx.x == 0)
        *sum = Scalar(0.0);

    for (int start = 0; start < nblocks; start += blockDim.x)
        {
        __syncthreads();
        if (start + threadIdx.x < nblocks)
            smem[threadIdx.x] = sum_partial[start + threadIdx.x];
        else
            smem[threadIdx.x] = Scalar(0.0);

        __syncthreads();

        // reduce the sum
        int offs = blockDim.x >> 1;
        while (offs > 0)
            {
            if (threadIdx.x < offs)
                smem[threadIdx.x] += smem[threadIdx.x + offs];
            offs >>= 1;
            __syncthreads();
            }

        if (threadIdx.x == 0)
            {
            *sum += smem[0];
            }
        }
    }

void gpu_compute_pe(unsigned int n_wave_vectors,
                    Scalar* d_sum_partial,
                    Scalar* d_sum,
                    const hipfftComplex* d_fourier_mesh,
                    const Scalar* d_inf_f,
                    const unsigned int block_size,
                    const uint3 mesh_dim,
                    const bool exclude_dc)
    {
    unsigned int n_blocks = n_wave_vectors / block_size + 1;

    unsigned int shared_size = (unsigned int)(block_size * sizeof(Scalar));

    dim3 grid(n_blocks, 1, 1);

    hipLaunchKernelGGL((kernel_calculate_pe_partial),
                       dim3(grid),
                       dim3(block_size),
                       shared_size,
                       0,
                       n_wave_vectors,
                       d_sum_partial,
                       d_fourier_mesh,
                       d_inf_f,
                       exclude_dc);

    // calculate final sum of mesh values
    const unsigned int final_block_size = 256;
    shared_size = final_block_size * sizeof(Scalar);
    hipLaunchKernelGGL((kernel_final_reduce_pe),
                       dim3(1),
                       dim3(final_block_size),
                       shared_size,
                       0,
                       d_sum_partial,
                       n_blocks,
                       d_sum);
    }

__global__ void kernel_calculate_virial_partial(int n_wave_vectors,
                                                Scalar* sum_virial_partial,
                                                const Scalar* d_mesh_virial)
    {
    HIP_DYNAMIC_SHARED(Scalar, sdata)

    unsigned int j;

    j = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned int tidx = threadIdx.x;

    Scalar mySum_xx = Scalar(0.0);
    Scalar mySum_xy = Scalar(0.0);
    Scalar mySum_xz = Scalar(0.0);
    Scalar mySum_yy = Scalar(0.0);
    Scalar mySum_yz = Scalar(0.0);
    Scalar mySum_zz = Scalar(0.0);

    if (j < n_wave_vectors)
        {
        mySum_xx = d_mesh_virial[0 * n_wave_vectors + j];
        mySum_xy = d_mesh_virial[1 * n_wave_vectors + j];
        mySum_xz = d_mesh_virial[2 * n_wave_vectors + j];
        mySum_yy = d_mesh_virial[3 * n_wave_vectors + j];
        mySum_yz = d_mesh_virial[4 * n_wave_vectors + j];
        mySum_zz = d_mesh_virial[5 * n_wave_vectors + j];
        }

    sdata[0 * blockDim.x + tidx] = mySum_xx;
    sdata[1 * blockDim.x + tidx] = mySum_xy;
    sdata[2 * blockDim.x + tidx] = mySum_xz;
    sdata[3 * blockDim.x + tidx] = mySum_yy;
    sdata[4 * blockDim.x + tidx] = mySum_yz;
    sdata[5 * blockDim.x + tidx] = mySum_zz;

    __syncthreads();

    // reduce the sum
    int offs = blockDim.x >> 1;
    while (offs > 0)
        {
        if (tidx < offs)
            {
            sdata[0 * blockDim.x + tidx] += sdata[0 * blockDim.x + tidx + offs];
            sdata[1 * blockDim.x + tidx] += sdata[1 * blockDim.x + tidx + offs];
            sdata[2 * blockDim.x + tidx] += sdata[2 * blockDim.x + tidx + offs];
            sdata[3 * blockDim.x + tidx] += sdata[3 * blockDim.x + tidx + offs];
            sdata[4 * blockDim.x + tidx] += sdata[4 * blockDim.x + tidx + offs];
            sdata[5 * blockDim.x + tidx] += sdata[5 * blockDim.x + tidx + offs];
            }
        offs >>= 1;
        __syncthreads();
        }

    // write result to global memory
    if (tidx == 0)
        {
        sum_virial_partial[0 * gridDim.x + blockIdx.x] = sdata[0 * blockDim.x];
        sum_virial_partial[1 * gridDim.x + blockIdx.x] = sdata[1 * blockDim.x];
        sum_virial_partial[2 * gridDim.x + blockIdx.x] = sdata[2 * blockDim.x];
        sum_virial_partial[3 * gridDim.x + blockIdx.x] = sdata[3 * blockDim.x];
        sum_virial_partial[4 * gridDim.x + blockIdx.x] = sdata[4 * blockDim.x];
        sum_virial_partial[5 * gridDim.x + blockIdx.x] = sdata[5 * blockDim.x];
        }
    }

__global__ void
kernel_final_reduce_virial(Scalar* sum_virial_partial, unsigned int nblocks, Scalar* sum_virial)
    {
    HIP_DYNAMIC_SHARED(Scalar, smem)

    if (threadIdx.x == 0)
        {
        sum_virial[0] = Scalar(0.0);
        sum_virial[1] = Scalar(0.0);
        sum_virial[2] = Scalar(0.0);
        sum_virial[3] = Scalar(0.0);
        sum_virial[4] = Scalar(0.0);
        sum_virial[5] = Scalar(0.0);
        }

    for (int start = 0; start < nblocks; start += blockDim.x)
        {
        __syncthreads();
        if (start + threadIdx.x < nblocks)
            {
            smem[0 * blockDim.x + threadIdx.x]
                = sum_virial_partial[0 * nblocks + start + threadIdx.x];
            smem[1 * blockDim.x + threadIdx.x]
                = sum_virial_partial[1 * nblocks + start + threadIdx.x];
            smem[2 * blockDim.x + threadIdx.x]
                = sum_virial_partial[2 * nblocks + start + threadIdx.x];
            smem[3 * blockDim.x + threadIdx.x]
                = sum_virial_partial[3 * nblocks + start + threadIdx.x];
            smem[4 * blockDim.x + threadIdx.x]
                = sum_virial_partial[4 * nblocks + start + threadIdx.x];
            smem[5 * blockDim.x + threadIdx.x]
                = sum_virial_partial[5 * nblocks + start + threadIdx.x];
            }
        else
            {
            smem[0 * blockDim.x + threadIdx.x] = Scalar(0.0);
            smem[1 * blockDim.x + threadIdx.x] = Scalar(0.0);
            smem[2 * blockDim.x + threadIdx.x] = Scalar(0.0);
            smem[3 * blockDim.x + threadIdx.x] = Scalar(0.0);
            smem[4 * blockDim.x + threadIdx.x] = Scalar(0.0);
            smem[5 * blockDim.x + threadIdx.x] = Scalar(0.0);
            }

        __syncthreads();

        // reduce the sum
        int offs = blockDim.x >> 1;
        while (offs > 0)
            {
            if (threadIdx.x < offs)
                {
                smem[0 * blockDim.x + threadIdx.x] += smem[0 * blockDim.x + threadIdx.x + offs];
                smem[1 * blockDim.x + threadIdx.x] += smem[1 * blockDim.x + threadIdx.x + offs];
                smem[2 * blockDim.x + threadIdx.x] += smem[2 * blockDim.x + threadIdx.x + offs];
                smem[3 * blockDim.x + threadIdx.x] += smem[3 * blockDim.x + threadIdx.x + offs];
                smem[4 * blockDim.x + threadIdx.x] += smem[4 * blockDim.x + threadIdx.x + offs];
                smem[5 * blockDim.x + threadIdx.x] += smem[5 * blockDim.x + threadIdx.x + offs];
                }
            offs >>= 1;
            __syncthreads();
            }

        if (threadIdx.x == 0)
            {
            sum_virial[0] += smem[0 * blockDim.x];
            sum_virial[1] += smem[1 * blockDim.x];
            sum_virial[2] += smem[2 * blockDim.x];
            sum_virial[3] += smem[3 * blockDim.x];
            sum_virial[4] += smem[4 * blockDim.x];
            sum_virial[5] += smem[5 * blockDim.x];
            }
        }
    }

void gpu_compute_virial(unsigned int n_wave_vectors,
                        Scalar* d_sum_virial_partial,
                        Scalar* d_sum_virial,
                        const Scalar* d_mesh_virial,
                        const unsigned int block_size)
    {
    unsigned int n_blocks = n_wave_vectors / block_size + 1;

    unsigned int shared_size = (unsigned int)(6 * block_size * sizeof(Scalar));

    dim3 grid(n_blocks, 1, 1);

    hipLaunchKernelGGL((kernel_calculate_virial_partial),
                       dim3(grid),
                       dim3(block_size),
                       shared_size,
                       0,
                       n_wave_vectors,
                       d_sum_virial_partial,
                       d_mesh_virial);

    // calculate final virial values
    const unsigned int final_block_size = 256;
    shared_size = 6 * final_block_size * sizeof(Scalar);
    hipLaunchKernelGGL((kernel_final_reduce_virial),
                       dim3(1),
                       dim3(final_block_size),
                       shared_size,
                       0,
                       d_sum_virial_partial,
                       n_blocks,
                       d_sum_virial);
    }

template<bool local_fft>
__global__ void gpu_compute_influence_function_kernel(const uint3 mesh_dim,
                                                      const unsigned int n_wave_vectors,
                                                      const uint3 global_dim,
                                                      Scalar* d_inf_f,
                                                      Scalar3* d_k,
                                                      const Scalar3 b1,
                                                      const Scalar3 b2,
                                                      const Scalar3 b3,
                                                      const uint3 pidx,
                                                      const uint3 pdim,
                                                      int nbx,
                                                      int nby,
                                                      int nbz,
                                                      const Scalar* gf_b,
                                                      int order,
                                                      Scalar kappa,
                                                      Scalar alpha)
    {
    unsigned int kidx;

    kidx = blockDim.x * blockIdx.x + threadIdx.x;

    if (kidx >= n_wave_vectors)
        return;

    int l, m, n;
    if (local_fft)
        {
        // use row-major layout
        int ny = mesh_dim.y;
        int nx = mesh_dim.x;
        n = kidx / ny / nx;
        m = (kidx - n * ny * nx) / nx;
        l = kidx % nx;
        }
#ifdef ENABLE_MPI
    else
        {
        // local layout: row-major
        int ny = mesh_dim.y;
        int nx = mesh_dim.x;
        int n_local = kidx / ny / nx;
        int m_local = (kidx - n_local * ny * nx) / nx;
        int l_local = kidx % nx;

        // cyclic distribution
        l = l_local * pdim.x + pidx.x;
        m = m_local * pdim.y + pidx.y;
        n = n_local * pdim.z + pidx.z;
        }
#endif

    // compute Miller indices
    if (l >= (int)(global_dim.x / 2 + global_dim.x % 2))
        l -= (int)global_dim.x;
    if (m >= (int)(global_dim.y / 2 + global_dim.y % 2))
        m -= (int)global_dim.y;
    if (n >= (int)(global_dim.z / 2 + global_dim.z % 2))
        n -= (int)global_dim.z;

    Scalar val;
    Scalar3 kval = (Scalar)l * b1 + (Scalar)m * b2 + (Scalar)n * b3;

    Scalar3 kH = Scalar(2.0 * M_PI)
                 * make_scalar3(Scalar(1.0) / (Scalar)global_dim.x,
                                Scalar(1.0) / (Scalar)global_dim.y,
                                Scalar(1.0) / (Scalar)global_dim.z);

    Scalar snx = fast::sin(Scalar(0.5) * l * kH.x);
    Scalar snx2 = snx * snx;

    Scalar sny = fast::sin(Scalar(0.5) * m * kH.y);
    Scalar sny2 = sny * sny;

    Scalar snz = fast::sin(Scalar(0.5) * n * kH.z);
    Scalar snz2 = snz * snz;

    Scalar sx(0.0), sy(0.0), sz(0.0);
    for (int iorder = order - 1; iorder >= 0; iorder--)
        {
        sx = gf_b[iorder] + sx * snx2;
        sy = gf_b[iorder] + sy * sny2;
        sz = gf_b[iorder] + sz * snz2;
        }
    Scalar denominator = sx * sy * sz;
    denominator *= denominator;

    if (l != 0 || m != 0 || n != 0)
        {
        Scalar sum1(0.0);
        Scalar numerator = Scalar(4.0 * M_PI) / dot(kval, kval);

        for (int ix = -nbx; ix <= nbx; ix++)
            {
            Scalar qx = ((Scalar)l + (Scalar)ix * global_dim.x);
            Scalar3 knx = qx * b1;

            Scalar argx = Scalar(0.5) * qx * kH.x;
            Scalar wxs = gpu_sinc(argx);
            Scalar wx(1.0);
            for (int iorder = 0; iorder < order; ++iorder)
                {
                wx *= wxs;
                }

            for (int iy = -nby; iy <= nby; iy++)
                {
                Scalar qy = ((Scalar)m + (Scalar)iy * global_dim.y);
                Scalar3 kny = qy * b2;

                Scalar argy = Scalar(0.5) * qy * kH.y;
                Scalar wys = gpu_sinc(argy);
                Scalar wy(1.0);
                for (int iorder = 0; iorder < order; ++iorder)
                    {
                    wy *= wys;
                    }

                for (int iz = -nbz; iz <= nbz; iz++)
                    {
                    Scalar qz = ((Scalar)n + (Scalar)iz * global_dim.z);
                    Scalar3 knz = qz * b3;

                    Scalar argz = Scalar(0.5) * qz * kH.z;
                    Scalar wzs = gpu_sinc(argz);
                    Scalar wz(1.0);
                    for (int iorder = 0; iorder < order; ++iorder)
                        {
                        wz *= wzs;
                        }

                    Scalar3 kn = knx + kny + knz;
                    Scalar dot1 = dot(kn, kval);
                    Scalar dot2 = dot(kn, kn) + alpha * alpha;

                    Scalar arg_gauss = Scalar(0.25) * dot2 / kappa / kappa;
                    Scalar gauss = exp(-arg_gauss);

                    sum1 += (dot1 / dot2) * gauss * wx * wx * wy * wy * wz * wz;
                    }
                }
            }
        val = numerator * sum1 / denominator;
        }
    else
        {
        val = Scalar(0.0);
        }

    // write out result
    d_inf_f[kidx] = val;
    d_k[kidx] = kval;
    }

void gpu_compute_influence_function(const uint3 mesh_dim,
                                    const uint3 global_dim,
                                    Scalar* d_inf_f,
                                    Scalar3* d_k,
                                    const BoxDim& global_box,
                                    const bool local_fft,
                                    const uint3 pidx,
                                    const uint3 pdim,
                                    const Scalar EPS_HOC,
                                    Scalar kappa,
                                    Scalar alpha,
                                    const Scalar* d_gf_b,
                                    int order,
                                    unsigned int block_size)
    {
    // compute reciprocal lattice vectors
    Scalar3 a1 = global_box.getLatticeVector(0);
    Scalar3 a2 = global_box.getLatticeVector(1);
    Scalar3 a3 = global_box.getLatticeVector(2);

    Scalar V_box = global_box.getVolume();
    Scalar3 b1 = Scalar(2.0 * M_PI)
                 * make_scalar3(a2.y * a3.z - a2.z * a3.y,
                                a2.z * a3.x - a2.x * a3.z,
                                a2.x * a3.y - a2.y * a3.x)
                 / V_box;
    Scalar3 b2 = Scalar(2.0 * M_PI)
                 * make_scalar3(a3.y * a1.z - a3.z * a1.y,
                                a3.z * a1.x - a3.x * a1.z,
                                a3.x * a1.y - a3.y * a1.x)
                 / V_box;
    Scalar3 b3 = Scalar(2.0 * M_PI)
                 * make_scalar3(a1.y * a2.z - a1.z * a2.y,
                                a1.z * a2.x - a1.x * a2.z,
                                a1.x * a2.y - a1.y * a2.x)
                 / V_box;

    unsigned int num_wave_vectors = mesh_dim.x * mesh_dim.y * mesh_dim.z;

    Scalar3 L = global_box.getL();
    Scalar temp = floor(((kappa * L.x / (M_PI * global_dim.x)) * pow(-log(EPS_HOC), 0.25)));
    int nbx = (int)temp;
    temp = floor(((kappa * L.y / (M_PI * global_dim.y)) * pow(-log(EPS_HOC), 0.25)));
    int nby = (int)temp;
    temp = floor(((kappa * L.z / (M_PI * global_dim.z)) * pow(-log(EPS_HOC), 0.25)));
    int nbz = (int)temp;

    if (local_fft)
        {
        unsigned int max_block_size;
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_influence_function_kernel<true>);
        max_block_size = attr.maxThreadsPerBlock;

        unsigned int run_block_size = min(max_block_size, block_size);

        unsigned int n_blocks = num_wave_vectors / run_block_size;
        if (num_wave_vectors % run_block_size)
            n_blocks += 1;

        dim3 grid(n_blocks, 1, 1);

        hipLaunchKernelGGL((gpu_compute_influence_function_kernel<true>),
                           dim3(grid),
                           dim3(run_block_size),
                           0,
                           0,
                           mesh_dim,
                           num_wave_vectors,
                           global_dim,
                           d_inf_f,
                           d_k,
                           b1,
                           b2,
                           b3,
                           pidx,
                           pdim,
                           nbx,
                           nby,
                           nbz,
                           d_gf_b,
                           order,
                           kappa,
                           alpha);
        }
#ifdef ENABLE_MPI
    else
        {
        unsigned int max_block_size;
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_influence_function_kernel<false>);
        max_block_size = attr.maxThreadsPerBlock;

        unsigned int run_block_size = min(max_block_size, block_size);

        unsigned int n_blocks = num_wave_vectors / run_block_size;
        if (num_wave_vectors % run_block_size)
            n_blocks += 1;

        dim3 grid(n_blocks, 1, 1);

        hipLaunchKernelGGL((gpu_compute_influence_function_kernel<false>),
                           dim3(grid),
                           dim3(run_block_size),
                           0,
                           0,
                           mesh_dim,
                           num_wave_vectors,
                           global_dim,
                           d_inf_f,
                           d_k,
                           b1,
                           b2,
                           b3,
                           pidx,
                           pdim,
                           nbx,
                           nby,
                           nbz,
                           d_gf_b,
                           order,
                           kappa,
                           alpha);
        }
#endif
    }

//! The developer has chosen not to document this function
__global__ void gpu_fix_exclusions_kernel(Scalar4* d_force,
                                          Scalar* d_virial,
                                          const size_t virial_pitch,
                                          const Scalar4* d_pos,
                                          const Scalar* d_charge,
                                          const BoxDim box,
                                          const unsigned int* d_n_neigh,
                                          const unsigned int* d_nlist,
                                          const Index2D nli,
                                          Scalar kappa,
                                          Scalar alpha,
                                          unsigned int* d_group_members,
                                          unsigned int group_size)
    {
    // start by identifying which particle we are to handle
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];
        const Scalar sqrtpi = sqrtf(M_PI);
        unsigned int n_neigh = d_n_neigh[idx];
        Scalar4 postypei = __ldg(d_pos + idx);
        Scalar3 posi = make_scalar3(postypei.x, postypei.y, postypei.z);

        Scalar qi = __ldg(d_charge + idx);
        // initialize the force to 0
        Scalar4 force = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
        Scalar virial[6];
        for (unsigned int i = 0; i < 6; i++)
            virial[i] = Scalar(0.0);
        unsigned int cur_j = 0;
        // prefetch neighbor index
        unsigned int next_j = d_nlist[nli(idx, 0)];

        for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
            {
                {
                // read the current neighbor index (MEM TRANSFER: 4 bytes)
                // prefetch the next value and set the current one
                cur_j = next_j;
                if (neigh_idx + 1 < n_neigh)
                    next_j = d_nlist[nli(idx, neigh_idx + 1)];

                // get the neighbor's position (MEM TRANSFER: 16 bytes)
                Scalar4 postypej = __ldg(d_pos + cur_j);
                Scalar3 posj = make_scalar3(postypej.x, postypej.y, postypej.z);

                Scalar qj = __ldg(d_charge + cur_j);

                // calculate dr (with periodic boundary conditions) (FLOPS: 3)
                Scalar3 dx = posi - posj;

                // apply periodic boundary conditions: (FLOPS 12)
                dx = box.minImage(dx);

                // calculate r squared (FLOPS: 5)
                Scalar rsq = dot(dx, dx);
                Scalar r = sqrtf(rsq);
                Scalar qiqj = qi * qj;
                Scalar expfac = fast::exp(-alpha * r);
                Scalar arg1 = kappa * r - alpha / Scalar(2.0) / kappa;
                Scalar arg2 = kappa * r + alpha / Scalar(2.0) / kappa;
                Scalar erffac = (::erf(arg1) * expfac + expfac - fast::erfc(arg2) * exp(alpha * r))
                                / (Scalar(2.0) * r);

                Scalar force_divr
                    = qiqj
                      * (expfac * Scalar(2.0) * kappa / sqrtpi * fast::exp(-arg1 * arg1)
                         - Scalar(0.5) * alpha
                               * (expfac * ::erfc(arg1) + fast::exp(alpha * r) * fast::erfc(arg2))
                         - erffac)
                      / rsq;

                // subtract long-range part of pair-interaction
                Scalar pair_eng = -qiqj * erffac;

                Scalar force_div2r = Scalar(0.5) * force_divr;
                virial[0] += dx.x * dx.x * force_div2r;
                virial[1] += dx.x * dx.y * force_div2r;
                virial[2] += dx.x * dx.z * force_div2r;
                virial[3] += dx.y * dx.y * force_div2r;
                virial[4] += dx.y * dx.z * force_div2r;
                virial[5] += dx.z * dx.z * force_div2r;

                force.x += dx.x * force_divr;
                force.y += dx.y * force_divr;
                force.z += dx.z * force_divr;

                force.w += pair_eng;
                }
            }
        force.w *= Scalar(0.5);
        d_force[idx].x += force.x;
        d_force[idx].y += force.y;
        d_force[idx].z += force.z;
        d_force[idx].w += force.w;
        for (unsigned int i = 0; i < 6; i++)
            d_virial[i * virial_pitch + idx] += virial[i];
        }
    }

//! The developer has chosen not to document this function
hipError_t gpu_fix_exclusions(Scalar4* d_force,
                              Scalar* d_virial,
                              const size_t virial_pitch,
                              const unsigned int Nmax,
                              const Scalar4* d_pos,
                              const Scalar* d_charge,
                              const BoxDim& box,
                              const unsigned int* d_n_ex,
                              const unsigned int* d_exlist,
                              const Index2D nex,
                              Scalar kappa,
                              Scalar alpha,
                              unsigned int* d_group_members,
                              unsigned int group_size,
                              int block_size)
    {
    dim3 grid(group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    hipLaunchKernelGGL((gpu_fix_exclusions_kernel),
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       d_force,
                       d_virial,
                       virial_pitch,
                       d_pos,
                       d_charge,
                       box,
                       d_n_ex,
                       d_exlist,
                       nex,
                       kappa,
                       alpha,
                       d_group_members,
                       group_size);
    return hipSuccess;
    }

    } // namespace kernel
    } // end namespace md
    } // end namespace hoomd
