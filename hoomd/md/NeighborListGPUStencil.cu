#include "hip/hip_runtime.h"
// Copyright (c) 2009-2023 The Regents of the University of Michigan.
// Part of HOOMD-blue, released under the BSD 3-Clause License.

#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "NeighborListGPUStencil.cuh"
#include "hoomd/TextureTools.h"
#include "hoomd/WarpTools.cuh"
#include <hipcub/hipcub.hpp>

/*! \file NeighborListGPUStencil.cu
    \brief Defines GPU kernel code for O(N) neighbor list generation on the GPU with multiple bin
   stencils
*/

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
//! Kernel call for generating neighbor list on the GPU using multiple stencils (Kepler optimized
//! version)
/*! \tparam filter_body Set to true to enable body filtering.
    \tparam threads_per_particle Number of threads cooperatively computing the neighbor list
    \param d_nlist Neighbor list data structure to write
    \param d_n_neigh Number of neighbors to write
    \param d_last_updated_pos Particle positions at this update are written to this array
    \param d_conditions Conditions array for writing overflow condition
    \param d_Nmax Maximum number of neighbors per type
    \param d_head_list List of indexes to access \a d_nlist
    \param d_pos Particle positions
    \param d_body Particle body indices
    \param N Number of particles
    \param d_cell_size Number of particles in each cell
    \param d_cell_xyzf Cell contents (xyzf array from CellList with flag=type)
    \param d_cell_type_body Cell contents (TypeBody array from CellList with)
    \param ci Cell indexer for indexing cells
    \param cli Cell list indexer for indexing into d_cell_xyzf
    \param d_stencil 2D array of stencil offsets per type
    \param d_n_stencil Number of stencils per type
    \param stencil_idx Indexer into \a d_stencil
    \param box Simulation box dimensions
    \param d_r_cut Cutoff radius stored by pair type r_cut(i,j)
    \param r_buff The maximum radius for which to include particles as neighbors
    \param ntypes Number of particle types
    \param ghost_width Width of ghost cell layer

    \note optimized for Kepler
*/
template<unsigned char filter_body, int threads_per_particle>
__global__ void gpu_compute_nlist_stencil_kernel(unsigned int* d_nlist,
                                                 unsigned int* d_n_neigh,
                                                 Scalar4* d_last_updated_pos,
                                                 unsigned int* d_conditions,
                                                 const unsigned int* d_Nmax,
                                                 const size_t* d_head_list,
                                                 const unsigned int* d_pid_map,
                                                 const Scalar4* d_pos,
                                                 const unsigned int* d_body,
                                                 const unsigned int N,
                                                 const unsigned int* d_cell_size,
                                                 const Scalar4* d_cell_xyzf,
                                                 const uint2* d_cell_type_body,
                                                 const Index3D ci,
                                                 const Index2D cli,
                                                 const Scalar4* d_stencil,
                                                 const unsigned int* d_n_stencil,
                                                 const Index2D stencil_idx,
                                                 const BoxDim box,
                                                 const Scalar* d_r_cut,
                                                 const Scalar r_buff,
                                                 const unsigned int ntypes,
                                                 const Scalar3 ghost_width)
    {
    // cache the r_listsq parameters into shared memory
    Index2D typpair_idx(ntypes);
    const unsigned int num_typ_parameters = typpair_idx.getNumElements();

    // shared data for per type pair parameters
    HIP_DYNAMIC_SHARED(unsigned char, s_data)

    // pointer for the r_listsq data
    Scalar* s_r_list = (Scalar*)(&s_data[0]);
    unsigned int* s_Nmax = (unsigned int*)(&s_data[sizeof(Scalar) * num_typ_parameters]);

    // load in the per type pair r_list
    for (unsigned int cur_offset = 0; cur_offset < num_typ_parameters; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < num_typ_parameters)
            {
            Scalar r_cut = d_r_cut[cur_offset + threadIdx.x];
            // force the r_list(i,j) to a skippable value if r_cut(i,j) is skippable
            s_r_list[cur_offset + threadIdx.x]
                = (r_cut > Scalar(0.0)) ? r_cut + r_buff : Scalar(-1.0);
            }
        if (cur_offset + threadIdx.x < ntypes)
            {
            s_Nmax[cur_offset + threadIdx.x] = d_Nmax[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();

    // each set of threads_per_particle threads is going to compute the neighbor list for a single
    // particle
    const int idx
        = blockIdx.x * (blockDim.x / threads_per_particle) + threadIdx.x / threads_per_particle;

    // one thread per particle
    if (idx >= N)
        return;

    // get the write particle id
    int my_pidx = d_pid_map[idx];

    Scalar4 my_postype = d_pos[my_pidx];
    Scalar3 my_pos = make_scalar3(my_postype.x, my_postype.y, my_postype.z);

    unsigned int my_type = __scalar_as_int(my_postype.w);
    unsigned int my_body = d_body[my_pidx];
    size_t my_head = d_head_list[my_pidx];

    Scalar3 f = box.makeFraction(my_pos, ghost_width);

    // find the bin each particle belongs in
    int ib = (int)(f.x * ci.getW());
    int jb = (int)(f.y * ci.getH());
    int kb = (int)(f.z * ci.getD());

    uchar3 periodic = box.getPeriodic();

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    int my_cell = ci(ib, jb, kb);

    // number of available stencils
    unsigned int n_stencil = d_n_stencil[my_type];

    // index of current stencil (-1 to initialize)
    int cur_adj = -1;
    Scalar cell_dist2 = 0.0;

    // current cell (0 to initialize)
    unsigned int neigh_cell = 0;

    // size of current cell (0 to initialize)
    unsigned int neigh_size = 0;

    // current index in cell
    int cur_offset = threadIdx.x % threads_per_particle;

    bool done = false;

    // total number of neighbors
    unsigned int nneigh = 0;

    while (!done)
        {
        // initialize with default
        unsigned int neighbor;
        unsigned char has_neighbor = 0;

        // advance neighbor cell
        while (cur_offset >= neigh_size && !done)
            {
            cur_offset -= neigh_size;
            cur_adj++;

            if (cur_adj < n_stencil)
                {
                // compute the stenciled cell cartesian coordinates
                Scalar4 stencil = __ldg(d_stencil + stencil_idx(cur_adj, my_type));
                int sib = ib + __scalar_as_int(stencil.x);
                int sjb = jb + __scalar_as_int(stencil.y);
                int skb = kb + __scalar_as_int(stencil.z);
                cell_dist2 = stencil.w;

                // wrap through the boundary
                if (sib >= (int)ci.getW() && periodic.x)
                    sib -= ci.getW();
                if (sib < 0 && periodic.x)
                    sib += ci.getW();
                if (sjb >= (int)ci.getH() && periodic.y)
                    sjb -= ci.getH();
                if (sjb < 0 && periodic.y)
                    sjb += ci.getH();
                if (skb >= (int)ci.getD() && periodic.z)
                    skb -= ci.getD();
                if (skb < 0 && periodic.z)
                    skb += ci.getD();

                neigh_cell = ci(sib, sjb, skb);
                neigh_size = d_cell_size[neigh_cell];
                }
            else
                {
                // we are past the end of the cell neighbors
                done = true;
                }
            }

        // check for a neighbor if thread is still working
        if (!done)
            {
            // use a do {} while(0) loop to process this particle so we can break for exclusions
            // in microbenchmarks, this is was faster than using bool exclude because it saved flops
            // it's a little easier to read than having 4 levels of if{} statements nested
            do
                {
                // read in the particle type and body
                const uint2 neigh_type_body = __ldg(d_cell_type_body + cli(cur_offset, neigh_cell));
                const unsigned int type_j = neigh_type_body.x;
                const unsigned int body_j = neigh_type_body.y;

                // skip any particles belonging to the same rigid body if requested
                if (filter_body && my_body != 0xffffffff && my_body == body_j)
                    break;

                // compute the rlist based on the particle type we're interacting with
                Scalar r_list = s_r_list[typpair_idx(my_type, type_j)];
                if (r_list <= Scalar(0.0))
                    break;
                Scalar r_listsq = r_list * r_list;

                // compare the check distance to the minimum cell distance, and pass without
                // distance check if unnecessary
                if (cell_dist2 > r_listsq)
                    break;

                // only load in the particle position and id if distance check is required
                const Scalar4 neigh_xyzf = __ldg(d_cell_xyzf + cli(cur_offset, neigh_cell));
                const Scalar3 neigh_pos = make_scalar3(neigh_xyzf.x, neigh_xyzf.y, neigh_xyzf.z);
                unsigned int cur_neigh = __scalar_as_int(neigh_xyzf.w);

                // a particle cannot neighbor itself
                if (my_pidx == (int)cur_neigh)
                    break;

                Scalar3 dx = my_pos - neigh_pos;
                dx = box.minImage(dx);

                Scalar dr_sq = dot(dx, dx);

                if (dr_sq <= r_listsq)
                    {
                    neighbor = cur_neigh;
                    has_neighbor = 1;
                    }
                } while (0); // particle is processed exactly once

            // advance cur_offset
            cur_offset += threads_per_particle;
            }

        // now that possible neighbor checks are finished, done (for the cta) depends only on first
        // thread neighbor list only needs to get written into if thread 0 is not done
        done = hoomd::detail::WarpScan<bool, threads_per_particle>().Broadcast(done, 0);
        if (!done)
            {
            // scan over flags
            unsigned char k(0), n(0);
            hoomd::detail::WarpScan<unsigned char, threads_per_particle>().ExclusiveSum(
                has_neighbor,
                k,
                n);

            // write neighbor if it fits in list
            if (has_neighbor && (nneigh + k) < s_Nmax[my_type])
                d_nlist[my_head + nneigh + k] = neighbor;

            // increment total neighbor count
            nneigh += n;
            }
        } // end while

    if (threadIdx.x % threads_per_particle == 0)
        {
        // flag if we need to grow the neighbor list
        if (nneigh >= s_Nmax[my_type])
            atomicMax(&d_conditions[my_type], nneigh);

        d_n_neigh[my_pidx] = nneigh;
        d_last_updated_pos[my_pidx] = my_postype;
        }
    }

//! determine maximum possible block size
template<typename T> int get_max_block_size_stencil(T func)
    {
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))func);
    int max_threads = attr.maxThreadsPerBlock;
    // number of threads has to be multiple of warp size
    max_threads -= max_threads % max_threads_per_particle;
    return max_threads;
    }

//! recursive template to launch neighborlist with given template parameters
/* \tparam cur_tpp Number of threads per particle (assumed to be power of two) */
template<int cur_tpp>
inline void stencil_launcher(unsigned int* d_nlist,
                             unsigned int* d_n_neigh,
                             Scalar4* d_last_updated_pos,
                             unsigned int* d_conditions,
                             const unsigned int* d_Nmax,
                             const size_t* d_head_list,
                             const unsigned int* d_pid_map,
                             const Scalar4* d_pos,
                             const unsigned int* d_body,
                             const unsigned int N,
                             const unsigned int* d_cell_size,
                             const Scalar4* d_cell_xyzf,
                             const uint2* d_cell_type_body,
                             const Index3D& ci,
                             const Index2D& cli,
                             const Scalar4* d_stencil,
                             const unsigned int* d_n_stencil,
                             const Index2D& stencil_idx,
                             const BoxDim& box,
                             const Scalar* d_r_cut,
                             const Scalar r_buff,
                             const unsigned int ntypes,
                             const Scalar3& ghost_width,
                             bool filter_body,
                             const unsigned int threads_per_particle,
                             const unsigned int block_size,
                             const hipDeviceProp_t& devprop)
    {
    // shared memory = r_listsq + Nmax + stuff needed for neighborlist (computed below)
    Index2D typpair_idx(ntypes);
    unsigned int shared_size = (unsigned int)(sizeof(Scalar) * typpair_idx.getNumElements()
                                              + sizeof(unsigned int) * ntypes);

    if (shared_size > devprop.sharedMemPerBlock)
        {
        throw std::runtime_error("Neighborlist r_cut matrix exceeds the available shared memory "
                                 "per block.");
        }

    if (threads_per_particle == cur_tpp && cur_tpp != 0)
        {
        if (!filter_body)
            {
            unsigned int max_block_size;
            max_block_size
                = get_max_block_size_stencil(gpu_compute_nlist_stencil_kernel<0, cur_tpp>);

            unsigned int run_block_size
                = (block_size < max_block_size) ? block_size : max_block_size;
            dim3 grid(N / (block_size / threads_per_particle) + 1);
            hipLaunchKernelGGL((gpu_compute_nlist_stencil_kernel<0, cur_tpp>),
                               dim3(grid),
                               dim3(run_block_size),
                               shared_size,
                               0,
                               d_nlist,
                               d_n_neigh,
                               d_last_updated_pos,
                               d_conditions,
                               d_Nmax,
                               d_head_list,
                               d_pid_map,
                               d_pos,
                               d_body,
                               N,
                               d_cell_size,
                               d_cell_xyzf,
                               d_cell_type_body,
                               ci,
                               cli,
                               d_stencil,
                               d_n_stencil,
                               stencil_idx,
                               box,
                               d_r_cut,
                               r_buff,
                               ntypes,
                               ghost_width);
            }
        else if (filter_body)
            {
            unsigned int max_block_size;
            max_block_size
                = get_max_block_size_stencil(gpu_compute_nlist_stencil_kernel<1, cur_tpp>);

            unsigned int run_block_size
                = (block_size < max_block_size) ? block_size : max_block_size;
            dim3 grid(N / (block_size / threads_per_particle) + 1);
            hipLaunchKernelGGL((gpu_compute_nlist_stencil_kernel<1, cur_tpp>),
                               dim3(grid),
                               dim3(run_block_size),
                               shared_size,
                               0,
                               d_nlist,
                               d_n_neigh,
                               d_last_updated_pos,
                               d_conditions,
                               d_Nmax,
                               d_head_list,
                               d_pid_map,
                               d_pos,
                               d_body,
                               N,
                               d_cell_size,
                               d_cell_xyzf,
                               d_cell_type_body,
                               ci,
                               cli,
                               d_stencil,
                               d_n_stencil,
                               stencil_idx,
                               box,
                               d_r_cut,
                               r_buff,
                               ntypes,
                               ghost_width);
            }
        }
    else
        {
        stencil_launcher<cur_tpp / 2>(d_nlist,
                                      d_n_neigh,
                                      d_last_updated_pos,
                                      d_conditions,
                                      d_Nmax,
                                      d_head_list,
                                      d_pid_map,
                                      d_pos,
                                      d_body,
                                      N,
                                      d_cell_size,
                                      d_cell_xyzf,
                                      d_cell_type_body,
                                      ci,
                                      cli,
                                      d_stencil,
                                      d_n_stencil,
                                      stencil_idx,
                                      box,
                                      d_r_cut,
                                      r_buff,
                                      ntypes,
                                      ghost_width,
                                      filter_body,
                                      threads_per_particle,
                                      block_size,
                                      devprop);
        }
    }

//! template specialization to terminate recursion
template<>
inline void stencil_launcher<min_threads_per_particle / 2>(unsigned int* d_nlist,
                                                           unsigned int* d_n_neigh,
                                                           Scalar4* d_last_updated_pos,
                                                           unsigned int* d_conditions,
                                                           const unsigned int* d_Nmax,
                                                           const size_t* d_head_list,
                                                           const unsigned int* d_pid_map,
                                                           const Scalar4* d_pos,
                                                           const unsigned int* d_body,
                                                           const unsigned int N,
                                                           const unsigned int* d_cell_size,
                                                           const Scalar4* d_cell_xyzf,
                                                           const uint2* d_cell_type_body,
                                                           const Index3D& ci,
                                                           const Index2D& cli,
                                                           const Scalar4* d_stencil,
                                                           const unsigned int* d_n_stencil,
                                                           const Index2D& stencil_idx,
                                                           const BoxDim& box,
                                                           const Scalar* d_r_cut,
                                                           const Scalar r_buff,
                                                           const unsigned int ntypes,
                                                           const Scalar3& ghost_width,
                                                           bool filter_body,
                                                           const unsigned int threads_per_particle,
                                                           const unsigned int block_size,
                                                           const hipDeviceProp_t& devprop)
    {
    }

hipError_t gpu_compute_nlist_stencil(unsigned int* d_nlist,
                                     unsigned int* d_n_neigh,
                                     Scalar4* d_last_updated_pos,
                                     unsigned int* d_conditions,
                                     const unsigned int* d_Nmax,
                                     const size_t* d_head_list,
                                     const unsigned int* d_pid_map,
                                     const Scalar4* d_pos,
                                     const unsigned int* d_body,
                                     const unsigned int N,
                                     const unsigned int* d_cell_size,
                                     const Scalar4* d_cell_xyzf,
                                     const uint2* d_cell_type_body,
                                     const Index3D& ci,
                                     const Index2D& cli,
                                     const Scalar4* d_stencil,
                                     const unsigned int* d_n_stencil,
                                     const Index2D& stencil_idx,
                                     const BoxDim& box,
                                     const Scalar* d_r_cut,
                                     const Scalar r_buff,
                                     const unsigned int ntypes,
                                     const Scalar3& ghost_width,
                                     bool filter_body,
                                     const unsigned int threads_per_particle,
                                     const unsigned int block_size,
                                     const hipDeviceProp_t& devprop)
    {
    stencil_launcher<max_threads_per_particle>(d_nlist,
                                               d_n_neigh,
                                               d_last_updated_pos,
                                               d_conditions,
                                               d_Nmax,
                                               d_head_list,
                                               d_pid_map,
                                               d_pos,
                                               d_body,
                                               N,
                                               d_cell_size,
                                               d_cell_xyzf,
                                               d_cell_type_body,
                                               ci,
                                               cli,
                                               d_stencil,
                                               d_n_stencil,
                                               stencil_idx,
                                               box,
                                               d_r_cut,
                                               r_buff,
                                               ntypes,
                                               ghost_width,
                                               filter_body,
                                               threads_per_particle,
                                               block_size,
                                               devprop);
    return hipSuccess;
    }

/*!
 * \param d_pids Unsorted particle indexes
 * \param d_types Unsorted particle types
 * \param d_pos Particle position array
 * \param N Number of particles
 *
 * \a d_pids and \a d_types are trivially initialized to their current (unsorted) values. They are
 * later sorted in gpu_compute_nlist_stencil_sort_types().
 */
__global__ void gpu_compute_nlist_stencil_fill_types_kernel(unsigned int* d_pids,
                                                            unsigned int* d_types,
                                                            const Scalar4* d_pos,
                                                            const unsigned int N)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    Scalar4 pos_i = d_pos[idx];
    unsigned int type = __scalar_as_int(pos_i.w);
    d_types[idx] = type;
    d_pids[idx] = idx;
    }

/*!
 * \param d_pids Unsorted particle indexes
 * \param d_types Unsorted particle types
 * \param d_pos Particle position array
 * \param N Number of particles
 */
hipError_t gpu_compute_nlist_stencil_fill_types(unsigned int* d_pids,
                                                unsigned int* d_types,
                                                const Scalar4* d_pos,
                                                const unsigned int N)
    {
    const unsigned int block_size = 128;

    hipLaunchKernelGGL((gpu_compute_nlist_stencil_fill_types_kernel),
                       dim3(N / block_size + 1),
                       dim3(block_size),
                       0,
                       0,
                       d_pids,
                       d_types,
                       d_pos,
                       N);

    return hipSuccess;
    }

/*!
 * \param d_pids Array of unsorted particle indexes
 * \param d_pids_alt Double buffer for particle indexes
 * \param d_types Array of unsorted particle types
 * \param d_types_alt Double buffer for particle types
 * \param d_tmp_storage Temporary allocation for sorting
 * \param tmp_storage_bytes Size of temporary allocation
 * \param swap Flag to swap the sorted particle indexes into the correct buffer
 * \param N number of particles
 *
 * This wrapper calls the CUB radix sorting methods, and so it needs to be called twice. Initially,
 * \a d_tmp_storage should be NULL, and the necessary temporary storage is saved into \a
 * tmp_storage_bytes. This space must then be allocated into \a d_tmp_storage, and on the second
 * call, the sorting is performed.
 */
void gpu_compute_nlist_stencil_sort_types(unsigned int* d_pids,
                                          unsigned int* d_pids_alt,
                                          unsigned int* d_types,
                                          unsigned int* d_types_alt,
                                          void* d_tmp_storage,
                                          size_t& tmp_storage_bytes,
                                          bool& swap,
                                          const unsigned int N)
    {
    hipcub::DoubleBuffer<unsigned int> d_keys(d_types, d_types_alt);
    hipcub::DoubleBuffer<unsigned int> d_vals(d_pids, d_pids_alt);
    hipcub::DeviceRadixSort::SortPairs(d_tmp_storage, tmp_storage_bytes, d_keys, d_vals, N);
    if (d_tmp_storage != NULL)
        {
        swap = (d_vals.selector == 1);
        }
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
