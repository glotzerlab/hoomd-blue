#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: mphoward

/*!
 * \file mpcd/SlitGeometryFillerGPU.cu
 * \brief Defines GPU functions and kernels used by mpcd::SlitGeometryFillerGPU
 */

#include "ParticleDataUtilities.h"
#include "SlitGeometryFillerGPU.cuh"
#include "hoomd/RNGIdentifiers.h"
#include "hoomd/RandomNumbers.h"

namespace hoomd
    {
namespace mpcd
    {
namespace gpu
    {
namespace kernel
    {
/*!
 * \param d_pos Particle positions
 * \param d_vel Particle velocities
 * \param d_tag Particle tags
 * \param geom Slit geometry to fill
 * \param z_min Lower bound to lower fill region
 * \param z_max Upper bound to upper fill region
 * \param box Local simulation box
 * \param type Type of fill particles
 * \param N_lo Number of particles to fill in lower region
 * \param N_hi Number of particles to fill in upper region
 * \param first_tag First tag of filled particles
 * \param first_idx First (local) particle index of filled particles
 * \param vel_factor Scale factor for uniform normal velocities consistent with particle mass /
 * temperature \param timestep Current timestep \param seed User seed to PRNG for drawing velocities
 *
 * \b Implementation:
 *
 * Using one thread per particle (in both slabs), the thread is assigned to fill either the lower
 * or upper region. This defines a local cuboid of volume to fill. The thread index is translated
 * into a particle tag and local particle index. A random position is drawn within the cuboid. A
 * random velocity is drawn consistent with the speed of the moving wall.
 */
__global__ void slit_draw_particles(Scalar4* d_pos,
                                    Scalar4* d_vel,
                                    unsigned int* d_tag,
                                    const mpcd::detail::SlitGeometry geom,
                                    const Scalar z_min,
                                    const Scalar z_max,
                                    const BoxDim box,
                                    const unsigned int type,
                                    const unsigned int N_lo,
                                    const unsigned int N_tot,
                                    const unsigned int first_tag,
                                    const unsigned int first_idx,
                                    const Scalar vel_factor,
                                    const uint64_t timestep,
                                    const uint16_t seed)
    {
    // one thread per particle
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_tot)
        return;

    // determine the fill region based on current index
    signed char sign = (idx >= N_lo) - (idx < N_lo);
    Scalar3 lo = box.getLo();
    Scalar3 hi = box.getHi();
    if (sign == -1) // bottom
        {
        lo.z = z_min;
        hi.z = -geom.getH();
        }
    else // top
        {
        lo.z = geom.getH();
        hi.z = z_max;
        }

    // particle tag and index
    const unsigned int tag = first_tag + idx;
    const unsigned int pidx = first_idx + idx;
    d_tag[pidx] = tag;

    // initialize random number generator for positions and velocity
    hoomd::RandomGenerator rng(
        hoomd::Seed(hoomd::RNGIdentifier::SlitGeometryFiller, timestep, seed),
        hoomd::Counter(tag));
    d_pos[pidx] = make_scalar4(hoomd::UniformDistribution<Scalar>(lo.x, hi.x)(rng),
                               hoomd::UniformDistribution<Scalar>(lo.y, hi.y)(rng),
                               hoomd::UniformDistribution<Scalar>(lo.z, hi.z)(rng),
                               __int_as_scalar(type));

    hoomd::NormalDistribution<Scalar> gen(vel_factor, 0.0);
    Scalar3 vel;
    gen(vel.x, vel.y, rng);
    vel.z = gen(rng);
    // TODO: should these be given zero net-momentum contribution (relative to the frame of
    // reference?)
    d_vel[pidx] = make_scalar4(vel.x + sign * geom.getVelocity(),
                               vel.y,
                               vel.z,
                               __int_as_scalar(mpcd::detail::NO_CELL));
    }
    } // end namespace kernel

/*!
 * \param d_pos Particle positions
 * \param d_vel Particle velocities
 * \param d_tag Particle tags
 * \param geom Slit geometry to fill
 * \param z_min Lower bound to lower fill region
 * \param z_max Upper bound to upper fill region
 * \param box Local simulation box
 * \param mass Mass of fill particles
 * \param type Type of fill particles
 * \param N_lo Number of particles to fill in lower region
 * \param N_hi Number of particles to fill in upper region
 * \param first_tag First tag of filled particles
 * \param first_idx First (local) particle index of filled particles
 * \param kT Temperature for fill particles
 * \param timestep Current timestep
 * \param seed User seed to PRNG for drawing velocities
 * \param block_size Number of threads per block
 *
 * \sa kernel::slit_draw_particles
 */
hipError_t slit_draw_particles(Scalar4* d_pos,
                                Scalar4* d_vel,
                                unsigned int* d_tag,
                                const mpcd::detail::SlitGeometry& geom,
                                const Scalar z_min,
                                const Scalar z_max,
                                const BoxDim& box,
                                const Scalar mass,
                                const unsigned int type,
                                const unsigned int N_lo,
                                const unsigned int N_hi,
                                const unsigned int first_tag,
                                const unsigned int first_idx,
                                const Scalar kT,
                                const uint64_t timestep,
                                const uint16_t seed,
                                const unsigned int block_size)
    {
    const unsigned int N_tot = N_lo + N_hi;
    if (N_tot == 0)
        return hipSuccess;

    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))kernel::slit_draw_particles);
    max_block_size = attr.maxThreadsPerBlock;

    // precompute factor for rescaling the velocities since it is the same for all particles
    const Scalar vel_factor = fast::sqrt(kT / mass);

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(N_tot / run_block_size + 1);
    kernel::slit_draw_particles<<<grid, run_block_size>>>(d_pos,
                                                          d_vel,
                                                          d_tag,
                                                          geom,
                                                          z_min,
                                                          z_max,
                                                          box,
                                                          type,
                                                          N_lo,
                                                          N_tot,
                                                          first_tag,
                                                          first_idx,
                                                          vel_factor,
                                                          timestep,
                                                          seed);

    return hipSuccess;
    }

    } // end namespace gpu
    } // end namespace mpcd
    } // end namespace hoomd
