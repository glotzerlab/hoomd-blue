#include "hip/hip_runtime.h"
// Copyright (c) 2009-2024 The Regents of the University of Michigan.
// Part of HOOMD-blue, released under the BSD 3-Clause License.

#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file NeighborListGPU.cu
    \brief Defines GPU kernel code for neighbor list processing on the GPU
*/

#include "NeighborListGPU.cuh"

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wconversion"
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#pragma GCC diagnostic pop

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
/*! \param d_result Device pointer to a single uint. Will be set to 1 if an update is needed
    \param d_last_pos Particle positions at the time the nlist was last updated
    \param d_pos Current particle positions
    \param nwork Number of particles this GPU processes
    \param box Box dimensions
    \param d_rcut_max The maximum rcut(i,j) that any particle of type i participates in
    \param r_buff The buffer size that particles can move in
    \param ntypes The number of particle types
    \param lambda_min Minimum contraction of deformation tensor
    \param lambda Diagonal deformation tensor (for orthorhombic boundaries)
    \param checkn

    gpu_nlist_needs_update_check_new_kernel() executes one thread per particle. Every particle's
   current position is compared to its last position. If the particle has moved a distance more than
   the buffer width, then *d_result is set to \a checkn.
*/
__global__ void gpu_nlist_needs_update_check_new_kernel(unsigned int* d_result,
                                                        const Scalar4* d_last_pos,
                                                        const Scalar4* d_pos,
                                                        const unsigned int nwork,
                                                        const BoxDim box,
                                                        const Scalar* d_rcut_max,
                                                        const Scalar r_buff,
                                                        const unsigned int ntypes,
                                                        const Scalar lambda_min,
                                                        const Scalar3 lambda,
                                                        const unsigned int checkn,
                                                        const unsigned int offset)
    {
    // each thread will compare vs it's old position to see if the list needs updating
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < nwork)
        {
        // get particle index
        idx += offset;

        Scalar4 cur_postype = d_pos[idx];
        Scalar3 cur_pos = make_scalar3(cur_postype.x, cur_postype.y, cur_postype.z);
        const unsigned int cur_type = __scalar_as_int(cur_postype.w);
        Scalar4 last_postype = d_last_pos[idx];
        Scalar3 last_pos = make_scalar3(last_postype.x, last_postype.y, last_postype.z);

        Scalar3 dx = cur_pos - lambda * last_pos;
        dx = box.minImage(dx);

        const Scalar rmin = __ldg(d_rcut_max + cur_type);
        const Scalar rmax = rmin + r_buff;
        const Scalar delta_max = (rmax * lambda_min - rmin) / Scalar(2.0);
        Scalar maxshiftsq = (delta_max > 0) ? delta_max * delta_max : 0.0f;

        if (dot(dx, dx) >= maxshiftsq)
#if (__CUDA_ARCH__ >= 600)
            atomicMax_system(d_result, checkn);
#else
            atomicMax(d_result, checkn);
#endif
        }
    }

hipError_t gpu_nlist_needs_update_check_new(unsigned int* d_result,
                                            const Scalar4* d_last_pos,
                                            const Scalar4* d_pos,
                                            const unsigned int N,
                                            const BoxDim& box,
                                            const Scalar* d_rcut_max,
                                            const Scalar r_buff,
                                            const unsigned int ntypes,
                                            const Scalar lambda_min,
                                            const Scalar3 lambda,
                                            const unsigned int checkn,
                                            const GPUPartition& gpu_partition)
    {
    unsigned int block_size = 128;

    // iterate over active GPUs in reverse order
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);
        unsigned int nwork = range.second - range.first;

        int n_blocks = nwork / block_size + 1;
        hipLaunchKernelGGL((gpu_nlist_needs_update_check_new_kernel),
                           dim3(n_blocks),
                           dim3(block_size),
                           0,
                           0,
                           d_result,
                           d_last_pos,
                           d_pos,
                           nwork,
                           box,
                           d_rcut_max,
                           r_buff,
                           ntypes,
                           lambda_min,
                           lambda,
                           checkn,
                           range.first);
        }

    return hipSuccess;
    }

//! Number of elements of the exclusion list to process in each batch
const unsigned int FILTER_BATCH_SIZE = 4;

/*! \param d_n_neigh Number of neighbors for each particle (read/write)
    \param d_nlist Neighbor list for each particle (read/write)
    \param nli Indexer for indexing into d_nlist
    \param d_n_ex Number of exclusions for each particle
    \param d_ex_list List of exclusions for each particle
    \param exli Indexer for indexing into d_ex_list
    \param N Number of particles
    \param ex_start Start filtering the nlist from exclusion number \a ex_start

    gpu_nlist_filter_kernel() processes the neighbor list \a d_nlist and removes any entries that
   are excluded. To allow for an arbitrary large number of exclusions, these are processed in batch
   sizes of FILTER_BATCH_SIZE. The kernel must be called multiple times in order to fully remove all
   exclusions from the nlist.

    \note The driver gpu_nlist_filter properly makes as many calls as are necessary, it only needs
   to be called once.

    \b Implementation

    One thread is run for each particle. Exclusions \a ex_start, \a ex_start + 1, ... are loaded in
   for that particle (or the thread returns if there are no exclusions past that point). The thread
   then loops over the neighbor list, comparing each entry to the list of exclusions. If the entry
   is not excluded, it is written back out. \a d_n_neigh is updated to reflect the current number of
   particles in the list at the end of the kernel call.
*/
__global__ void gpu_nlist_filter_kernel(unsigned int* d_n_neigh,
                                        unsigned int* d_nlist,
                                        const size_t* d_head_list,
                                        const unsigned int* d_n_ex,
                                        const unsigned int* d_ex_list,
                                        const Index2D exli,
                                        const unsigned int N,
                                        const unsigned int ex_start)
    {
    // compute the particle index this thread operates on
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // quit now if this thread is processing past the end of the particle list
    if (idx >= N)
        return;

    const unsigned int n_neigh = d_n_neigh[idx];
    const unsigned int n_ex = d_n_ex[idx];
    unsigned int new_n_neigh = 0;

    // quit now if the ex_start flag is past the end of n_ex
    if (ex_start >= n_ex)
        return;

    // count the number of exclusions to process in this thread
    const unsigned int n_ex_process = n_ex - ex_start;

    // load the exclusion list into "local" memory - fully unrolled loops should dump this into
    // registers
    unsigned int l_ex_list[FILTER_BATCH_SIZE];
#pragma unroll
    for (unsigned int cur_ex_idx = 0; cur_ex_idx < FILTER_BATCH_SIZE; cur_ex_idx++)
        {
        if (cur_ex_idx < n_ex_process)
            l_ex_list[cur_ex_idx] = d_ex_list[exli(idx, cur_ex_idx + ex_start)];
        else
            l_ex_list[cur_ex_idx] = 0xffffffff;
        }

    // loop over the list, regenerating it as we go
    const size_t my_head = d_head_list[idx];
    for (unsigned int cur_neigh_idx = 0; cur_neigh_idx < n_neigh; cur_neigh_idx++)
        {
        unsigned int cur_neigh = d_nlist[my_head + cur_neigh_idx];

        // test if excluded
        bool excluded = false;
#pragma unroll
        for (unsigned int cur_ex_idx = 0; cur_ex_idx < FILTER_BATCH_SIZE; cur_ex_idx++)
            {
            if (cur_neigh == l_ex_list[cur_ex_idx])
                excluded = true;
            }

        // add it back to the list if it is not excluded
        if (!excluded)
            {
            if (new_n_neigh != cur_neigh_idx)
                d_nlist[my_head + new_n_neigh] = cur_neigh;
            new_n_neigh++;
            }
        }

    // update the number of neighbors
    d_n_neigh[idx] = new_n_neigh;
    }

hipError_t gpu_nlist_filter(unsigned int* d_n_neigh,
                            unsigned int* d_nlist,
                            const size_t* d_head_list,
                            const unsigned int* d_n_ex,
                            const unsigned int* d_ex_list,
                            const Index2D& exli,
                            const unsigned int N,
                            const unsigned int block_size)
    {
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_nlist_filter_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    // determine parameters for kernel launch
    int n_blocks = N / run_block_size + 1;

    // split the processing of the full exclusion list up into a number of batches
    unsigned int n_batches = (unsigned int)ceil(double(exli.getH()) / double(FILTER_BATCH_SIZE));
    unsigned int ex_start = 0;
    for (unsigned int batch = 0; batch < n_batches; batch++)
        {
        hipLaunchKernelGGL((gpu_nlist_filter_kernel),
                           dim3(n_blocks),
                           dim3(run_block_size),
                           0,
                           0,
                           d_n_neigh,
                           d_nlist,
                           d_head_list,
                           d_n_ex,
                           d_ex_list,
                           exli,
                           N,
                           ex_start);

        ex_start += FILTER_BATCH_SIZE;
        }

    return hipSuccess;
    }

//! GPU kernel to update the exclusions list
__global__ void gpu_update_exclusion_list_kernel(const unsigned int* tags,
                                                 const unsigned int* rtags,
                                                 const unsigned int* n_ex_tag,
                                                 const unsigned int* ex_list_tag,
                                                 const Index2D ex_list_tag_indexer,
                                                 unsigned int* n_ex_idx,
                                                 unsigned int* ex_list_idx,
                                                 const Index2D ex_list_indexer,
                                                 const unsigned int N)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    unsigned int tag = tags[idx];

    unsigned int n = n_ex_tag[tag];

    // copy over number of exclusions
    n_ex_idx[idx] = n;

    for (unsigned int offset = 0; offset < n; offset++)
        {
        unsigned int ex_tag = ex_list_tag[ex_list_tag_indexer(tag, offset)];
        unsigned int ex_idx = rtags[ex_tag];

        ex_list_idx[ex_list_indexer(idx, offset)] = ex_idx;
        }
    }

//! GPU function to update the exclusion list on the device
/*! \param d_tag Array of particle tags
    \param d_rtag Array of reverse-lookup tag->idx
    \param d_n_ex_tag List of number of exclusions per tag
    \param d_ex_list_tag 2D Exclusion list per tag
    \param ex_list_tag_indexer Indexer for per-tag exclusion list
    \param d_n_ex_idx List of number of exclusions per idx
    \param d_ex_list_idx Exclusion list per idx
    \param ex_list_indexer Indexer for per-idx exclusion list
    \param N number of particles
 */
hipError_t gpu_update_exclusion_list(const unsigned int* d_tag,
                                     const unsigned int* d_rtag,
                                     const unsigned int* d_n_ex_tag,
                                     const unsigned int* d_ex_list_tag,
                                     const Index2D& ex_list_tag_indexer,
                                     unsigned int* d_n_ex_idx,
                                     unsigned int* d_ex_list_idx,
                                     const Index2D& ex_list_indexer,
                                     const unsigned int N)
    {
    unsigned int block_size = 256;

    hipLaunchKernelGGL((gpu_update_exclusion_list_kernel),
                       dim3(N / block_size + 1),
                       dim3(block_size),
                       0,
                       0,
                       d_tag,
                       d_rtag,
                       d_n_ex_tag,
                       d_ex_list_tag,
                       ex_list_tag_indexer,
                       d_n_ex_idx,
                       d_ex_list_idx,
                       ex_list_indexer,
                       N);

    return hipSuccess;
    }

//! GPU kernel to do a preliminary sizing on particles
/*!
 * \param d_head_list The head list of indexes to overwrite
 * \param d_req_size_nlist Flag for the required size of the neighbor list to overwrite
 * \param d_Nmax The number of neighbors to size per particle type
 * \param d_pos Particle positions and types
 * \param N the number of particles on this rank
 * \param ntypes the number of types in the system
 *
 * This kernel initializes the head list with the number of neighbors that each type expects from
 * d_Nmax. A prefix sum is then performed in gpu_nlist_build_head_list() to accumulate starting
 * indices.
 */
__global__ void gpu_nlist_init_head_list_kernel(size_t* d_head_list,
                                                size_t* d_req_size_nlist,
                                                const unsigned int* d_Nmax,
                                                const Scalar4* d_pos,
                                                const unsigned int N,
                                                const unsigned int ntypes)
    {
    // particle index
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per particle
    if (idx >= N)
        return;

    const Scalar4 postype_i = d_pos[idx];
    const unsigned int type_i = __scalar_as_int(postype_i.w);
    const unsigned int Nmax_i = __ldg(d_Nmax + type_i);

    d_head_list[idx] = Nmax_i;

    // last thread presets its number of particles in the memory req as well
    if (idx == (N - 1))
        {
        *d_req_size_nlist = Nmax_i;
        }
    }

/*!
 * \param d_req_size_nlist Flag for the total size of the neighbor list
 * \param d_head_list The complete particle head list
 * \param N the number of particles on this rank
 *
 * A single thread on the device is needed to complete the exclusive scan and find the size of the
 * neighbor list. Because gpu_nlist_init_head_list_kernel() already set the number of neighbors for
 * the last particle in d_req_size_nlist, the head index of the last particle is added to this
 * number to get the total size.
 */
__global__ void gpu_nlist_get_nlist_size_kernel(size_t* d_req_size_nlist,
                                                const size_t* d_head_list,
                                                const unsigned int N)
    {
    *d_req_size_nlist += d_head_list[N - 1];
    }

/*!
 * \param d_head_list The head list of indexes to compute for reading the neighbor list
 * \param d_req_size_nlist Flag for the total size of the neighbor list
 * \param d_Nmax The number of neighbors to size per particle type
 * \param d_pos Particle positions and types
 * \param N the number of particles on this rank
 * \param ntypes the number of types in the system
 * \param block_size Number of threads per block for gpu_nlist_init_head_list_kernel()
 *
 * \return hipSuccess on completion
 *
 * \b Implementation
 * \a d_head_list is filled with the number of neighbors per particle. An exclusive prefix sum is
 * performed in place on \a d_head_list using the thrust libraries and a single thread is used to
 * perform compute the total size of the neighbor list while still on device.
 */
hipError_t gpu_nlist_build_head_list(size_t* d_head_list,
                                     size_t* d_req_size_nlist,
                                     const unsigned int* d_Nmax,
                                     const Scalar4* d_pos,
                                     const unsigned int N,
                                     const unsigned int ntypes,
                                     const unsigned int block_size)
    {
    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_nlist_init_head_list_kernel);
    max_block_size = attr.maxThreadsPerBlock;

    unsigned int run_block_size = min(block_size, max_block_size);

    // initialize each particle with its number of neighbors
    hipLaunchKernelGGL((gpu_nlist_init_head_list_kernel),
                       dim3(N / run_block_size + 1),
                       dim3(run_block_size),
                       0,
                       0,
                       d_head_list,
                       d_req_size_nlist,
                       d_Nmax,
                       d_pos,
                       N,
                       ntypes);

    thrust::device_ptr<size_t> t_head_list = thrust::device_pointer_cast(d_head_list);
    thrust::exclusive_scan(t_head_list, t_head_list + N, t_head_list);

    hipLaunchKernelGGL((gpu_nlist_get_nlist_size_kernel),
                       dim3(1),
                       dim3(1),
                       0,
                       0,
                       d_req_size_nlist,
                       d_head_list,
                       N);

    return hipSuccess;
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
