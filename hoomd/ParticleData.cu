#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: jglaser

#include "ParticleData.cuh"

/*! \file ParticleData.cu
    \brief ImplementsGPU kernel code and data structure functions used by ParticleData
*/

#ifdef ENABLE_MPI

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wconversion"
#include <hipcub/hipcub.hpp>

#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scatter.h>
#pragma GCC diagnostic pop

namespace hoomd
    {
namespace kernel
    {
//! Kernel to partition particle data
__global__ void gpu_scatter_particle_data_kernel(const unsigned int nwork,
                                                 const Scalar4* d_pos,
                                                 const Scalar4* d_vel,
                                                 const Scalar3* d_accel,
                                                 const Scalar* d_charge,
                                                 const Scalar* d_diameter,
                                                 const int3* d_image,
                                                 const unsigned int* d_body,
                                                 const Scalar4* d_orientation,
                                                 const Scalar4* d_angmom,
                                                 const Scalar3* d_inertia,
                                                 const Scalar4* d_net_force,
                                                 const Scalar4* d_net_torque,
                                                 const Scalar* d_net_virial,
                                                 unsigned int net_virial_pitch,
                                                 const unsigned int* d_tag,
                                                 unsigned int* d_rtag,
                                                 Scalar4* d_pos_alt,
                                                 Scalar4* d_vel_alt,
                                                 Scalar3* d_accel_alt,
                                                 Scalar* d_charge_alt,
                                                 Scalar* d_diameter_alt,
                                                 int3* d_image_alt,
                                                 unsigned int* d_body_alt,
                                                 Scalar4* d_orientation_alt,
                                                 Scalar4* d_angmom_alt,
                                                 Scalar3* d_inertia_alt,
                                                 Scalar4* d_net_force_alt,
                                                 Scalar4* d_net_torque_alt,
                                                 Scalar* d_net_virial_alt,
                                                 unsigned int* d_tag_alt,
                                                 detail::pdata_element* d_out,
                                                 unsigned int* d_comm_flags,
                                                 unsigned int* d_comm_flags_out,
                                                 const unsigned int* d_scan,
                                                 const unsigned int offset)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= nwork)
        return;
    idx += offset;
    bool remove = d_comm_flags[idx];

    unsigned int scan_remove = d_scan[idx];
    unsigned int scan_keep = idx - scan_remove;

    if (remove)
        {
        detail::pdata_element p;
        p.pos = d_pos[idx];
        p.vel = d_vel[idx];
        p.accel = d_accel[idx];
        p.charge = d_charge[idx];
        p.diameter = d_diameter[idx];
        p.image = d_image[idx];
        p.body = d_body[idx];
        p.orientation = d_orientation[idx];
        p.angmom = d_angmom[idx];
        p.inertia = d_inertia[idx];
        p.net_force = d_net_force[idx];
        p.net_torque = d_net_torque[idx];
        for (unsigned int j = 0; j < 6; ++j)
            p.net_virial[j] = d_net_virial[j * net_virial_pitch + idx];
        p.tag = d_tag[idx];
        d_out[scan_remove] = p;
        d_comm_flags_out[scan_remove] = d_comm_flags[idx];

        // reset communication flags
        d_comm_flags[idx] = 0;

        // reset rtag
        d_rtag[p.tag] = NOT_LOCAL;
        }
    else
        {
        d_pos_alt[scan_keep] = d_pos[idx];
        d_vel_alt[scan_keep] = d_vel[idx];
        d_accel_alt[scan_keep] = d_accel[idx];
        d_charge_alt[scan_keep] = d_charge[idx];
        d_diameter_alt[scan_keep] = d_diameter[idx];
        d_image_alt[scan_keep] = d_image[idx];
        d_body_alt[scan_keep] = d_body[idx];
        d_orientation_alt[scan_keep] = d_orientation[idx];
        d_angmom_alt[scan_keep] = d_angmom[idx];
        d_inertia_alt[scan_keep] = d_inertia[idx];
        d_net_force_alt[scan_keep] = d_net_force[idx];
        d_net_torque_alt[scan_keep] = d_net_torque[idx];
        for (unsigned int j = 0; j < 6; ++j)
            d_net_virial_alt[j * net_virial_pitch + scan_keep]
                = d_net_virial[j * net_virial_pitch + idx];
        unsigned int tag = d_tag[idx];
        d_tag_alt[scan_keep] = tag;

        // update rtag
        d_rtag[tag] = scan_keep;
        }
    }

__global__ void
gpu_select_sent_particles(unsigned int N, unsigned int* d_comm_flags, unsigned int* d_tmp)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;
    d_tmp[idx] = d_comm_flags[idx] ? 1 : 0;
    }

/*! \param N Number of local particles
    \param d_pos Device array of particle positions
    \param d_vel Device array of particle velocities
    \param d_accel Device array of particle accelerations
    \param d_charge Device array of particle charges
    \param d_diameter Device array of particle diameters
    \param d_image Device array of particle images
    \param d_body Device array of particle body tags
    \param d_orientation Device array of particle orientations
    \param d_angmom Device array of particle angular momenta
    \param d_inertia Device array of particle moments of inertia
    \param d_net_force Net force
    \param d_net_torque Net torque
    \param d_net_virial Net virial
    \param net_virial_pitch Pitch of net virial array
    \param d_tag Device array of particle tags
    \param d_rtag Device array for reverse-lookup table
    \param d_pos_alt Device array of particle positions (output)
    \param d_vel_alt Device array of particle velocities (output)
    \param d_accel_alt Device array of particle accelerations (output)
    \param d_charge_alt Device array of particle charges (output)
    \param d_diameter_alt Device array of particle diameters (output)
    \param d_image_alt Device array of particle images (output)
    \param d_body_alt Device array of particle body tags (output)
    \param d_orientation_alt Device array of particle orientations (output)
    \param d_angmom_alt Device array of particle angular momenta (output)
    \param d_inertia Device array of particle moments of inertia (output)
    \param d_net_force Net force (output)
    \param d_net_torque Net torque (output)
    \param d_net_virial Net virial (output)
    \param d_out Output array for packed particle data
    \param max_n_out Maximum number of elements to write to output array

    \returns Number of elements marked for removal
 */
unsigned int gpu_pdata_remove(const unsigned int N,
                              const Scalar4* d_pos,
                              const Scalar4* d_vel,
                              const Scalar3* d_accel,
                              const Scalar* d_charge,
                              const Scalar* d_diameter,
                              const int3* d_image,
                              const unsigned int* d_body,
                              const Scalar4* d_orientation,
                              const Scalar4* d_angmom,
                              const Scalar3* d_inertia,
                              const Scalar4* d_net_force,
                              const Scalar4* d_net_torque,
                              const Scalar* d_net_virial,
                              unsigned int net_virial_pitch,
                              const unsigned int* d_tag,
                              unsigned int* d_rtag,
                              Scalar4* d_pos_alt,
                              Scalar4* d_vel_alt,
                              Scalar3* d_accel_alt,
                              Scalar* d_charge_alt,
                              Scalar* d_diameter_alt,
                              int3* d_image_alt,
                              unsigned int* d_body_alt,
                              Scalar4* d_orientation_alt,
                              Scalar4* d_angmom_alt,
                              Scalar3* d_inertia_alt,
                              Scalar4* d_net_force_alt,
                              Scalar4* d_net_torque_alt,
                              Scalar* d_net_virial_alt,
                              unsigned int* d_tag_alt,
                              detail::pdata_element* d_out,
                              unsigned int* d_comm_flags,
                              unsigned int* d_comm_flags_out,
                              unsigned int max_n_out,
                              unsigned int* d_tmp,
                              CachedAllocator& alloc,
                              GPUPartition& gpu_partition)
    {
    if (!N)
        return 0;

    assert(d_pos);
    assert(d_vel);
    assert(d_accel);
    assert(d_charge);
    assert(d_diameter);
    assert(d_image);
    assert(d_body);
    assert(d_orientation);
    assert(d_angmom);
    assert(d_inertia);
    assert(d_net_force);
    assert(d_net_torque);
    assert(d_net_virial);
    assert(d_tag);
    assert(d_rtag);
    assert(d_pos_alt);
    assert(d_vel_alt);
    assert(d_accel_alt);
    assert(d_charge_alt);
    assert(d_diameter_alt);
    assert(d_image_alt);
    assert(d_body_alt);
    assert(d_orientation_alt);
    assert(d_angmom_alt);
    assert(d_inertia_alt);
    assert(d_net_force_alt);
    assert(d_net_torque_alt);
    assert(d_net_virial_alt);
    assert(d_tag_alt);
    assert(d_out);
    assert(d_comm_flags);
    assert(d_comm_flags_out);
    assert(d_tmp);

    unsigned int n_out;

    // partition particle data into local and removed particles
    unsigned int block_size = 256;
    unsigned int n_blocks = N / block_size + 1;

    // select nonzero communication flags
    hipLaunchKernelGGL(gpu_select_sent_particles,
                       dim3(n_blocks),
                       dim3(block_size),
                       0,
                       0,
                       N,
                       d_comm_flags,
                       d_tmp);

    // perform a scan over the array of ones and zeroes
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    // determine size of temporary storage
    unsigned int* d_scan = alloc.getTemporaryBuffer<unsigned int>(N);
    assert(d_scan);

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_tmp, d_scan, N);

    d_temp_storage = alloc.getTemporaryBuffer<char>(temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_tmp, d_scan, N);
    alloc.deallocate((char*)d_temp_storage);

    // determine total number of sent particles
    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    unsigned int* d_n_out = (unsigned int*)alloc.getTemporaryBuffer<unsigned int>(1);
    assert(d_n_out);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_tmp, d_n_out, N);
    d_temp_storage = alloc.allocate(temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_tmp, d_n_out, N);
    alloc.deallocate((char*)d_temp_storage);
    hipMemcpy(&n_out, d_n_out, sizeof(unsigned int), hipMemcpyDeviceToHost);
    alloc.deallocate((char*)d_n_out);

    // Don't write past end of buffer
    if (n_out <= max_n_out)
        {
        // partition particle data into local and removed particles
        for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
            {
            auto range = gpu_partition.getRangeAndSetGPU(idev);

            unsigned int nwork = range.second - range.first;
            unsigned int offset = range.first;

            unsigned int block_size = 256;
            unsigned int n_blocks = nwork / block_size + 1;

            hipLaunchKernelGGL(gpu_scatter_particle_data_kernel,
                               dim3(n_blocks),
                               dim3(block_size),
                               0,
                               0,
                               nwork,
                               d_pos,
                               d_vel,
                               d_accel,
                               d_charge,
                               d_diameter,
                               d_image,
                               d_body,
                               d_orientation,
                               d_angmom,
                               d_inertia,
                               d_net_force,
                               d_net_torque,
                               d_net_virial,
                               net_virial_pitch,
                               d_tag,
                               d_rtag,
                               d_pos_alt,
                               d_vel_alt,
                               d_accel_alt,
                               d_charge_alt,
                               d_diameter_alt,
                               d_image_alt,
                               d_body_alt,
                               d_orientation_alt,
                               d_angmom_alt,
                               d_inertia_alt,
                               d_net_force_alt,
                               d_net_torque_alt,
                               d_net_virial_alt,
                               d_tag_alt,
                               d_out,
                               d_comm_flags,
                               d_comm_flags_out,
                               d_scan,
                               offset);
            }
        }

    // free temp buf
    alloc.deallocate((char*)d_scan);

    // return elements written to output stream
    return n_out;
    }

__global__ void gpu_pdata_add_particles_kernel(unsigned int old_nparticles,
                                               unsigned int num_add_ptls,
                                               Scalar4* d_pos,
                                               Scalar4* d_vel,
                                               Scalar3* d_accel,
                                               Scalar* d_charge,
                                               Scalar* d_diameter,
                                               int3* d_image,
                                               unsigned int* d_body,
                                               Scalar4* d_orientation,
                                               Scalar4* d_angmom,
                                               Scalar3* d_inertia,
                                               Scalar4* d_net_force,
                                               Scalar4* d_net_torque,
                                               Scalar* d_net_virial,
                                               unsigned int net_virial_pitch,
                                               unsigned int* d_tag,
                                               unsigned int* d_rtag,
                                               const detail::pdata_element* d_in,
                                               unsigned int* d_comm_flags)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= num_add_ptls)
        return;

    detail::pdata_element p = d_in[idx];

    unsigned int add_idx = old_nparticles + idx;
    d_pos[add_idx] = p.pos;
    d_vel[add_idx] = p.vel;
    d_accel[add_idx] = p.accel;
    d_charge[add_idx] = p.charge;
    d_diameter[add_idx] = p.diameter;
    d_image[add_idx] = p.image;
    d_body[add_idx] = p.body;
    d_orientation[add_idx] = p.orientation;
    d_angmom[add_idx] = p.angmom;
    d_inertia[add_idx] = p.inertia;
    d_net_force[add_idx] = p.net_force;
    d_net_torque[add_idx] = p.net_torque;
    for (unsigned int j = 0; j < 6; ++j)
        d_net_virial[j * net_virial_pitch + add_idx] = p.net_virial[j];
    d_tag[add_idx] = p.tag;
    d_rtag[p.tag] = add_idx;
    d_comm_flags[add_idx] = 0;
    }

/*! \param old_nparticles old local particle count
    \param num_add_ptls Number of particles in input array
    \param d_pos Device array of particle positions
    \param d_vel Device iarray of particle velocities
    \param d_accel Device array of particle accelerations
    \param d_charge Device array of particle charges
    \param d_diameter Device array of particle diameters
    \param d_image Device array of particle images
    \param d_body Device array of particle body tags
    \param d_orientation Device array of particle orientations
    \param d_angmom Device array of particle angular momenta
    \param d_inertia Device array of particle moments of inertia
    \param d_net_force Net force
    \param d_net_torque Net torque
    \param d_net_virial Net virial
    \param d_tag Device array of particle tags
    \param d_rtag Device array for reverse-lookup table
    \param d_in Device array of packed input particle data
    \param d_comm_flags Device array of communication flags (pdata)
*/
void gpu_pdata_add_particles(const unsigned int old_nparticles,
                             const unsigned int num_add_ptls,
                             Scalar4* d_pos,
                             Scalar4* d_vel,
                             Scalar3* d_accel,
                             Scalar* d_charge,
                             Scalar* d_diameter,
                             int3* d_image,
                             unsigned int* d_body,
                             Scalar4* d_orientation,
                             Scalar4* d_angmom,
                             Scalar3* d_inertia,
                             Scalar4* d_net_force,
                             Scalar4* d_net_torque,
                             Scalar* d_net_virial,
                             unsigned int net_virial_pitch,
                             unsigned int* d_tag,
                             unsigned int* d_rtag,
                             const detail::pdata_element* d_in,
                             unsigned int* d_comm_flags)
    {
    assert(d_pos);
    assert(d_vel);
    assert(d_accel);
    assert(d_charge);
    assert(d_diameter);
    assert(d_image);
    assert(d_body);
    assert(d_orientation);
    assert(d_angmom);
    assert(d_inertia);
    assert(d_net_force);
    assert(d_net_torque);
    assert(d_net_virial);
    assert(d_tag);
    assert(d_rtag);
    assert(d_in);

    unsigned int block_size = 256;
    unsigned int n_blocks = num_add_ptls / block_size + 1;

    hipLaunchKernelGGL(gpu_pdata_add_particles_kernel,
                       dim3(n_blocks),
                       dim3(block_size),
                       0,
                       0,
                       old_nparticles,
                       num_add_ptls,
                       d_pos,
                       d_vel,
                       d_accel,
                       d_charge,
                       d_diameter,
                       d_image,
                       d_body,
                       d_orientation,
                       d_angmom,
                       d_inertia,
                       d_net_force,
                       d_net_torque,
                       d_net_virial,
                       net_virial_pitch,
                       d_tag,
                       d_rtag,
                       d_in,
                       d_comm_flags);
    }

    } // end namespace kernel

    } // end namespace hoomd

#endif // ENABLE_MPI
