// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file EwaldDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "AllDriverPotentialPairGPU.cuh"
#include "EvaluatorPairEwald.h"

namespace hoomd
    {
namespace md
    {
namespace kernel
    {
hipError_t gpu_compute_ewald_forces(const pair_args_t& pair_args,
                                    const EvaluatorPairEwald::param_type* d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairEwald>(pair_args, d_params);
    }

    } // end namespace kernel
    } // end namespace md
    } // end namespace hoomd
